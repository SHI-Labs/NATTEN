#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 Ali Hassani.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 **************************************************************************************************/
/*! \file
    \brief Neighborhood-Neighborhood kernel for 1D data.
           Applies neighborhood attention weights to neighborhood values.
*/

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/cuda/HIPContext.h>
#include <ATen/ATen.h>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/AccumulateType.h>
#include <hip/hip_fp16.h>

#include "natten_commons.cuh"

namespace natten {

template<class scalar_t>
using Tensor4D = typename torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits>;

template <int KS, int NS, int DILATION, typename scalar_t>
__global__ void neighborhood_neighborhood_1d(           // AV     / Q-grad
    const Tensor4D<scalar_t> weights,                   // attn   / d_attn
    const Tensor4D<scalar_t> values,                    // value  / key
    Tensor4D<scalar_t> output,                          // output / d_query
    const int length,
    const int heads,
    const int kernel_size_in,
    const int dilation_in,
    const int dim,
    const int totalElements) {
    const int KERNEL_SIZE = (KS>1) ? KS : kernel_size_in;
    const int NEIGHBORHOOD_SIZE = (NS>0) ? NS : KERNEL_SIZE / 2;
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < totalElements){
        int indtmp1 = linearIndex/dim;
        const int d = linearIndex - indtmp1 * dim;
        int indtmp2 = indtmp1/length;
        const int i = indtmp1 - indtmp2 * length;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/heads;
        const int h = indtmp1 - indtmp2 * heads;
        const int b = indtmp2;

        const int ni = get_window_start(i, length, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        scalar_t updt = scalar_t(0);
        int weightsOffset = b * weights.stride(0) + h * weights.stride(1) + i * weights.stride(2);
        const int valuesOffset = b * values.stride(0) + h * values.stride(1) + d;
        #pragma unroll
        for (int xi=ni; xi < ni + KERNEL_SIZE * dilation; xi+=dilation){
            const int valuesIndex = valuesOffset + xi * values.stride(2);
            updt += weights.data()[weightsOffset] * values.data()[valuesIndex];
            ++weightsOffset;
        }
        output.data()[linearIndex] = updt;
    }
}

template <int KS, int NS, int DILATION, typename scalar_t>
__global__ void neighborhood_neighborhood_1d_fp16(           // AV     / Q-grad
    const Tensor4D<scalar_t> weights,                        // attn   / d_attn
    const Tensor4D<scalar_t> values,                         // value  / key
    Tensor4D<scalar_t> output,                               // output / d_query
    const int length,
    const int heads,
    const int kernel_size_in,
    const int dilation_in,
    const int dimhalf,
    const int totalElements) {
    const int KERNEL_SIZE = (KS>1) ? KS : kernel_size_in;
    const int NEIGHBORHOOD_SIZE = (NS>0) ? NS : KERNEL_SIZE / 2;
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < totalElements){
        __half2* values2 = reinterpret_cast<__half2*>(values.data());
        __half2* output2 = reinterpret_cast<__half2*>(output.data());
        int indtmp1 = linearIndex/dimhalf;
        const int d = linearIndex - indtmp1 * dimhalf;
        int indtmp2 = indtmp1/length;
        const int i = indtmp1 - indtmp2 * length;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/heads;
        const int h = indtmp1 - indtmp2 * heads;
        const int b = indtmp2;

        const int ni = get_window_start(i, length, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        __half2 updt = __float2half2_rn(0.f);
        int weightsOffset = b * weights.stride(0) + h * weights.stride(1) + i * weights.stride(2);
        const int valuesOffset = b * (dimhalf * length * heads) + h * (dimhalf * length) + d;
        #pragma unroll
        for (int xi=ni; xi < ni + KERNEL_SIZE * dilation; xi+=dilation){
            const int valuesIndex = valuesOffset + xi * dimhalf;
            scalar_t a = weights.data()[weightsOffset];
            updt = __hfma2(__halves2half2(a, a), values2[valuesIndex], updt);
            ++weightsOffset;
        }
        output2[linearIndex] = updt;
    }
}

} // namespace natten
