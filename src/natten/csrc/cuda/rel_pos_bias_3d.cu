#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 Ali Hassani.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 **************************************************************************************************/
/*! \file
    \brief Relative positional bias backward pass kernel for 3D data.
*/

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/cuda/HIPContext.h>
#include <ATen/ATen.h>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/AccumulateType.h>
#include <hip/hip_fp16.h>

#include "natten_commons.cuh"

namespace natten {

template <int KERNEL_SIZE, int KERNEL_SIZE_D, int NEIGHBORHOOD_SIZE, int NEIGHBORHOOD_SIZE_D, typename scalar_t>
__global__ void rel_pos_bias_gradient_3d(
    torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_rpb,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::DefaultPtrTraits> d_attn,
    const int depth,
    const int height,
    const int width,
    const int dilation,
    const int dilation_d,
    const int batch_size,
    const int d_rpb_numel,
    const int totalThreads) {
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < totalThreads){
        int indtmp1 = linearIndex/KERNEL_SIZE;
        const int kj = linearIndex - indtmp1 * KERNEL_SIZE;
        int indtmp2 = indtmp1/KERNEL_SIZE;
        const int ki = indtmp1 - indtmp2 * KERNEL_SIZE;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/KERNEL_SIZE_D;
        const int kk = indtmp1 - indtmp2 * KERNEL_SIZE_D;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/width;
        const int j = indtmp1 - indtmp2 * width;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/height;
        const int i = indtmp1 - indtmp2 * height;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/depth;
        const int k = indtmp1 - indtmp2 * depth;
        const int h = indtmp2;
        const int pi = get_pb_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pk = get_pb_start(k, depth, KERNEL_SIZE_D, NEIGHBORHOOD_SIZE_D, dilation_d);
        scalar_t d_rpb_update = scalar_t(0);
        int attnOffset = h * d_attn.stride(1) + k * d_attn.stride(2) + i * d_attn.stride(3) + j * d_attn.stride(4) + ((kk*KERNEL_SIZE*KERNEL_SIZE)+(ki*KERNEL_SIZE)+kj);
        #pragma unroll
        for (int b=0; b < batch_size; ++b){
            d_rpb_update += d_attn.data()[attnOffset];
            attnOffset += d_attn.stride(0);
        }
        const int index = h * d_rpb.stride(0) + (pk+kk) * d_rpb.stride(1) + (pi+ki) * d_rpb.stride(2) + (pj+kj) * d_rpb.stride(3);
        at::native::fastAtomicAdd(d_rpb.data(), index, d_rpb_numel, static_cast<scalar_t>(d_rpb_update), true);
    }
}

template <int KERNEL_SIZE, int KERNEL_SIZE_D, int NEIGHBORHOOD_SIZE, int NEIGHBORHOOD_SIZE_D, typename scalar_t>
__global__ void rel_pos_bias_gradient_3d_fp16(
    torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_rpb,
    const torch::PackedTensorAccessor32<scalar_t,6,torch::DefaultPtrTraits> d_attn,
    const int depth,
    const int height,
    const int width,
    const int dilation,
    const int dilation_d,
    const int batch_size,
    const int d_rpb_numel,
    const int totalThreads) {
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < totalThreads){
        int indtmp1 = linearIndex/KERNEL_SIZE;
        const int kj = linearIndex - indtmp1 * KERNEL_SIZE;
        int indtmp2 = indtmp1/KERNEL_SIZE;
        const int ki = indtmp1 - indtmp2 * KERNEL_SIZE;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/KERNEL_SIZE_D;
        const int kk = indtmp1 - indtmp2 * KERNEL_SIZE_D;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/width;
        const int j = indtmp1 - indtmp2 * width;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/height;
        const int i = indtmp1 - indtmp2 * height;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/depth;
        const int k = indtmp1 - indtmp2 * depth;
        const int h = indtmp2;
        const int pi = get_pb_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pk = get_pb_start(k, depth, KERNEL_SIZE_D, NEIGHBORHOOD_SIZE_D, dilation_d);
        float d_rpb_update = float(0.0);
        int attnOffset = h * d_attn.stride(1) + k * d_attn.stride(2) + i * d_attn.stride(3) + j * d_attn.stride(4) + ((kk*KERNEL_SIZE*KERNEL_SIZE)+(ki*KERNEL_SIZE)+kj);
        #pragma unroll
        for (int b=0; b < batch_size; ++b){
            d_rpb_update += static_cast<float>(d_attn.data()[attnOffset]);
            attnOffset += d_attn.stride(0);
        }
        const int index = h * d_rpb.stride(0) + (pk+kk) * d_rpb.stride(1) + (pi+ki) * d_rpb.stride(2) + (pj+kj) * d_rpb.stride(3);
        at::native::fastAtomicAdd(d_rpb.data(), index, d_rpb_numel, static_cast<scalar_t>(d_rpb_update), true);
    }
}

} // namespace natten
