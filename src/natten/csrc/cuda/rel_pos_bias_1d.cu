#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 Ali Hassani.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 **************************************************************************************************/
/*! \file
    \brief Relative positional bias backward pass kernel for 1D data.
*/

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/cuda/HIPContext.h>
#include <ATen/ATen.h>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/AccumulateType.h>
#include <hip/hip_fp16.h>

#include "natten_commons.cuh"

namespace natten {

template <int KS, int NS, int DILATION, typename scalar_t>
__global__ void rel_pos_bias_gradient_1d(
    torch::PackedTensorAccessor32<scalar_t,2,torch::DefaultPtrTraits> d_rpb,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_attn,
    const int length,
    const int kernel_size_in,
    const int dilation_in,
    const int batch_size,
    const int d_rpb_numel,
    const int totalThreads) {
    const int KERNEL_SIZE = (KS>1) ? KS : kernel_size_in;
    const int NEIGHBORHOOD_SIZE = (NS>0) ? NS : KERNEL_SIZE / 2;
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < totalThreads){
        int indtmp1 = linearIndex/KERNEL_SIZE;
        const int ki = linearIndex - indtmp1 * KERNEL_SIZE;
        const int h = indtmp1/length;
        const int i = indtmp1 - h * length;
        const int pi = get_pb_start(i, length, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        scalar_t d_rpb_update = scalar_t(0);
        int attnOffset = h * d_attn.stride(1) + i * d_attn.stride(2) + ki;
        #pragma unroll
        for (int b=0; b < batch_size; ++b){
            d_rpb_update += static_cast<float>(d_attn.data()[attnOffset]);
            attnOffset += d_attn.stride(0);
        }
        const int index = h * d_rpb.stride(0) + (pi+ki) * d_rpb.stride(1);
        at::native::fastAtomicAdd(d_rpb.data(), index, d_rpb_numel, static_cast<scalar_t>(d_rpb_update), true);
    }
}

template <int KS, int NS, int DILATION, typename scalar_t>
__global__ void rel_pos_bias_gradient_1d_fp16(
    torch::PackedTensorAccessor32<scalar_t,2,torch::DefaultPtrTraits> d_rpb,
    const torch::PackedTensorAccessor32<scalar_t,4,torch::DefaultPtrTraits> d_attn,
    const int length,
    const int kernel_size_in,
    const int dilation_in,
    const int batch_size,
    const int d_rpb_numel,
    const int totalThreads) {
    const int KERNEL_SIZE = (KS>1) ? KS : kernel_size_in;
    const int NEIGHBORHOOD_SIZE = (NS>0) ? NS : KERNEL_SIZE / 2;
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < totalThreads){
        int indtmp1 = linearIndex/KERNEL_SIZE;
        const int ki = linearIndex - indtmp1 * KERNEL_SIZE;
        const int h = indtmp1/length;
        const int i = indtmp1 - h * length;
        const int pi = get_pb_start(i, length, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        float d_rpb_update = scalar_t(0);
        int attnOffset = h * d_attn.stride(1) + i * d_attn.stride(2) + ki;
        #pragma unroll
        for (int b=0; b < batch_size; ++b){
            d_rpb_update += static_cast<float>(d_attn.data()[attnOffset]);
            attnOffset += d_attn.stride(0);
        }
        const int index = h * d_rpb.stride(0) + (pi+ki) * d_rpb.stride(1);
        at::native::fastAtomicAdd(d_rpb.data(), index, d_rpb_numel, static_cast<scalar_t>(d_rpb_update), true);
    }
}

} // namespace natten
