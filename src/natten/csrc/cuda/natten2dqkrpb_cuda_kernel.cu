#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 Ali Hassani.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 **************************************************************************************************/

#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <ATen/cuda/HIPContext.h>
#include <ATen/ATen.h>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/AccumulateType.h>
#include <hip/hip_fp16.h>

#include "natten_commons.cuh"

namespace natten {

#define WARP_SIZE 32

#define KERNEL_SIZE_13 13
#define KERNEL_SIZE_11 11
#define KERNEL_SIZE_9 9
#define KERNEL_SIZE_7 7
#define KERNEL_SIZE_5 5
#define KERNEL_SIZE_3 3
#define NEIGHBORHOOD_SIZE_13 6
#define NEIGHBORHOOD_SIZE_11 5
#define NEIGHBORHOOD_SIZE_9 4
#define NEIGHBORHOOD_SIZE_7 3
#define NEIGHBORHOOD_SIZE_5 2
#define NEIGHBORHOOD_SIZE_3 1
// Always keep batchthreads 1, because we want each thread block to process one 1 sample 1 head
#define BATCHTHREADS_13 1
#define BATCHTHREADS_11 1
#define BATCHTHREADS_9 1
#define BATCHTHREADS_7 1
#define BATCHTHREADS_5 1
#define BATCHTHREADS_3 1
// Tile is the number of pixels across each axis that are processed within a single threadblock
// So far the best tile size for Kernel size 7 is 3x3.
#define TILE_9 3
#define TILE_7 3
#define TILE_5 4
#define TILE_3 7

#define TILE_11_X 2
#define TILE_11_Y 3
#define TILE_13_X 2
#define TILE_13_Y 3
// Each of the 3x3 pixels has 7x7 key neighbors in this case, therefore the tile size for keys will 7 + 3 - 1 = 9x9
#define KTILE_9 11
#define KTILE_7 9
#define KTILE_5 8
#define KTILE_3 9

#define KTILE_11_X 12
#define KTILE_11_Y 13
#define KTILE_13_X 14
#define KTILE_13_Y 15
// 7x7 kernel, and we want each threadblock to process the entire neighborhood for each QUERY in its tile,
// so we'll have 7x7 * 3x3 = 21x21
// Also keep in mind these 21 threads are across each axis, so it's 21x21 threads total
// 21x21 = 441 < 1024
// Ensure it's less than 1024, which is the max number of threads per threadblock
#define XYTHREADS_9 27
#define XYTHREADS_7 21
#define XYTHREADS_5 20
#define XYTHREADS_3 21

#define XTHREADS_11 33
#define YTHREADS_11 22
#define XTHREADS_13 39
#define YTHREADS_13 26

// DIM is fixed at 32 for now
#define DIM_32 32
#define DIMHALF_32 16 // FP16 stored in half2 => half the dims
// There's 32 * 3x3 QUERY cells to store, and 32 * 10x10 KEY cells
// The former is 288 < 441 threads, so each thread can copy over one QUERY cell exactly, and we'll have empty threads too
// But that's not the case for the latter, which is 3200 and it's not < 441
// But we can have each thread load more cells instead. 8 is optimal since it will maximize utility
// So copy 8 dims per KEY pixel in each thread
#define KITERS_32 8
#define KHALFITERS_32 4 // FP16 stored in half2 => half the dims
// and DIM = 32 / 8 = 4, hence 4 is the stride.
#define KSTRIDE_32 4
// For kernel size 5, we have to do 2 query dims per thread, because we have fewer threads in each threadblock than the total
// number of queries.
// For kernel size 3, we have to read 2 query dims per thread
#define QITERS_5 2
#define QSTRIDE_5 16
#define QITERS_3 4
#define QSTRIDE_3 8
#define QITERS_3_HALF 2
#define QSTRIDE_3_HALF 8

// This is just for the other kernels that are not using SMEM
#define CUDA_NUM_THREADS_Q 512
#define CUDA_NUM_THREADS_K 512
#define CUDA_NUM_THREADS_RPB 64
#define CUDA_NUM_THREADS_Q16 512
#define CUDA_NUM_THREADS_K16 256
#define CUDA_NUM_THREADS_RPB16 64


template <int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t>
__global__ void natten2dqkrpb_cuda_forward_kernel_fp16(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> rpb,
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> attn,
    const int height,
    const int width,
    const int batch_size,
    const int heads,
    const int dilation_in,
    const int dimhalf) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (z < batch_size * heads){
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        if (x < height * width){
            const int y = blockIdx.y * blockDim.y + threadIdx.y;
            if (y < KERNEL_SIZE * KERNEL_SIZE){
                __half2* query2 = reinterpret_cast<__half2*>(query.data());
                __half2* key2 = reinterpret_cast<__half2*>(key.data());
                const int b = z / heads;
                const int h = z - b * heads;
                const int ki = y / KERNEL_SIZE;
                const int kj = y - ki * KERNEL_SIZE;
                const int i = x / width;
                const int j = x - i * width;
                const int ni = get_window_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
                const int nj = get_window_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
                const int pi = get_pb_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
                const int pj = get_pb_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
                __half2 updt = __float2half2_rn(0.f);
                const int stride2 = dimhalf * width;
                const int batchHeadOffset = b * (stride2*height*heads) + h * (stride2*height);
                const int queryOffset = batchHeadOffset + i * stride2 + j * dimhalf;
                const int keyOffset = batchHeadOffset + (ki*dilation+ni) * stride2 + (kj*dilation+nj) * dimhalf;
                #pragma unroll
                for (int dimOffset=0; dimOffset < dimhalf; ++dimOffset)
                    updt = __hfma2(query2[queryOffset+dimOffset], key2[keyOffset+dimOffset], updt);
                const int index = b * attn.stride(0) + h * attn.stride(1) + i * attn.stride(2) + j * attn.stride(3) + y * attn.stride(4);
                const int rpbIndex = h * rpb.stride(0) + (pi+ki) * rpb.stride(1) + (pj+kj) * rpb.stride(2);
                attn.data()[index] = static_cast<scalar_t>(__hadd(updt.x, updt.y)) + rpb.data()[rpbIndex];
            }
        }
    }
}


template <int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t>
__global__ void natten2dqkrpb_cuda_forward_kernel_fp32(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> rpb,
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> attn,
    const int height,
    const int width,
    const int batch_size,
    const int heads,
    const int dilation_in,
    const int dim) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (z < batch_size * heads){
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        if (x < height * width){
            const int y = blockIdx.y * blockDim.y + threadIdx.y;
            if (y < KERNEL_SIZE * KERNEL_SIZE){
                const int b = z / heads;
                const int h = z - b * heads;
                const int ki = y / KERNEL_SIZE;
                const int kj = y - ki * KERNEL_SIZE;
                const int i = x / width;
                const int j = x - i * width;
                const int ni = get_window_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
                const int nj = get_window_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
                const int pi = get_pb_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
                const int pj = get_pb_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
                scalar_t updt = scalar_t(0);
                const int batchHeadOffset = b * query.stride(0) + h * query.stride(1);
                const int queryOffset = batchHeadOffset + i * query.stride(2) + j * query.stride(3);
                const int keyOffset = batchHeadOffset + (ki*dilation+ni) * key.stride(2) + (kj*dilation+nj) * key.stride(3);
                #pragma unroll
                for (int dimOffset=0; dimOffset < dim; ++dimOffset)
                    updt += query.data()[queryOffset+dimOffset] * key.data()[keyOffset+dimOffset];
                const int index = b * attn.stride(0) + h * attn.stride(1) + i * attn.stride(2) + j * attn.stride(3) + y * attn.stride(4);
                const int rpbIndex = h * rpb.stride(0) + (pi+ki) * rpb.stride(1) + (pj+kj) * rpb.stride(2);
                updt += rpb.data()[rpbIndex];
                attn.data()[index] = updt;
            }
        }
    }
}


/* TODO: FIX BANK CONFLICTS */
template <int DILATION, typename scalar_t>
__global__ void natten2dqkrpb_cuda_forward_kernel_fp16_3x3_32(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> rpb,
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> attn,
    const int height,
    const int width,
    const int batch_size,
    const int heads,
    const int dilation_in) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    // Because batch heads have stride 1 per threadblock, we can just use blockIdx since blockDim will be 1 and threadIdx will
    // always be 0.
    // const int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int z = blockIdx.z;
    const int b = z / heads;
    const int h = z - b * heads;
    // Not needed again because it will always be true.
    // if (z < batch_size * heads)
    // {
    const int lti = threadIdx.y * (TILE_3*KERNEL_SIZE_3) + threadIdx.x;
    const int stride2 = DIMHALF_32 * width;
    const int batchHeadOffset = b * (stride2*height*heads) + h * (stride2*height);
    const int si = int(blockIdx.y / dilation) * (TILE_3 * dilation) + (blockIdx.y % dilation);
    const int sj = int(blockIdx.x / dilation) * (TILE_3 * dilation) + (blockIdx.x % dilation);
    const int sni = get_window_start(si, height, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
    const int snj = get_window_start(sj, width, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
    __shared__ __half2 tile[TILE_3*TILE_3][DIM_32+3];
    __shared__ __half2 kTile[KTILE_3*KTILE_3][DIM_32+3];
    __half2* query2 = reinterpret_cast<__half2*>(query.data());
    __half2* key2 = reinterpret_cast<__half2*>(key.data());

    /* query tile */
    const int qtx = lti / QSTRIDE_3_HALF;
    const int qty = (lti - qtx * QSTRIDE_3_HALF) * QITERS_3_HALF;
    if (qtx < TILE_3*TILE_3)
    {
        int qi = qtx / TILE_3;
        const int qj = (qtx - qi * TILE_3) * dilation + sj;
        qi = qi * dilation + si;
        if (qi < height && qj < width){
            #pragma unroll
            for (int ti=0; ti < QITERS_3_HALF; ++ti)
                tile[qtx][qty+ti] = query2[batchHeadOffset + qi * stride2 + qj * DIMHALF_32 + qty+ti];
        }
    }
    /* key tile */
    const int ktx = lti / KSTRIDE_32;
    const int kty = (lti - ktx * KSTRIDE_32) * KHALFITERS_32;
    if (ktx < KTILE_3*KTILE_3)
    {
        int bi = ktx / KTILE_3;
        const int bj = (ktx - bi * KTILE_3) * dilation + snj;
        bi = bi * dilation + sni;
        if (bi < height && bj < width){
            const int keyOffset = batchHeadOffset + bi * stride2 + bj * DIMHALF_32 + kty;
            #pragma unroll
            for (int ti=0; ti < KHALFITERS_32; ++ti)
                kTile[ktx][kty + ti] = key2[keyOffset + ti];
        }
    }
    __syncthreads();
    const int ii = threadIdx.y / KERNEL_SIZE_3;
    const int ki = threadIdx.y - ii * KERNEL_SIZE_3;
    const int jj = threadIdx.x / KERNEL_SIZE_3;
    const int kj = threadIdx.x - jj * KERNEL_SIZE_3;
    const int i = si + ii*dilation, j = sj + jj*dilation;
    if (i < height && j < width){
        const int ni = get_window_start(i, height, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
        const int nj = get_window_start(j, width, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
        const int pi = get_pb_start(i, height, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
        __half2 updt = __float2half2_rn(0.f);
        const int queryIdx = ii*TILE_3 + jj;
        const int keyIdx = int((ni+ki*dilation - sni)/dilation)*KTILE_3 + int((nj+kj*dilation - snj)/dilation);

        #pragma unroll
        for (int dimOffset=0; dimOffset < DIMHALF_32; ++dimOffset)
            updt = __hfma2(tile[queryIdx][dimOffset], kTile[keyIdx][dimOffset], updt);
        const int index = b * attn.stride(0) + h * attn.stride(1) + i * attn.stride(2) + j * attn.stride(3) + ki*KERNEL_SIZE_3+kj;
        const int rpbIndex = h * rpb.stride(0) + (pi+ki) * rpb.stride(1) + (pj+kj) * rpb.stride(2);
        attn.data()[index] = static_cast<scalar_t>(__hadd(updt.x, updt.y)) + rpb.data()[rpbIndex];
    }
    //}
}

/* TODO: CHECK BANK CONFLICTS */
template <int DILATION, typename scalar_t>
__global__ void natten2dqkrpb_cuda_forward_kernel_fp32_3x3_32(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> rpb,
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> attn,
    const int height,
    const int width,
    const int batch_size,
    const int heads,
    const int dilation_in) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    // Because batch heads have stride 1 per threadblock, we can just use blockIdx since blockDim will be 1 and threadIdx will
    // always be 0.
    // const int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int z = blockIdx.z;
    const int b = z / heads;
    const int h = z - b * heads;
    // Not needed again because it will always be true.
    // if (z < batch_size * heads)
    // {
    const int lti = threadIdx.y * (TILE_3*KERNEL_SIZE_3) + threadIdx.x;
    const int batchHeadOffset = b * query.stride(0) + h * query.stride(1);
    const int si = int(blockIdx.y / dilation) * (TILE_3 * dilation) + (blockIdx.y % dilation);
    const int sj = int(blockIdx.x / dilation) * (TILE_3 * dilation) + (blockIdx.x % dilation);
    const int sni = get_window_start(si, height, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
    const int snj = get_window_start(sj, width, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
    __shared__ scalar_t tile[TILE_3*TILE_3][DIM_32+3];
    __shared__ scalar_t kTile[KTILE_3*KTILE_3][DIM_32+3];

    /* query tile */
    const int qtx = lti / QSTRIDE_3;
    const int qty = (lti - qtx * QSTRIDE_3) * QITERS_3;
    if (qtx < TILE_3*TILE_3)
    {
        int qi = qtx / TILE_3;
        const int qj = (qtx - qi * TILE_3) * dilation + sj;
        qi = qi * dilation + si;
        if (qi < height && qj < width){
            #pragma unroll
            for (int ti=0; ti < QITERS_3; ++ti)
                tile[qtx][qty+ti] = query.data()[batchHeadOffset + qi * query.stride(2) + qj * query.stride(3) + qty+ti];
        }
    }
    /* key tile */
    const int ktx = lti / KSTRIDE_32;
    const int kty = (lti - ktx * KSTRIDE_32) * KITERS_32;
    if (ktx < KTILE_3*KTILE_3)
    {
        int bi = ktx / KTILE_3;
        const int bj = (ktx - bi * KTILE_3) * dilation + snj;
        bi = bi * dilation + sni;
        if (bi < height && bj < width){
            const int keyOffset = batchHeadOffset + bi * query.stride(2) + bj * query.stride(3) + kty;
            #pragma unroll
            for (int ti=0; ti < KITERS_32; ++ti)
                kTile[ktx][kty + ti] = key.data()[keyOffset + ti];
        }
    }
    __syncthreads();
    const int ii = threadIdx.y / KERNEL_SIZE_3;
    const int ki = threadIdx.y - ii * KERNEL_SIZE_3;
    const int jj = threadIdx.x / KERNEL_SIZE_3;
    const int kj = threadIdx.x - jj * KERNEL_SIZE_3;
    const int i = si + ii*dilation, j = sj + jj*dilation;
    if (i < height && j < width){
        const int ni = get_window_start(i, height, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
        const int nj = get_window_start(j, width, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
        const int pi = get_pb_start(i, height, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE_3, NEIGHBORHOOD_SIZE_3, dilation);
        scalar_t updt = scalar_t(0);
        const int queryIdx = ii*TILE_3 + jj;
        const int keyIdx = int((ni+ki*dilation - sni)/dilation)*KTILE_3 + int((nj+kj*dilation - snj)/dilation);

        #pragma unroll
        for (int dimOffset=0; dimOffset < DIM_32; ++dimOffset)
            updt += tile[queryIdx][dimOffset] * kTile[keyIdx][dimOffset];

        const int index = b * attn.stride(0) + h * attn.stride(1) + i * attn.stride(2) + j * attn.stride(3) + ki*KERNEL_SIZE_3+kj;
        const int rpbIndex = h * rpb.stride(0) + (pi+ki) * rpb.stride(1) + (pj+kj) * rpb.stride(2);
        updt += rpb.data()[rpbIndex];
        attn.data()[index] = updt;
    }
    //}
}


/* TODO: FIX BANK CONFLICTS */
template <int DILATION, typename scalar_t>
__global__ void natten2dqkrpb_cuda_forward_kernel_fp16_5x5_32(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> rpb,
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> attn,
    const int height,
    const int width,
    const int batch_size,
    const int heads,
    const int dilation_in) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    // Because batch heads have stride 1 per threadblock, we can just use blockIdx since blockDim will be 1 and threadIdx will
    // always be 0.
    // const int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int z = blockIdx.z;
    const int b = z / heads;
    const int h = z - b * heads;
    // Not needed again because it will always be true.
    // if (z < batch_size * heads)
    // {
    const int lti = threadIdx.y * (TILE_5*KERNEL_SIZE_5) + threadIdx.x;
    const int stride2 = DIMHALF_32 * width;
    const int batchHeadOffset = b * (stride2*height*heads) + h * (stride2*height);
    const int si = int(blockIdx.y / dilation) * (TILE_5 * dilation) + (blockIdx.y % dilation);
    const int sj = int(blockIdx.x / dilation) * (TILE_5 * dilation) + (blockIdx.x % dilation);
    const int sni = get_window_start(si, height, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
    const int snj = get_window_start(sj, width, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
    __shared__ __half2 tile[TILE_5*TILE_5][DIM_32+3];
    __shared__ __half2 kTile[KTILE_5*KTILE_5][DIM_32+3];
    __half2* query2 = reinterpret_cast<__half2*>(query.data());
    __half2* key2 = reinterpret_cast<__half2*>(key.data());

    /* query tile */
    const int qtx = lti / DIMHALF_32;
    const int qty = lti - qtx * DIMHALF_32;
    if (qtx < TILE_5*TILE_5)
    {
        int qi = qtx / TILE_5;
        const int qj = (qtx - qi * TILE_5) * dilation + sj;
        qi = qi * dilation + si;
        if (qi < height && qj < width){
            tile[qtx][qty] = query2[batchHeadOffset + qi * stride2 + qj * DIMHALF_32 + qty];
        }
    }
    /* key tile */
    const int ktx = lti / KSTRIDE_32;
    const int kty = (lti - ktx * KSTRIDE_32) * KHALFITERS_32;
    if (ktx < KTILE_5*KTILE_5)
    {
        int bi = ktx / KTILE_5;
        const int bj = (ktx - bi * KTILE_5) * dilation + snj;
        bi = bi * dilation + sni;
        if (bi < height && bj < width){
            const int keyOffset = batchHeadOffset + bi * stride2 + bj * DIMHALF_32 + kty;
            #pragma unroll
            for (int ti=0; ti < KHALFITERS_32; ++ti)
                kTile[ktx][kty + ti] = key2[keyOffset + ti];
        }
    }
    __syncthreads();
    const int ii = threadIdx.y / KERNEL_SIZE_5;
    const int ki = threadIdx.y - ii * KERNEL_SIZE_5;
    const int jj = threadIdx.x / KERNEL_SIZE_5;
    const int kj = threadIdx.x - jj * KERNEL_SIZE_5;
    const int i = si + ii*dilation, j = sj + jj*dilation;
    if (i < height && j < width){
        const int ni = get_window_start(i, height, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
        const int nj = get_window_start(j, width, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
        const int pi = get_pb_start(i, height, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
        __half2 updt = __float2half2_rn(0.f);
        const int queryIdx = ii*TILE_5 + jj;
        const int keyIdx = int((ni+ki*dilation - sni)/dilation)*KTILE_5 + int((nj+kj*dilation - snj)/dilation);

        #pragma unroll
        for (int dimOffset=0; dimOffset < DIMHALF_32; ++dimOffset)
            updt = __hfma2(tile[queryIdx][dimOffset], kTile[keyIdx][dimOffset], updt);
        const int index = b * attn.stride(0) + h * attn.stride(1) + i * attn.stride(2) + j * attn.stride(3) + ki*KERNEL_SIZE_5+kj;
        const int rpbIndex = h * rpb.stride(0) + (pi+ki) * rpb.stride(1) + (pj+kj) * rpb.stride(2);
        attn.data()[index] = static_cast<scalar_t>(__hadd(updt.x, updt.y)) + rpb.data()[rpbIndex];
    }
    //}
}

/* TODO: CHECK BANK CONFLICTS */
template <int DILATION, typename scalar_t>
__global__ void natten2dqkrpb_cuda_forward_kernel_fp32_5x5_32(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> rpb,
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> attn,
    const int height,
    const int width,
    const int batch_size,
    const int heads,
    const int dilation_in) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    // Because batch heads have stride 1 per threadblock, we can just use blockIdx since blockDim will be 1 and threadIdx will
    // always be 0.
    // const int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int z = blockIdx.z;
    const int b = z / heads;
    const int h = z - b * heads;
    // Not needed again because it will always be true.
    // if (z < batch_size * heads)
    // {
    const int lti = threadIdx.y * (TILE_5*KERNEL_SIZE_5) + threadIdx.x;
    const int batchHeadOffset = b * query.stride(0) + h * query.stride(1);
    const int si = int(blockIdx.y / dilation) * (TILE_5 * dilation) + (blockIdx.y % dilation);
    const int sj = int(blockIdx.x / dilation) * (TILE_5 * dilation) + (blockIdx.x % dilation);
    const int sni = get_window_start(si, height, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
    const int snj = get_window_start(sj, width, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
    __shared__ scalar_t tile[TILE_5*TILE_5][DIM_32+3];
    __shared__ scalar_t kTile[KTILE_5*KTILE_5][DIM_32+3];

    /* query tile */
    const int qtx = lti / QSTRIDE_5;
    const int qty = (lti - qtx * QSTRIDE_5) * QITERS_5;
    if (qtx < TILE_5*TILE_5)
    {
        int qi = qtx / TILE_5;
        const int qj = (qtx - qi * TILE_5) * dilation + sj;
        qi = qi * dilation + si;
        if (qi < height && qj < width){
            #pragma unroll
            for (int ti=0; ti < QITERS_5; ++ti)
                tile[qtx][qty+ti] = query.data()[batchHeadOffset + qi * query.stride(2) + qj * query.stride(3) + qty+ti];
        }
    }
    /* key tile */
    const int ktx = lti / KSTRIDE_32;
    const int kty = (lti - ktx * KSTRIDE_32) * KITERS_32;
    if (ktx < KTILE_5*KTILE_5)
    {
        int bi = ktx / KTILE_5;
        const int bj = (ktx - bi * KTILE_5) * dilation + snj;
        bi = bi * dilation + sni;
        if (bi < height && bj < width){
            const int keyOffset = batchHeadOffset + bi * query.stride(2) + bj * query.stride(3) + kty;
            #pragma unroll
            for (int ti=0; ti < KITERS_32; ++ti)
                kTile[ktx][kty + ti] = key.data()[keyOffset + ti];
        }
    }
    __syncthreads();
    const int ii = threadIdx.y / KERNEL_SIZE_5;
    const int ki = threadIdx.y - ii * KERNEL_SIZE_5;
    const int jj = threadIdx.x / KERNEL_SIZE_5;
    const int kj = threadIdx.x - jj * KERNEL_SIZE_5;
    const int i = si + ii*dilation, j = sj + jj*dilation;
    if (i < height && j < width){
        const int ni = get_window_start(i, height, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
        const int nj = get_window_start(j, width, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
        const int pi = get_pb_start(i, height, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE_5, NEIGHBORHOOD_SIZE_5, dilation);
        scalar_t updt = scalar_t(0);
        const int queryIdx = ii*TILE_5 + jj;
        const int keyIdx = int((ni+ki*dilation - sni)/dilation)*KTILE_5 + int((nj+kj*dilation - snj)/dilation);

        #pragma unroll
        for (int dimOffset=0; dimOffset < DIM_32; ++dimOffset)
            updt += tile[queryIdx][dimOffset] * kTile[keyIdx][dimOffset];

        const int index = b * attn.stride(0) + h * attn.stride(1) + i * attn.stride(2) + j * attn.stride(3) + ki*KERNEL_SIZE_5+kj;
        const int rpbIndex = h * rpb.stride(0) + (pi+ki) * rpb.stride(1) + (pj+kj) * rpb.stride(2);
        updt += rpb.data()[rpbIndex];
        attn.data()[index] = updt;
    }
    //}
}


template <int TILE, int KTILE, int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t>
__global__ void natten2dqkrpb_cuda_forward_kernel_fp16_7x7_9x9_32(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> rpb,
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> attn,
    const int height,
    const int width,
    const int batch_size,
    const int heads,
    const int dilation_in) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    // Because batch heads have stride 1 per threadblock, we can just use blockIdx since blockDim will be 1 and threadIdx will
    // always be 0.
    // const int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int z = blockIdx.z;
    const int b = z / heads;
    const int h = z - b * heads;
    // Not needed again because it will always be true.
    // if (z < batch_size * heads)
    // {
    const int lti = threadIdx.y * (TILE*KERNEL_SIZE) + threadIdx.x;
    const int stride2 = DIMHALF_32 * width;
    const int batchHeadOffset = b * (stride2*height*heads) + h * (stride2*height);
    const int si = int(blockIdx.y / dilation) * (TILE * dilation) + (blockIdx.y % dilation);
    const int sj = int(blockIdx.x / dilation) * (TILE * dilation) + (blockIdx.x % dilation);
    const int sni = get_window_start(si, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
    const int snj = get_window_start(sj, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
    __shared__ __half2 tile[TILE*TILE][DIM_32+3];
    __shared__ __half2 kTile[KTILE*KTILE][DIM_32+3];
    __half2* query2 = reinterpret_cast<__half2*>(query.data());
    __half2* key2 = reinterpret_cast<__half2*>(key.data());

    /* query tile */
    const int qtx = lti / DIM_32;
    const int qtyp = lti - qtx * DIM_32;
    const int qdi = qtyp / KHALFITERS_32;
    const int qdj = qtyp - qdi * KHALFITERS_32;
    const int qty = qdi*KITERS_32+qdj;
    if (qtx < TILE*TILE && qtyp < DIMHALF_32)
    {
        int qi = qtx / TILE;
        const int qj = (qtx - qi * TILE) * dilation + sj;
        qi = qi * dilation + si;
        if (qi < height && qj < width)
            tile[qtx][qty] = query2[batchHeadOffset + qi * stride2 + qj * DIMHALF_32 + qtyp];
    }
    /* key tile */
    const int ktx = lti / KSTRIDE_32;
    const int kty = (lti - ktx * KSTRIDE_32) * KHALFITERS_32;
    if (ktx < KTILE*KTILE)
    {
        int bi = ktx / KTILE;
        const int bj = (ktx - bi * KTILE) * dilation + snj;
        bi = bi * dilation + sni;
        if (bi < height && bj < width){
            const int keyOffset = batchHeadOffset + bi * stride2 + bj * DIMHALF_32 + kty;
            #pragma unroll
            for (int ti=0; ti < KHALFITERS_32; ++ti)
                kTile[ktx][kty*2 + ti] = key2[keyOffset + ti];
        }
    }
    __syncthreads();
    const int ii = threadIdx.y / KERNEL_SIZE;
    const int ki = threadIdx.y - ii * KERNEL_SIZE;
    const int jj = threadIdx.x / KERNEL_SIZE;
    const int kj = threadIdx.x - jj * KERNEL_SIZE;
    const int i = si + ii*dilation, j = sj + jj*dilation;
    if (i < height && j < width){
        const int ni = get_window_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int nj = get_window_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pi = get_pb_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        __half2 updt = __float2half2_rn(0.f);
        const int queryIdx = ii*TILE + jj;
        const int keyIdx = int((ni+ki*dilation - sni)/dilation)*KTILE + int((nj+kj*dilation - snj)/dilation);

        #pragma unroll
        for (int di=0; di < KSTRIDE_32; ++di)
            #pragma unroll
            for (int dj=0; dj <KHALFITERS_32; ++dj)
                updt = __hfma2(tile[queryIdx][di*KITERS_32+dj], kTile[keyIdx][di*KITERS_32+dj], updt);
        const int index = b * attn.stride(0) + h * attn.stride(1) + i * attn.stride(2) + j * attn.stride(3) + ki*KERNEL_SIZE+kj;
        const int rpbIndex = h * rpb.stride(0) + (pi+ki) * rpb.stride(1) + (pj+kj) * rpb.stride(2);
        attn.data()[index] = static_cast<scalar_t>(__hadd(updt.x, updt.y)) + rpb.data()[rpbIndex];
    }
    //}
}

template <int TILE, int KTILE, int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t>
__global__ void natten2dqkrpb_cuda_forward_kernel_fp32_7x7_9x9_32(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> rpb,
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> attn,
    const int height,
    const int width,
    const int batch_size,
    const int heads,
    const int dilation_in) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    // Because batch heads have stride 1 per threadblock, we can just use blockIdx since blockDim will be 1 and threadIdx will
    // always be 0.
    // const int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int z = blockIdx.z;
    const int b = z / heads;
    const int h = z - b * heads;
    // Not needed again because it will always be true.
    // if (z < batch_size * heads)
    // {
    const int lti = threadIdx.y * (TILE*KERNEL_SIZE) + threadIdx.x;
    const int batchHeadOffset = b * query.stride(0) + h * query.stride(1);
    const int si = int(blockIdx.y / dilation) * (TILE * dilation) + (blockIdx.y % dilation);
    const int sj = int(blockIdx.x / dilation) * (TILE * dilation) + (blockIdx.x % dilation);
    const int sni = get_window_start(si, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
    const int snj = get_window_start(sj, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
    __shared__ scalar_t tile[TILE*TILE][DIM_32+3];
    __shared__ scalar_t kTile[KTILE*KTILE][DIM_32+3];

    /* query tile */
    const int qtx = lti / DIM_32;
    const int qty = lti - qtx * DIM_32;
    if (qtx < TILE*TILE)
    {
        int qi = qtx / TILE;
        const int qj = (qtx - qi * TILE) * dilation + sj;
        qi = qi * dilation + si;
        if (qi < height && qj < width)
            tile[qtx][qty] = query.data()[batchHeadOffset + qi * query.stride(2) + qj * query.stride(3) + qty];
    }
    /* key tile */
    const int ktx = lti / KSTRIDE_32;
    const int kty = (lti - ktx * KSTRIDE_32) * KITERS_32;
    if (ktx < KTILE*KTILE)
    {
        int bi = ktx / KTILE;
        const int bj = (ktx - bi * KTILE) * dilation + snj;
        bi = bi * dilation + sni;
        if (bi < height && bj < width){
            const int keyOffset = batchHeadOffset + bi * query.stride(2) + bj * query.stride(3) + kty;
            #pragma unroll
            for (int ti=0; ti < KITERS_32; ++ti)
                kTile[ktx][kty + ti] = key.data()[keyOffset + ti];
        }
    }
    __syncthreads();
    const int ii = threadIdx.y / KERNEL_SIZE;
    const int ki = threadIdx.y - ii * KERNEL_SIZE;
    const int jj = threadIdx.x / KERNEL_SIZE;
    const int kj = threadIdx.x - jj * KERNEL_SIZE;
    const int i = si + ii*dilation, j = sj + jj*dilation;
    if (i < height && j < width){
        const int ni = get_window_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int nj = get_window_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pi = get_pb_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        scalar_t updt = scalar_t(0);
        const int queryIdx = ii*TILE + jj;
        const int keyIdx = int((ni+ki*dilation - sni)/dilation)*KTILE + int((nj+kj*dilation - snj)/dilation);

        #pragma unroll
        for (int dimOffset=0; dimOffset < DIM_32; ++dimOffset)
            updt += tile[queryIdx][dimOffset] * kTile[keyIdx][dimOffset];

        const int index = b * attn.stride(0) + h * attn.stride(1) + i * attn.stride(2) + j * attn.stride(3) + ki*KERNEL_SIZE+kj;
        const int rpbIndex = h * rpb.stride(0) + (pi+ki) * rpb.stride(1) + (pj+kj) * rpb.stride(2);
        updt += rpb.data()[rpbIndex];
        attn.data()[index] = updt;
    }
    //}
}


template <int TILEX, int TILEY, int KTILEX, int KTILEY, int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t, typename memscalar_t>
__global__ void natten2dqkrpb_cuda_forward_kernel_fp16_11x11_13x13_32(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> rpb,
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> attn,
    const int height,
    const int width,
    const int batch_size,
    const int heads,
    const int dilation_in) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    // Because batch heads have stride 1 per threadblock, we can just use blockIdx since blockDim will be 1 and threadIdx will
    // always be 0.
    // const int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int z = blockIdx.z;
    const int b = z / heads;
    const int h = z - b * heads;
    // Not needed again because it will always be true.
    // if (z < batch_size * heads)
    // {
    const int lti = threadIdx.y * (TILEY*KERNEL_SIZE) + threadIdx.x;
    const int stride2 = DIMHALF_32 * width;
    const int batchHeadOffset = b * (stride2*height*heads) + h * (stride2*height);
    const int si = int(blockIdx.y / dilation) * (TILEX * dilation) + (blockIdx.y % dilation);
    const int sj = int(blockIdx.x / dilation) * (TILEY * dilation) + (blockIdx.x % dilation);
    const int sni = get_window_start(si, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
    const int snj = get_window_start(sj, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
    __shared__ __half2 tile[TILEX*TILEY][DIM_32+3];
    __shared__ __half2 kTile[KTILEX*KTILEY][DIM_32+3];
    __half2* query2 = reinterpret_cast<__half2*>(query.data());
    __half2* key2 = reinterpret_cast<__half2*>(key.data());

    /* query tile */
    const int qtx = lti / DIM_32;
    const int qtyp = lti - qtx * DIM_32;
    const int qdi = qtyp / KHALFITERS_32;
    const int qdj = qtyp - qdi * KHALFITERS_32;
    const int qty = qdi*KITERS_32+qdj;
    if (qtx < TILEX*TILEY && qtyp < DIMHALF_32)
    {
        int qi = qtx / TILEY;
        const int qj = (qtx - qi * TILEY) * dilation + sj;
        qi =  qi * dilation + si;
        if (qi < height && qj < width)
            tile[qtx][qty] = query2[batchHeadOffset + qi * stride2 + qj * DIMHALF_32 + qtyp];
    }
    /* key tile */
    const int ktx = lti / KSTRIDE_32;
    const int kty = (lti - ktx * KSTRIDE_32) * KHALFITERS_32;
    if (ktx < KTILEX*KTILEY)
    {
        int bi = ktx / KTILEY;
        const int bj = (ktx - bi * KTILEY) * dilation + snj;
        bi = bi * dilation + sni;
        if (bi < height && bj < width){
            const int keyOffset = batchHeadOffset + bi * stride2 + bj * DIMHALF_32 + kty;
            #pragma unroll
            for (int ti=0; ti < KHALFITERS_32; ++ti)
                kTile[ktx][kty*2 + ti] = key2[keyOffset + ti];
        }
    }
    __syncthreads();
    const int ii = threadIdx.y / KERNEL_SIZE;
    const int ki = threadIdx.y - ii * KERNEL_SIZE;
    const int jj = threadIdx.x / KERNEL_SIZE;
    const int kj = threadIdx.x - jj * KERNEL_SIZE;
    const int i = si + ii*dilation, j = sj + jj*dilation;
    if (i < height && j < width){
        const int ni = get_window_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int nj = get_window_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pi = get_pb_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        __half2 updt = __float2half2_rn(0.f);
        const int queryIdx = ii*TILEY + jj;
        const int keyIdx = int((ni+ki*dilation - sni)/dilation)*KTILEY + int((nj+kj*dilation - snj)/dilation);

        #pragma unroll
        for (int di=0; di < KSTRIDE_32; ++di)
            #pragma unroll
            for (int dj=0; dj <KHALFITERS_32; ++dj)
                updt = __hfma2(tile[queryIdx][di*KITERS_32+dj], kTile[keyIdx][di*KITERS_32+dj], updt);
        const int index = b * attn.stride(0) + h * attn.stride(1) + i * attn.stride(2) + j * attn.stride(3) + ki*KERNEL_SIZE+kj;
        const int rpbIndex = h * rpb.stride(0) + (pi+ki) * rpb.stride(1) + (pj+kj) * rpb.stride(2);
        attn.data()[index] = static_cast<scalar_t>(__hadd(updt.x, updt.y)) + rpb.data()[rpbIndex];
    }
    //}
}

template <int TILEX, int TILEY, int KTILEX, int KTILEY, int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t, typename memscalar_t>
__global__ void natten2dqkrpb_cuda_forward_kernel_fp32_11x11_13x13_32(
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> rpb,
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> attn,
    const int height,
    const int width,
    const int batch_size,
    const int heads,
    const int dilation_in) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    // Because batch heads have stride 1 per threadblock, we can just use blockIdx since blockDim will be 1 and threadIdx will
    // always be 0.
    // const int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int z = blockIdx.z;
    const int b = z / heads;
    const int h = z - b * heads;
    // Not needed again because it will always be true.
    // if (z < batch_size * heads)
    // {
    const int lti = threadIdx.y * (TILEY*KERNEL_SIZE) + threadIdx.x;
    const int batchHeadOffset = b * query.stride(0) + h * query.stride(1);
    const int si = int(blockIdx.y / dilation) * (TILEX * dilation) + (blockIdx.y % dilation);
    const int sj = int(blockIdx.x / dilation) * (TILEY * dilation) + (blockIdx.x % dilation);
    const int sni = get_window_start(si, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
    const int snj = get_window_start(sj, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
    __shared__ memscalar_t tile[TILEX*TILEY][DIM_32+3];
    __shared__ memscalar_t kTile[KTILEX*KTILEY][DIM_32+3];

    /* query tile */
    const int qtx = lti / DIM_32;
    const int qty = lti - qtx * DIM_32;
    if (qtx < TILEX*TILEY)
    {
        int qi = qtx / TILEY;
        const int qj = (qtx - qi * TILEY) * dilation + sj;
        qi =  qi * dilation + si;
        if (qi < height && qj < width)
            tile[qtx][qty] = query.data()[batchHeadOffset + qi * query.stride(2) + qj * query.stride(3) + qty];
    }
    /* key tile */
    const int ktx = lti / KSTRIDE_32;
    const int kty = (lti - ktx * KSTRIDE_32) * KITERS_32;
    if (ktx < KTILEX*KTILEY)
    {
        int bi = ktx / KTILEY;
        const int bj = (ktx - bi * KTILEY) * dilation + snj;
        bi = bi * dilation + sni;
        if (bi < height && bj < width){
            const int keyOffset = batchHeadOffset + bi * query.stride(2) + bj * query.stride(3) + kty;
            #pragma unroll
            for (int ti=0; ti < KITERS_32; ++ti)
                kTile[ktx][kty + ti] = key.data()[keyOffset + ti];
        }
    }
    __syncthreads();
    const int ii = threadIdx.y / KERNEL_SIZE;
    const int ki = threadIdx.y - ii * KERNEL_SIZE;
    const int jj = threadIdx.x / KERNEL_SIZE;
    const int kj = threadIdx.x - jj * KERNEL_SIZE;
    const int i = si + ii*dilation, j = sj + jj*dilation;
    if (i < height && j < width){
        const int ni = get_window_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int nj = get_window_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pi = get_pb_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        scalar_t updt = scalar_t(0);
        const int queryIdx = ii*TILEY + jj;
        const int keyIdx = int((ni+ki*dilation - sni)/dilation)*KTILEY + int((nj+kj*dilation - snj)/dilation);

        #pragma unroll
        for (int dimOffset=0; dimOffset < DIM_32; ++dimOffset)
            updt += tile[queryIdx][dimOffset] * kTile[keyIdx][dimOffset];

        const int index = b * attn.stride(0) + h * attn.stride(1) + i * attn.stride(2) + j * attn.stride(3) + ki*KERNEL_SIZE+kj;
        const int rpbIndex = h * rpb.stride(0) + (pi+ki) * rpb.stride(1) + (pj+kj) * rpb.stride(2);
        updt += rpb.data()[rpbIndex];
        attn.data()[index] = updt;
    }
    //}
}

template <int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t>
__global__ void natten2dq_cuda_backward_kernel_fp32(
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> d_query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> d_attn,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const int height,
    const int width,
    const int heads,
    const int dilation_in,
    const int dim,
    const int totalElements) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < totalElements){
        int indtmp1 = linearIndex/dim;
        const int d = linearIndex - indtmp1 * dim;
        int indtmp2 = indtmp1/width;
        const int j = indtmp1 - indtmp2 * width;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/height;
        const int i = indtmp1 - indtmp2 * height;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/heads;
        const int h = indtmp1 - indtmp2 * heads;
        const int b = indtmp2;
        const int ni = get_window_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int nj = get_window_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        scalar_t d_query_update = scalar_t(0);
        int attnOffset = b * d_attn.stride(0) + h * d_attn.stride(1) + i * d_attn.stride(2) + j * d_attn.stride(3);
        const int keyOffset = b * key.stride(0) + h * key.stride(1) + d;
        #pragma unroll
        for (int xi=ni; xi < ni + KERNEL_SIZE * dilation; xi+=dilation)
            #pragma unroll
            for (int xj=nj; xj < nj + KERNEL_SIZE * dilation; xj+=dilation){
                const int keyIndex = keyOffset + xi * key.stride(2) + xj * key.stride(3);
                d_query_update += d_attn.data()[attnOffset] * key.data()[keyIndex];
                ++attnOffset;
            }
        d_query.data()[linearIndex] = d_query_update;
    }
}

template <int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t>
__global__ void natten2dq_cuda_backward_kernel_fp16(
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> d_query,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> d_attn,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> key,
    const int height,
    const int width,
    const int heads,
    const int dilation_in,
    const int dimhalf,
    const int totalElements) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < totalElements){
        __half2* d_query2 = reinterpret_cast<__half2*>(d_query.data());
        __half2* key2 = reinterpret_cast<__half2*>(key.data());
        int indtmp1 = linearIndex/dimhalf;
        const int d = linearIndex - indtmp1 * dimhalf;
        int indtmp2 = indtmp1/width;
        const int j = indtmp1 - indtmp2 * width;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/height;
        const int i = indtmp1 - indtmp2 * height;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/heads;
        const int h = indtmp1 - indtmp2 * heads;
        const int b = indtmp2;
        const int ni = get_window_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int nj = get_window_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        __half2 d_query_update = __float2half2_rn(0.f);
        int attnOffset = b * d_attn.stride(0) + h * d_attn.stride(1) + i * d_attn.stride(2) + j * d_attn.stride(3);
        const int stride2 = dimhalf * width;
        const int keyOffset = b * (stride2 * height * heads) + h * (stride2 * height) + d;
        #pragma unroll
        for (int xi=ni; xi < ni + KERNEL_SIZE * dilation; xi+=dilation)
            #pragma unroll
            for (int xj=nj; xj < nj + KERNEL_SIZE * dilation; xj+=dilation){
                const int keyIndex = keyOffset + xi * stride2 + xj * dimhalf;
                scalar_t a = d_attn.data()[attnOffset];
                d_query_update = __hfma2(__halves2half2(a, a), key2[keyIndex], d_query_update);
                ++attnOffset;
            }
        d_query2[linearIndex] = d_query_update;
    }
}

template <int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t>
__global__ void natten2drpb_cuda_backward_kernel_fp16(
    torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> d_rpb,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> d_attn,
    const int height,
    const int width,
    const int dilation_in,
    const int batch_size,
    const int d_rpb_numel,
    const int totalThreads) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < totalThreads){
        int indtmp1 = linearIndex/KERNEL_SIZE;
        const int kj = linearIndex - indtmp1 * KERNEL_SIZE;
        int indtmp2 = indtmp1/KERNEL_SIZE;
        const int ki = indtmp1 - indtmp2 * KERNEL_SIZE;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/width;
        const int j = indtmp1 - indtmp2 * width;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/height;
        const int i = indtmp1 - indtmp2 * height;
        const int h = indtmp2;
        const int pi = get_pb_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        float d_rpb_update = scalar_t(0);
        int attnOffset = h * d_attn.stride(1) + i * d_attn.stride(2) + j * d_attn.stride(3) + (ki*KERNEL_SIZE+kj);
        #pragma unroll
        for (int b=0; b < batch_size; ++b){
            d_rpb_update += static_cast<float>(d_attn.data()[attnOffset]);
            attnOffset += d_attn.stride(0);
        }
        const int index = h * d_rpb.stride(0) + (pi+ki) * d_rpb.stride(1) + (pj+kj) * d_rpb.stride(2);
        at::native::fastAtomicAdd(d_rpb.data(), index, d_rpb_numel, static_cast<scalar_t>(d_rpb_update), true);
    }
}

template <int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t>
__global__ void natten2drpb_cuda_backward_kernel(
    torch::PackedTensorAccessor32<scalar_t,3,torch::DefaultPtrTraits> d_rpb,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> d_attn,
    const int height,
    const int width,
    const int dilation_in,
    const int batch_size,
    const int d_rpb_numel,
    const int totalThreads) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < totalThreads){
        int indtmp1 = linearIndex/KERNEL_SIZE;
        const int kj = linearIndex - indtmp1 * KERNEL_SIZE;
        int indtmp2 = indtmp1/KERNEL_SIZE;
        const int ki = indtmp1 - indtmp2 * KERNEL_SIZE;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/width;
        const int j = indtmp1 - indtmp2 * width;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/height;
        const int i = indtmp1 - indtmp2 * height;
        const int h = indtmp2;
        const int pi = get_pb_start(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int pj = get_pb_start(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        scalar_t d_rpb_update = scalar_t(0);
        int attnOffset = h * d_attn.stride(1) + i * d_attn.stride(2) + j * d_attn.stride(3) + (ki*KERNEL_SIZE+kj);
        #pragma unroll
        for (int b=0; b < batch_size; ++b){
            d_rpb_update += d_attn.data()[attnOffset];
            attnOffset += d_attn.stride(0);
        }
        const int index = h * d_rpb.stride(0) + (pi+ki) * d_rpb.stride(1) + (pj+kj) * d_rpb.stride(2);
        at::native::fastAtomicAdd(d_rpb.data(), index, d_rpb_numel, static_cast<scalar_t>(d_rpb_update), true);
    }
}

template <int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t>
__global__ void natten2dk_cuda_backward_kernel_fp16(
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> d_key,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> d_attn,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const int height,
    const int width,
    const int heads,
    const int dilation_in,
    const int dimhalf,
    const int d_key_numel) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < d_key_numel){
        __half2* d_key2 = reinterpret_cast<__half2*>(d_key.data());
        __half2* query2 = reinterpret_cast<__half2*>(query.data());
        int indtmp1 = linearIndex/dimhalf;
        const int d = linearIndex - indtmp1 * dimhalf;
        int indtmp2 = indtmp1/width;
        const int j = indtmp1 - indtmp2 * width;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/height;
        const int i = indtmp1 - indtmp2 * height;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/heads;
        const int h = indtmp1 - indtmp2 * heads;
        const int b = indtmp2;
        const int ni = get_backward_window_start(i, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int nj = get_backward_window_start(j, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int ei = get_backward_window_end(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int ej = get_backward_window_end(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int attnOffset = b * d_attn.stride(0) + h * d_attn.stride(1);
        const int stride2 = dimhalf * width;
        const int queryOffset = b * (stride2 * height * heads) + h * (stride2 * height) + d;
        __half2 d_key_update = __float2half2_rn(0.f);
        #pragma unroll
        for (int xi=ni; xi < ei; xi+=dilation){
            const int oni = get_window_start(xi, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
            #pragma unroll
            for (int xj=nj; xj < ej; xj+=dilation){
                const int onj = get_window_start(xj, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
                const int queryIndex = queryOffset + xi * stride2 + xj * dimhalf;
                const int attnIndex = attnOffset + xi * d_attn.stride(2) + xj * d_attn.stride(3) + int((i-oni)/dilation)*KERNEL_SIZE+int((j-onj)/dilation);
                scalar_t a = d_attn.data()[attnIndex];
                d_key_update = __hfma2(query2[queryIndex], __halves2half2(a, a), d_key_update);
            }
        }
        d_key2[linearIndex] = d_key_update;
    }
}

template <int KERNEL_SIZE, int NEIGHBORHOOD_SIZE, int DILATION, typename scalar_t>
__global__ void natten2dk_cuda_backward_kernel_fp32(
    torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> d_key,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> d_attn,
    const torch::PackedTensorAccessor32<scalar_t,5,torch::DefaultPtrTraits> query,
    const int height,
    const int width,
    const int heads,
    const int dilation_in,
    const int dim,
    const int d_key_numel) {
    const int dilation = (DILATION>0) ? DILATION : dilation_in;
    const int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (linearIndex < d_key_numel){
        int indtmp1 = linearIndex/dim;
        const int d = linearIndex - indtmp1 * dim;
        int indtmp2 = indtmp1/width;
        const int j = indtmp1 - indtmp2 * width;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/height;
        const int i = indtmp1 - indtmp2 * height;
        indtmp1 = indtmp2;
        indtmp2 = indtmp1/heads;
        const int h = indtmp1 - indtmp2 * heads;
        const int b = indtmp2;
        const int ni = get_backward_window_start(i, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int nj = get_backward_window_start(j, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int ei = get_backward_window_end(i, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int ej = get_backward_window_end(j, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
        const int attnOffset = b * d_attn.stride(0) + h * d_attn.stride(1);
        const int queryOffset = b * query.stride(0) + h * query.stride(1) + d;
        scalar_t d_key_update = scalar_t(0);
        #pragma unroll
        for (int xi=ni; xi < ei; xi+=dilation){
            const int oni = get_window_start(xi, height, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
            #pragma unroll
            for (int xj=nj; xj < ej; xj+=dilation){
                const int onj = get_window_start(xj, width, KERNEL_SIZE, NEIGHBORHOOD_SIZE, dilation);
                const int queryIndex = queryOffset + xi * query.stride(2) + xj * query.stride(3);
                const int attnIndex = attnOffset + xi * d_attn.stride(2) + xj * d_attn.stride(3) + int((i-oni)/dilation)*KERNEL_SIZE+int((j-onj)/dilation);
                d_key_update += query.data()[queryIndex] * d_attn.data()[attnIndex];
            }
        }
        d_key.data()[linearIndex] = d_key_update;
    }
}

torch::Tensor natten2dqkrpb_cuda_forward(
    const torch::Tensor &query,
    const torch::Tensor &key,
    const torch::Tensor &rpb,
    const int dilation) {
    int64_t batch_size = query.size(0);
    int64_t heads = query.size(1);
    int64_t height = query.size(2);
    int64_t width = query.size(3);
    int64_t dim = query.size(4);
    int64_t RPB_MAX = rpb.size(1);
    int kernel_size = (RPB_MAX + 1) / 2;
    int kernel_size_sq = pow(kernel_size, 2);
    int zsize = batch_size * heads;
    int xsize = height * width;
    CHECK_FEATMAP(height, width, kernel_size, dilation);
    CHECK_KERNELSIZE("natten2dqkrpb_cuda_forward", kernel_size);
    int KERNELTHREADS = min(CUDA_NUM_THREADS, kernel_size_sq);
    int PIXELTHREADS = min(int(CUDA_NUM_THREADS / KERNELTHREADS), xsize);
    int BATCHTHREADS = max(1, CUDA_NUM_THREADS / (PIXELTHREADS * KERNELTHREADS));

    auto attn = torch::zeros(
            {batch_size, heads, height, width, kernel_size_sq}, query.options());

    const auto stream = c10::cuda::getCurrentCUDAStream();
    const dim3 blocks(
            (xsize + PIXELTHREADS - 1) / PIXELTHREADS,
            (kernel_size_sq + KERNELTHREADS - 1) / KERNELTHREADS,
            (zsize + BATCHTHREADS - 1) / BATCHTHREADS);
    const dim3 threads(PIXELTHREADS, KERNELTHREADS, BATCHTHREADS);
    AT_DISPATCH_FLOATING_TYPES(query.scalar_type(), "natten2dqkrpb_cuda_forward", ([&] {
        const auto query_a = query.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto key_a = key.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto rpb_a = rpb.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
        auto attn_a = attn.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        LAUNCH_DNA_KNS(kernel_size, dilation, natten2dqkrpb_cuda_forward_kernel_fp32, 
                blocks, threads, 0, stream, 
                query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation, dim);
    }));
    return attn;
}

torch::Tensor natten2dqkrpb_cuda_forward_fp16(
    const torch::Tensor &query,
    const torch::Tensor &key,
    const torch::Tensor &rpb,
    const int dilation) {
    int64_t batch_size = query.size(0);
    int64_t heads = query.size(1);
    int64_t height = query.size(2);
    int64_t width = query.size(3);
    int64_t dimhalf = query.size(4) / 2;
    int64_t RPB_MAX = rpb.size(1);
    int kernel_size = (RPB_MAX + 1) / 2;
    int kernel_size_sq = pow(kernel_size, 2);
    int zsize = batch_size * heads;
    int xsize = height * width;
    CHECK_FEATMAP(height, width, kernel_size, dilation);
    CHECK_KERNELSIZE("natten2dqkrpb_cuda_forward_fp16", kernel_size);
    TORCH_CHECK(dimhalf*2 == query.size(4), "Dims per head must be an even number in FP16.");
    int KERNELTHREADS = min(CUDA_NUM_THREADS, kernel_size_sq);
    int PIXELTHREADS = min(int(CUDA_NUM_THREADS / KERNELTHREADS), xsize);
    int BATCHTHREADS = max(1, CUDA_NUM_THREADS / (PIXELTHREADS * KERNELTHREADS));

    auto attn = torch::zeros(
            {batch_size, heads, height, width, kernel_size_sq}, query.options());

    const auto stream = c10::cuda::getCurrentCUDAStream();
    const dim3 blocks(
            (xsize + PIXELTHREADS - 1) / PIXELTHREADS,
            (kernel_size_sq + KERNELTHREADS - 1) / KERNELTHREADS,
            (zsize + BATCHTHREADS - 1) / BATCHTHREADS);
    const dim3 threads(PIXELTHREADS, KERNELTHREADS, BATCHTHREADS);
    AT_DISPATCH_HALF_TYPES(at::kHalf, query.scalar_type(), "natten2dqkrpb_cuda_forward_fp16", ([&] {
        const auto query_a = query.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto key_a = key.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto rpb_a = rpb.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
        auto attn_a = attn.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        LAUNCH_DNA_KNS(kernel_size, dilation, natten2dqkrpb_cuda_forward_kernel_fp16, 
                blocks, threads, 0, stream, 
                query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation, dimhalf);
    }));
    return attn;
}

torch::Tensor natten2dqkrpb_cuda_forward_tiled_32(
    const torch::Tensor &query,
    const torch::Tensor &key,
    const torch::Tensor &rpb,
    const int dilation) {
    int64_t batch_size = query.size(0);
    int64_t heads = query.size(1);
    int64_t height = query.size(2);
    int64_t width = query.size(3);
    int64_t dim = query.size(4);
    int64_t RPB_MAX = rpb.size(1);
    int kernel_size = (RPB_MAX + 1) / 2;
    CHECK_FEATMAP(height, width, kernel_size, dilation);
    TORCH_CHECK(dim == DIM_32, "natten2dqkrpb_cuda_forward_fp32_tiled_32", " only supports 32-dim attention heads.");
    TORCH_CHECK(kernel_size == KERNEL_SIZE_7 || kernel_size == KERNEL_SIZE_3 || kernel_size == KERNEL_SIZE_5 ||
            kernel_size == KERNEL_SIZE_9 || kernel_size == KERNEL_SIZE_11 || kernel_size == KERNEL_SIZE_13,
            "natten2dqkrpb_cuda_forward_fp32_tiled_32", " only supports kernel sizes 3, 5, 7, 9, 11, and 13.");
    int xsize = width * kernel_size;
    int ysize = height * kernel_size;
    int zsize = batch_size * heads;

    auto attn = torch::zeros({batch_size, heads, height, width, kernel_size*kernel_size}, query.options());

    const auto stream = c10::cuda::getCurrentCUDAStream();
    int XTHREADS = -1;
    int YTHREADS = -1;
    int BATCHTHREADS = -1;
    if (kernel_size == KERNEL_SIZE_7)
    {
        XTHREADS = XYTHREADS_7;
        YTHREADS = XYTHREADS_7;
        BATCHTHREADS = BATCHTHREADS_7;
    }
    else if (kernel_size == KERNEL_SIZE_3)
    {
        XTHREADS = XYTHREADS_3;
        YTHREADS = XYTHREADS_3;
        BATCHTHREADS = BATCHTHREADS_3;
    }
    else if (kernel_size == KERNEL_SIZE_5)
    {
        XTHREADS = XYTHREADS_5;
        YTHREADS = XYTHREADS_5;
        BATCHTHREADS = BATCHTHREADS_5;
    }
    else if (kernel_size == KERNEL_SIZE_9)
    {
        XTHREADS = XYTHREADS_9;
        YTHREADS = XYTHREADS_9;
        BATCHTHREADS = BATCHTHREADS_9;
    }
    else if (kernel_size == KERNEL_SIZE_11)
    {
        XTHREADS = XTHREADS_11;
        YTHREADS = YTHREADS_11;
        BATCHTHREADS = BATCHTHREADS_11;
    }
    else if (kernel_size == KERNEL_SIZE_13)
    {
        XTHREADS = XTHREADS_13;
        YTHREADS = YTHREADS_13;
        BATCHTHREADS = BATCHTHREADS_13;
    }
    const dim3 blocks(
            (xsize + XTHREADS*dilation - 1) / XTHREADS,
            (ysize + YTHREADS*dilation - 1) / YTHREADS,
            (zsize + BATCHTHREADS - 1) / BATCHTHREADS);
    const dim3 threads(XTHREADS, YTHREADS, BATCHTHREADS);
    AT_DISPATCH_FLOATING_TYPES(query.scalar_type(), "natten2dqkrpb_cuda_forward_fp32_tiled_32", ([&] {
        const auto query_a = query.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto key_a = key.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto rpb_a = rpb.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
        auto attn_a = attn.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        if (kernel_size == KERNEL_SIZE_7)
            LAUNCH_DNA_KNS_TILED79(TILE_7, KTILE_7, KERNEL_SIZE_7, NEIGHBORHOOD_SIZE_7, dilation,
                    natten2dqkrpb_cuda_forward_kernel_fp32_7x7_9x9_32, blocks, threads, 0, stream, 
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
        else if (kernel_size == KERNEL_SIZE_9)
            LAUNCH_DNA_KNS_TILED79(TILE_9, KTILE_9, KERNEL_SIZE_9, NEIGHBORHOOD_SIZE_9, dilation,
                    natten2dqkrpb_cuda_forward_kernel_fp32_7x7_9x9_32, blocks, threads, 0, stream, 
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
        else if (kernel_size == KERNEL_SIZE_3)
            LAUNCH_DNA_DS(dilation, natten2dqkrpb_cuda_forward_kernel_fp32_3x3_32,
                    blocks, threads, 0, stream,
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
        else if (kernel_size == KERNEL_SIZE_5)
            LAUNCH_DNA_DS(dilation, natten2dqkrpb_cuda_forward_kernel_fp32_5x5_32,
                    blocks, threads, 0, stream,
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
        else if (kernel_size == KERNEL_SIZE_11)
            LAUNCH_DNA_KNS_TILED1113(TILE_11_X, TILE_11_Y, KTILE_11_X, KTILE_11_Y, 
                    KERNEL_SIZE_11, NEIGHBORHOOD_SIZE_11, dilation, scalar_t,
                    natten2dqkrpb_cuda_forward_kernel_fp32_11x11_13x13_32,
                    blocks, threads, 0, stream,
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
        else if (kernel_size == KERNEL_SIZE_13)
            LAUNCH_DNA_KNS_TILED1113(TILE_13_X, TILE_13_Y, KTILE_13_X, KTILE_13_Y, 
                    KERNEL_SIZE_13, NEIGHBORHOOD_SIZE_13, dilation, float,
                    natten2dqkrpb_cuda_forward_kernel_fp32_11x11_13x13_32,
                    blocks, threads, 0, stream,
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
    }));
    return attn;
}

torch::Tensor natten2dqkrpb_cuda_forward_fp16_tiled_32(
    const torch::Tensor &query,
    const torch::Tensor &key,
    const torch::Tensor &rpb,
    const int dilation) {
    int64_t batch_size = query.size(0);
    int64_t heads = query.size(1);
    int64_t height = query.size(2);
    int64_t width = query.size(3);
    int64_t dimhalf = query.size(4) / 2;
    int64_t RPB_MAX = rpb.size(1);
    int kernel_size = (RPB_MAX + 1) / 2;
    CHECK_FEATMAP(height, width, kernel_size, dilation);
    TORCH_CHECK(dimhalf*2 == query.size(4), "Dims per head must be an even number in FP16.");
    TORCH_CHECK(dimhalf*2 == DIM_32, "natten2dqkrpb_cuda_forward_fp16_tiled_32", " only supports 32-dim attention heads.");
    TORCH_CHECK(kernel_size == KERNEL_SIZE_7 || kernel_size == KERNEL_SIZE_3 || kernel_size == KERNEL_SIZE_5 ||
            kernel_size == KERNEL_SIZE_9 || kernel_size == KERNEL_SIZE_11 || kernel_size == KERNEL_SIZE_13,
            "natten2dqkrpb_cuda_forward_fp16_tiled_32", " only supports kernel sizes 3, 5, 7, 9, 11, and 13.");
    int xsize = width * kernel_size;
    int ysize = height * kernel_size;
    int zsize = batch_size * heads;

    auto attn = torch::zeros({batch_size, heads, height, width, kernel_size*kernel_size}, query.options());

    const auto stream = c10::cuda::getCurrentCUDAStream();
    int XTHREADS = -1;
    int YTHREADS = -1;
    int BATCHTHREADS = -1;
    if (kernel_size == KERNEL_SIZE_7)
    {
        XTHREADS = XYTHREADS_7;
        YTHREADS = XYTHREADS_7;
        BATCHTHREADS = BATCHTHREADS_7;
    }
    else if (kernel_size == KERNEL_SIZE_3)
    {
        XTHREADS = XYTHREADS_3;
        YTHREADS = XYTHREADS_3;
        BATCHTHREADS = BATCHTHREADS_3;
    }
    else if (kernel_size == KERNEL_SIZE_5)
    {
        XTHREADS = XYTHREADS_5;
        YTHREADS = XYTHREADS_5;
        BATCHTHREADS = BATCHTHREADS_5;
    }
    else if (kernel_size == KERNEL_SIZE_9)
    {
        XTHREADS = XYTHREADS_9;
        YTHREADS = XYTHREADS_9;
        BATCHTHREADS = BATCHTHREADS_9;
    }
    else if (kernel_size == KERNEL_SIZE_11)
    {
        XTHREADS = XTHREADS_11;
        YTHREADS = YTHREADS_11;
        BATCHTHREADS = BATCHTHREADS_11;
    }
    else if (kernel_size == KERNEL_SIZE_13)
    {
        XTHREADS = XTHREADS_13;
        YTHREADS = YTHREADS_13;
        BATCHTHREADS = BATCHTHREADS_13;
    }
    const dim3 blocks(
            (xsize + XTHREADS*dilation - 1) / XTHREADS,
            (ysize + YTHREADS*dilation - 1) / YTHREADS,
            (zsize + BATCHTHREADS - 1) / BATCHTHREADS);
    const dim3 threads(XTHREADS, YTHREADS, BATCHTHREADS);
    AT_DISPATCH_HALF_TYPES(at::kHalf, query.scalar_type(), "natten2dqkrpb_cuda_forward_fp16_tiled_32", ([&] {
        const auto query_a = query.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto key_a = key.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto rpb_a = rpb.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
        auto attn_a = attn.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        if (kernel_size == KERNEL_SIZE_7)
            LAUNCH_DNA_KNS_TILED79(TILE_7, KTILE_7, KERNEL_SIZE_7, NEIGHBORHOOD_SIZE_7, dilation,
                    natten2dqkrpb_cuda_forward_kernel_fp16_7x7_9x9_32, blocks, threads, 0, stream, 
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
        else if (kernel_size == KERNEL_SIZE_9)
            LAUNCH_DNA_KNS_TILED79(TILE_9, KTILE_9, KERNEL_SIZE_9, NEIGHBORHOOD_SIZE_9, dilation,
                    natten2dqkrpb_cuda_forward_kernel_fp16_7x7_9x9_32, blocks, threads, 0, stream, 
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
        else if (kernel_size == KERNEL_SIZE_3)
            LAUNCH_DNA_DS(dilation, natten2dqkrpb_cuda_forward_kernel_fp16_3x3_32,
                    blocks, threads, 0, stream,
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
        else if (kernel_size == KERNEL_SIZE_5)
            LAUNCH_DNA_DS(dilation, natten2dqkrpb_cuda_forward_kernel_fp16_5x5_32,
                    blocks, threads, 0, stream,
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
        else if (kernel_size == KERNEL_SIZE_11)
            LAUNCH_DNA_KNS_TILED1113(TILE_11_X, TILE_11_Y, KTILE_11_X, KTILE_11_Y, 
                    KERNEL_SIZE_11, NEIGHBORHOOD_SIZE_11, dilation, scalar_t,
                    natten2dqkrpb_cuda_forward_kernel_fp16_11x11_13x13_32,
                    blocks, threads, 0, stream,
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
        else if (kernel_size == KERNEL_SIZE_13)
            LAUNCH_DNA_KNS_TILED1113(TILE_13_X, TILE_13_Y, KTILE_13_X, KTILE_13_Y, 
                    KERNEL_SIZE_13, NEIGHBORHOOD_SIZE_13, dilation, scalar_t,
                    natten2dqkrpb_cuda_forward_kernel_fp16_11x11_13x13_32,
                    blocks, threads, 0, stream,
                    query_a, key_a, rpb_a, attn_a, height, width, batch_size, heads, dilation);
    }));
    return attn;
}

std::vector<torch::Tensor> natten2dqkrpb_cuda_backward(
    const torch::Tensor &d_attn,
    const torch::Tensor &query,
    const torch::Tensor &key,
    const bool biasEnabled,
    const int dilation) {
    int64_t batch_size = query.size(0);
    int64_t heads = query.size(1);
    int64_t height = query.size(2);
    int64_t width = query.size(3);
    int64_t dim = query.size(4);
    int kernel_size_sq = d_attn.size(4);
    int kernel_size = sqrt(kernel_size_sq);
    CHECK_FEATMAP(height, width, kernel_size, dilation);
    CHECK_KERNELSIZE("natten2dqkrpb_cuda_backward", kernel_size);
    int64_t RPB_MAX = kernel_size * 2 - 1;
   
    auto d_query = torch::zeros_like(query);
    auto d_key = torch::zeros_like(key);
    at::Tensor d_rpb;
    if (biasEnabled)
        d_rpb = torch::zeros({heads, RPB_MAX, RPB_MAX}, d_attn.options());

    int32_t n_rpb = heads * height * width * kernel_size_sq;
    int blocks_rpb = GET_BLOCKS(n_rpb, CUDA_NUM_THREADS_RPB);
    dim3 grid_rpb(blocks_rpb);
    dim3 blockr(CUDA_NUM_THREADS_RPB);
    int32_t n_query = d_query.numel();
    int blocks_query = GET_BLOCKS(n_query, CUDA_NUM_THREADS_Q);
    dim3 grid_query(blocks_query);
    dim3 blockq(CUDA_NUM_THREADS_Q);
    int32_t n_key = d_key.numel();
    int blocks_key = GET_BLOCKS(n_key, CUDA_NUM_THREADS_K);
    dim3 grid_key(blocks_key);
    dim3 blockk(CUDA_NUM_THREADS_K);
    const auto stream = c10::cuda::getCurrentCUDAStream();
    AT_DISPATCH_FLOATING_TYPES(d_query.scalar_type(), "natten2dqkrpb_backward_cuda", ([&] {
        const auto d_attn_a = d_attn.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto query_a = query.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto key_a = key.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        auto d_query_a = d_query.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        auto d_key_a = d_key.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        if (biasEnabled) {
            auto d_rpb_a = d_rpb.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
            LAUNCH_DNA_KNS(kernel_size, dilation, natten2drpb_cuda_backward_kernel, grid_rpb, blockr, 0, stream,
                    d_rpb_a, d_attn_a, height, width, dilation, batch_size, d_rpb.numel(), n_rpb);
        }
        LAUNCH_DNA_KNS(kernel_size, dilation, natten2dq_cuda_backward_kernel_fp32, grid_query, blockq, 0, stream,
                d_query_a, d_attn_a, key_a, height, width, heads, dilation, dim, n_query);
        LAUNCH_DNA_KNS(kernel_size, dilation, natten2dk_cuda_backward_kernel_fp32, grid_key, blockk, 0, stream,
                d_key_a, d_attn_a, query_a, height, width, heads, dilation, dim, n_key);
    }));
    return {d_query, d_key, d_rpb};
}

std::vector<torch::Tensor> natten2dqkrpb_cuda_backward_fp16(
    const torch::Tensor &d_attn,
    const torch::Tensor &query,
    const torch::Tensor &key,
    const bool biasEnabled,
    const int dilation) {
    int64_t batch_size = query.size(0);
    int64_t heads = query.size(1);
    int64_t height = query.size(2);
    int64_t width = query.size(3);
    int64_t dimhalf = query.size(4) / 2;
    TORCH_CHECK(dimhalf*2 == query.size(4), "Dims per head must be an even number in FP16.");
    int64_t kernel_size_sq = d_attn.size(4);
    int kernel_size = sqrt(kernel_size_sq);
    CHECK_FEATMAP(height, width, kernel_size, dilation);
    CHECK_KERNELSIZE("natten2dqkrpb_cuda_backward_fp16", kernel_size);
    int64_t RPB_MAX = kernel_size * 2 - 1;
   
    auto d_query = torch::zeros_like(query);
    auto d_key = torch::zeros_like(key);
    at::Tensor d_rpb;
    if (biasEnabled)
        d_rpb = torch::zeros({heads, RPB_MAX, RPB_MAX}, d_attn.options());

    int32_t n_rpb = heads * height * width * kernel_size_sq;
    int blocks_rpb = GET_BLOCKS(n_rpb, CUDA_NUM_THREADS_RPB16);
    dim3 grid_rpb(blocks_rpb);
    dim3 blockr(CUDA_NUM_THREADS_RPB16);
    int32_t nhalf_query = d_query.numel() / 2;
    int blocks_query = GET_BLOCKS(nhalf_query, CUDA_NUM_THREADS_Q16);
    dim3 grid_query(blocks_query);
    dim3 blockq(CUDA_NUM_THREADS_Q16);
    int32_t nhalf_key = d_key.numel() / 2;
    int blocks_key = GET_BLOCKS(nhalf_key, CUDA_NUM_THREADS_K16);
    dim3 grid_key(blocks_key);
    dim3 blockk(CUDA_NUM_THREADS_K16);
    const auto stream = c10::cuda::getCurrentCUDAStream();
    AT_DISPATCH_HALF_TYPES(at::kHalf, d_query.scalar_type(), "natten2dqkrpb_backward_cuda_fp16", ([&] {
        const auto d_attn_a = d_attn.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto query_a = query.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        const auto key_a = key.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        auto d_query_a = d_query.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        auto d_key_a = d_key.packed_accessor32<scalar_t,5,torch::DefaultPtrTraits>();
        if (biasEnabled) {
            auto d_rpb_a = d_rpb.packed_accessor32<scalar_t,3,torch::DefaultPtrTraits>();
            LAUNCH_DNA_KNS(kernel_size, dilation, natten2drpb_cuda_backward_kernel_fp16, grid_rpb, blockr, 0, stream,
                    d_rpb_a, d_attn_a, height, width, dilation, batch_size, d_rpb.numel(), n_rpb);
        }
        LAUNCH_DNA_KNS(kernel_size, dilation, natten2dq_cuda_backward_kernel_fp16, grid_query, blockq, 0, stream,
                d_query_a, d_attn_a, key_a, height, width, heads, dilation, dimhalf, nhalf_query);
        LAUNCH_DNA_KNS(kernel_size, dilation, natten2dk_cuda_backward_kernel_fp16, grid_key, blockk, 0, stream,
                d_key_a, d_attn_a, query_a, height, width, heads, dilation, dimhalf, nhalf_key);
    }));
    return {d_query, d_key, d_rpb};
}
} // namespace natten
