
#include <hip/hip_runtime.h>
#ifdef NATTEN_WITH_CUTLASS
#ifdef NATTEN_WITH_BLACKWELL_FNA
#include <cuda_runtime.h>
#include <iostream>
#include <ATen/ATen.h>
#include <ATen/cuda/CUDAContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>
#include <natten/natten.h>
#include <natten/helpers.h>
#include <natten/cuda/fmha_blackwell/fmha_backward.cuh>
#include <natten_autogen/cuda/blackwell_fmha_bwd/kernels.h>
namespace natten { 
namespace cuda { 
namespace fmha_blackwell { 




void blackwell_fmha_backward_float16_128x128x32(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_q_aligned,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<32>>;
  using Kernel = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::half_t, GemmShape, false>;
  using KernelWithResidualMask = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::half_t, GemmShape, true>;

  bool no_mask_required = seqlen_q % get<0>(GemmShape{}) == 0 && seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void blackwell_fmha_backward_float16_128x128x64(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_q_aligned,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::half_t, GemmShape, false>;
  using KernelWithResidualMask = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::half_t, GemmShape, true>;

  bool no_mask_required = seqlen_q % get<0>(GemmShape{}) == 0 && seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void blackwell_fmha_backward_float16_128x128x128(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_q_aligned,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::half_t, GemmShape, false>;
  using KernelWithResidualMask = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::half_t, GemmShape, true>;

  bool no_mask_required = seqlen_q % get<0>(GemmShape{}) == 0 && seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void blackwell_fmha_backward_bfloat16_128x128x32(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_q_aligned,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<32>>;
  using Kernel = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::bfloat16_t, GemmShape, false>;
  using KernelWithResidualMask = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::bfloat16_t, GemmShape, true>;

  bool no_mask_required = seqlen_q % get<0>(GemmShape{}) == 0 && seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void blackwell_fmha_backward_bfloat16_128x128x64(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_q_aligned,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::bfloat16_t, GemmShape, false>;
  using KernelWithResidualMask = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::bfloat16_t, GemmShape, true>;

  bool no_mask_required = seqlen_q % get<0>(GemmShape{}) == 0 && seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void blackwell_fmha_backward_bfloat16_128x128x128(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_q_aligned,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::bfloat16_t, GemmShape, false>;
  using KernelWithResidualMask = natten::cuda::fmha_blackwell::KernelBackward<
    cutlass::bfloat16_t, GemmShape, true>;

  bool no_mask_required = seqlen_q % get<0>(GemmShape{}) == 0 && seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        ptr_dQ,
        ptr_dK,
        ptr_dV,
        ptr_dO,
        batch_size,
        seqlen_q,
        seqlen_q_aligned,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}


} // namespace fmha_blackwell 
} // namespace cuda 
} // namespace natten 
#endif 
#endif 

