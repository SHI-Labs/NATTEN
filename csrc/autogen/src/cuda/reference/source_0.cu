
#include <hip/hip_runtime.h>
#ifdef NATTEN_WITH_CUTLASS
#include <cuda_runtime.h>
#include <iostream>
#include <ATen/ATen.h>
#include <ATen/cuda/CUDAContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>
#include <natten/natten.h>
#include <natten/helpers.h>
#include <natten/cuda/reference/fna_reference_forward.hpp>
#include <natten/cuda/reference/fna_reference_backward.hpp>
#include <natten_autogen/cuda/reference/kernels.h>
namespace natten { 
namespace cuda { 
namespace reference { 




void reference_fna1d_float32_causal0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::false_type>;

  fna_reference_forward(
    static_cast<float*>(ptr_Q),
    static_cast<float*>(ptr_K),
    static_cast<float*>(ptr_V),
    static_cast<float*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna1d_float32_causal1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::true_type>;

  fna_reference_forward(
    static_cast<float*>(ptr_Q),
    static_cast<float*>(ptr_K),
    static_cast<float*>(ptr_V),
    static_cast<float*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna1d_float16_causal0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::false_type>;

  fna_reference_forward(
    static_cast<cutlass::half_t*>(ptr_Q),
    static_cast<cutlass::half_t*>(ptr_K),
    static_cast<cutlass::half_t*>(ptr_V),
    static_cast<cutlass::half_t*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna1d_float16_causal1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::true_type>;

  fna_reference_forward(
    static_cast<cutlass::half_t*>(ptr_Q),
    static_cast<cutlass::half_t*>(ptr_K),
    static_cast<cutlass::half_t*>(ptr_V),
    static_cast<cutlass::half_t*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna1d_bfloat16_causal0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::false_type>;

  fna_reference_forward(
    static_cast<cutlass::bfloat16_t*>(ptr_Q),
    static_cast<cutlass::bfloat16_t*>(ptr_K),
    static_cast<cutlass::bfloat16_t*>(ptr_V),
    static_cast<cutlass::bfloat16_t*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna1d_bfloat16_causal1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::true_type>;

  fna_reference_forward(
    static_cast<cutlass::bfloat16_t*>(ptr_Q),
    static_cast<cutlass::bfloat16_t*>(ptr_K),
    static_cast<cutlass::bfloat16_t*>(ptr_V),
    static_cast<cutlass::bfloat16_t*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna1d_backward_float32_causal0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_DO,
      void* ptr_DQ,
      void* ptr_DK,
      void* ptr_DV,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::false_type>;

  fna_reference_backward(
    static_cast<float*>(ptr_Q),
    static_cast<float*>(ptr_K),
    static_cast<float*>(ptr_V),
    static_cast<float*>(ptr_O),
    static_cast<float*>(ptr_DO),
    static_cast<float*>(ptr_DQ),
    static_cast<float*>(ptr_DK),
    static_cast<float*>(ptr_DV),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna1d_backward_float32_causal1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_DO,
      void* ptr_DQ,
      void* ptr_DK,
      void* ptr_DV,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::true_type>;

  fna_reference_backward(
    static_cast<float*>(ptr_Q),
    static_cast<float*>(ptr_K),
    static_cast<float*>(ptr_V),
    static_cast<float*>(ptr_O),
    static_cast<float*>(ptr_DO),
    static_cast<float*>(ptr_DQ),
    static_cast<float*>(ptr_DK),
    static_cast<float*>(ptr_DV),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna1d_backward_float16_causal0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_DO,
      void* ptr_DQ,
      void* ptr_DK,
      void* ptr_DV,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::false_type>;

  fna_reference_backward(
    static_cast<cutlass::half_t*>(ptr_Q),
    static_cast<cutlass::half_t*>(ptr_K),
    static_cast<cutlass::half_t*>(ptr_V),
    static_cast<cutlass::half_t*>(ptr_O),
    static_cast<cutlass::half_t*>(ptr_DO),
    static_cast<cutlass::half_t*>(ptr_DQ),
    static_cast<cutlass::half_t*>(ptr_DK),
    static_cast<cutlass::half_t*>(ptr_DV),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna1d_backward_float16_causal1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_DO,
      void* ptr_DQ,
      void* ptr_DK,
      void* ptr_DV,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::true_type>;

  fna_reference_backward(
    static_cast<cutlass::half_t*>(ptr_Q),
    static_cast<cutlass::half_t*>(ptr_K),
    static_cast<cutlass::half_t*>(ptr_V),
    static_cast<cutlass::half_t*>(ptr_O),
    static_cast<cutlass::half_t*>(ptr_DO),
    static_cast<cutlass::half_t*>(ptr_DQ),
    static_cast<cutlass::half_t*>(ptr_DK),
    static_cast<cutlass::half_t*>(ptr_DV),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna1d_backward_bfloat16_causal0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_DO,
      void* ptr_DQ,
      void* ptr_DK,
      void* ptr_DV,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::false_type>;

  fna_reference_backward(
    static_cast<cutlass::bfloat16_t*>(ptr_Q),
    static_cast<cutlass::bfloat16_t*>(ptr_K),
    static_cast<cutlass::bfloat16_t*>(ptr_V),
    static_cast<cutlass::bfloat16_t*>(ptr_O),
    static_cast<cutlass::bfloat16_t*>(ptr_DO),
    static_cast<cutlass::bfloat16_t*>(ptr_DQ),
    static_cast<cutlass::bfloat16_t*>(ptr_DK),
    static_cast<cutlass::bfloat16_t*>(ptr_DV),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna1d_backward_bfloat16_causal1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_DO,
      void* ptr_DQ,
      void* ptr_DK,
      void* ptr_DV,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int> qkv_shape,
      cute::tuple<int> window_size,
      cute::tuple<int> stride,
      cute::tuple<int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::true_type>;

  fna_reference_backward(
    static_cast<cutlass::bfloat16_t*>(ptr_Q),
    static_cast<cutlass::bfloat16_t*>(ptr_K),
    static_cast<cutlass::bfloat16_t*>(ptr_V),
    static_cast<cutlass::bfloat16_t*>(ptr_O),
    static_cast<cutlass::bfloat16_t*>(ptr_DO),
    static_cast<cutlass::bfloat16_t*>(ptr_DQ),
    static_cast<cutlass::bfloat16_t*>(ptr_DK),
    static_cast<cutlass::bfloat16_t*>(ptr_DV),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna2d_float32_causal0x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::false_type, cute::false_type>;

  fna_reference_forward(
    static_cast<float*>(ptr_Q),
    static_cast<float*>(ptr_K),
    static_cast<float*>(ptr_V),
    static_cast<float*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna2d_float32_causal0x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::false_type, cute::true_type>;

  fna_reference_forward(
    static_cast<float*>(ptr_Q),
    static_cast<float*>(ptr_K),
    static_cast<float*>(ptr_V),
    static_cast<float*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna2d_float32_causal1x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::true_type, cute::false_type>;

  fna_reference_forward(
    static_cast<float*>(ptr_Q),
    static_cast<float*>(ptr_K),
    static_cast<float*>(ptr_V),
    static_cast<float*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna2d_float32_causal1x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::true_type, cute::true_type>;

  fna_reference_forward(
    static_cast<float*>(ptr_Q),
    static_cast<float*>(ptr_K),
    static_cast<float*>(ptr_V),
    static_cast<float*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna2d_float16_causal0x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::false_type, cute::false_type>;

  fna_reference_forward(
    static_cast<cutlass::half_t*>(ptr_Q),
    static_cast<cutlass::half_t*>(ptr_K),
    static_cast<cutlass::half_t*>(ptr_V),
    static_cast<cutlass::half_t*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna2d_float16_causal0x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::false_type, cute::true_type>;

  fna_reference_forward(
    static_cast<cutlass::half_t*>(ptr_Q),
    static_cast<cutlass::half_t*>(ptr_K),
    static_cast<cutlass::half_t*>(ptr_V),
    static_cast<cutlass::half_t*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna2d_float16_causal1x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::true_type, cute::false_type>;

  fna_reference_forward(
    static_cast<cutlass::half_t*>(ptr_Q),
    static_cast<cutlass::half_t*>(ptr_K),
    static_cast<cutlass::half_t*>(ptr_V),
    static_cast<cutlass::half_t*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna2d_float16_causal1x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::true_type, cute::true_type>;

  fna_reference_forward(
    static_cast<cutlass::half_t*>(ptr_Q),
    static_cast<cutlass::half_t*>(ptr_K),
    static_cast<cutlass::half_t*>(ptr_V),
    static_cast<cutlass::half_t*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}





void reference_fna2d_bfloat16_causal0x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen,
      int heads,
      int dim,
      int num_additional_kv,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      float attn_scale,
      cudaStream_t stream) {

  using Causal = cute::tuple<cute::false_type, cute::false_type>;

  fna_reference_forward(
    static_cast<cutlass::bfloat16_t*>(ptr_Q),
    static_cast<cutlass::bfloat16_t*>(ptr_K),
    static_cast<cutlass::bfloat16_t*>(ptr_V),
    static_cast<cutlass::bfloat16_t*>(ptr_O),
    static_cast<float*>(ptr_LSE),
    batch_size,
    seqlen,
    heads,
    dim,
    num_additional_kv,
    qkv_shape,
    window_size,
    stride,
    dilation,
    Causal{},
    attn_scale,
    stream);
}


} // namespace reference 
} // namespace cuda 
} // namespace natten 
#endif 

