#include <hip/hip_runtime.h>
#include <natten/cuda/gemm/na2d.cuh>
#include <natten/dtypes.cuh>
#include <natten/config.h>
#include <natten/gemm_argpack.cuh>
namespace natten { 
namespace cuda { 
namespace gemm { 

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks3_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks3_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks3_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks5_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks5_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks5_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks7_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks7_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks7_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks9_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 7, 1, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks9_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 7, 1, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks9_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 7, 1, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks11_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 6, 1, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks11_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 6, 1, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks11_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 6, 1, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks13_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 10, 1, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks13_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 10, 1, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_ks13_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 8, 8, 4, 2, 10, 1, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks15_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 9, 1, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks15_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 9, 1, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks15_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 9, 1, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks17_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 11, 1, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks17_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 11, 1, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks17_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 11, 1, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks19_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 13, 1, 9>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks19_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 13, 1, 9>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks19_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 13, 1, 9>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks21_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 12, 1, 10>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks21_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 12, 1, 10>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks21_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 12, 1, 10>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks23_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 14, 1, 11>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks23_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 14, 1, 11>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks23_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 14, 1, 11>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks25_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 13, 1, 12>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks25_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 13, 1, 12>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks25_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 13, 1, 12>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_8x8x4_2_sm70_ks27_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 8, 8, 4, 2, 14, 1, 13>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

} 
} 
} 

