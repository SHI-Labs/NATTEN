#include <hip/hip_runtime.h>
#include <iostream>
#include <natten/config.h>
#include <natten/cuda/gemm/na2d.cuh>
#include <natten/dtypes.cuh>
#include <natten/gemm_argpack.cuh>
namespace natten { 
namespace cuda { 
namespace gemm { 

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks49_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 24>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks49_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 24>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks49_align1(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 24>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks51_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 25>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks51_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 25>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks51_align1(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 25>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks53_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 26>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks53_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 26>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks53_align1(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 26>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks55_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 27>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks55_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 27>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks55_align1(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 27>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks57_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 28>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks57_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 28>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks57_align1(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 28>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks59_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 29>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks59_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 29>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks59_align1(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 29>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks61_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 30>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks61_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 30>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks61_align1(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 30>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks63_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 31>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks63_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 31>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks63_align1(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 8, 4, 31>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);

    }
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks3_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 8, 4, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks3_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 8, 4, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks3_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 8, 4, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks5_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks5_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks5_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

} 
} 
} 

