#include <hip/hip_runtime.h>
#include <natten/cuda/gemm/na2d.cuh>
#include <natten/dtypes.cuh>
#include <natten/config.h>
#include <natten/gemm_argpack.cuh>
namespace natten { 
namespace cuda { 
namespace gemm { 

void na2d_pn_cuda_gemm_double_64x64x16_32x32x16_8x8x4_3_sm80_ks3_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 32, 16, 8, 8, 4, 3, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_64x64x16_32x32x16_8x8x4_3_sm80_ks5_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 32, 16, 8, 8, 4, 3, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_64x64x16_32x32x16_8x8x4_3_sm80_ks7_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 32, 16, 8, 8, 4, 3, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_64x64x16_32x32x16_8x8x4_3_sm80_ks9_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 32, 16, 8, 8, 4, 3, 7, 1, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_64x64x16_32x32x16_8x8x4_3_sm80_ks11_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 32, 16, 8, 8, 4, 3, 6, 1, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_64x64x16_32x32x16_8x8x4_3_sm80_ks13_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 32, 16, 8, 8, 4, 3, 10, 1, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_128x128x16_64x64x16_8x8x4_3_sm80_ks15_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 8, 8, 4, 3, 9, 1, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_128x128x16_64x64x16_8x8x4_3_sm80_ks17_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 8, 8, 4, 3, 11, 1, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_128x128x16_64x64x16_8x8x4_3_sm80_ks19_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 8, 8, 4, 3, 13, 1, 9>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_128x128x16_64x64x16_8x8x4_3_sm80_ks21_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 8, 8, 4, 3, 12, 1, 10>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_128x128x16_64x64x16_8x8x4_3_sm80_ks23_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 8, 8, 4, 3, 14, 1, 11>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_128x128x16_64x64x16_8x8x4_3_sm80_ks25_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 8, 8, 4, 3, 13, 1, 12>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_128x128x16_64x64x16_8x8x4_3_sm80_ks27_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 8, 8, 4, 3, 14, 1, 13>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_128x128x16_64x64x16_8x8x4_3_sm80_ks29_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 8, 8, 4, 3, 17, 1, 14>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_128x128x16_64x64x16_8x8x4_3_sm80_ks31_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 8, 8, 4, 3, 17, 1, 15>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_double_128x128x16_64x64x16_8x8x4_3_sm80_ks33_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 8, 8, 4, 3, 17, 1, 16>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, double>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float64>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks3_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks3_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks3_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks5_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks5_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks5_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks7_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks7_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks7_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks9_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 7, 1, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks9_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 7, 1, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks9_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 7, 1, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks11_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 6, 1, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks11_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 6, 1, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks11_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 6, 1, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks13_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 10, 1, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks13_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 10, 1, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_64x64x16_32x16x16_16x8x8_3_sm80_ks13_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 16, 32, 16, 16, 16, 8, 8, 3, 10, 1, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks15_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 9, 1, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks15_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 9, 1, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks15_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 9, 1, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks17_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 11, 1, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks17_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 11, 1, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks17_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 11, 1, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks19_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 13, 1, 9>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks19_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 13, 1, 9>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks19_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 13, 1, 9>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks21_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 12, 1, 10>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks21_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 12, 1, 10>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks21_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 12, 1, 10>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks23_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 14, 1, 11>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks23_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 14, 1, 11>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks23_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 14, 1, 11>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks25_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 13, 1, 12>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks25_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 13, 1, 12>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks25_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 13, 1, 12>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks27_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 14, 1, 13>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks27_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 14, 1, 13>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks27_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 14, 1, 13>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks29_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 17, 1, 14>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks29_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 17, 1, 14>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks29_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 17, 1, 14>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks31_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 17, 1, 15>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks31_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 17, 1, 15>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks31_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 17, 1, 15>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks33_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 17, 1, 16>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks33_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 17, 1, 16>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_float_128x128x16_64x64x16_16x8x8_3_sm80_ks33_align1(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 16, 64, 64, 16, 16, 8, 8, 3, 17, 1, 16>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, float>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 1, 1>;
    if (natten::kEnableGemmTF32) { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::tf32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    } else { 
      using DConfig = natten::gemm::detail::DTypeConfig<natten::float32>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

    }
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks3_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks3_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks3_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks5_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks5_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks5_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks7_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks7_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks7_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks9_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 7, 1, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks9_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 7, 1, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks9_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 7, 1, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks11_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 6, 1, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks11_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 6, 1, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks11_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 6, 1, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks13_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 10, 1, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks13_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 10, 1, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_64x64x32_32x32x32_16x8x16_3_sm80_ks13_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 10, 1, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks15_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 9, 1, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks15_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 9, 1, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks15_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 9, 1, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks17_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 11, 1, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks17_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 11, 1, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks17_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 11, 1, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks19_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 13, 1, 9>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks19_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 13, 1, 9>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks19_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 13, 1, 9>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks21_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 12, 1, 10>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks21_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 12, 1, 10>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks21_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 12, 1, 10>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks23_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 14, 1, 11>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks23_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 14, 1, 11>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks23_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 14, 1, 11>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks25_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 13, 1, 12>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks25_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 13, 1, 12>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks25_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 13, 1, 12>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks27_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 14, 1, 13>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks27_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 14, 1, 13>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks27_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 14, 1, 13>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks29_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 17, 1, 14>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks29_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 17, 1, 14>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks29_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 17, 1, 14>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks31_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 17, 1, 15>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks31_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 17, 1, 15>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks31_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 17, 1, 15>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks33_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 17, 1, 16>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks33_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 17, 1, 16>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x16_3_sm80_ks33_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 16, 3, 17, 1, 16>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_bfloat16_64x64x32_32x32x32_16x8x16_3_sm80_ks3_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_BF16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, bfloat16>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::bfloat16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_bfloat16_64x64x32_32x32x32_16x8x16_3_sm80_ks3_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_BF16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, bfloat16>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::bfloat16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_bfloat16_64x64x32_32x32x32_16x8x16_3_sm80_ks3_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_BF16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 7, 1, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, bfloat16>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::bfloat16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_bfloat16_64x64x32_32x32x32_16x8x16_3_sm80_ks5_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_BF16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, bfloat16>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::bfloat16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_bfloat16_64x64x32_32x32x32_16x8x16_3_sm80_ks5_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_BF16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, bfloat16>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::bfloat16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_bfloat16_64x64x32_32x32x32_16x8x16_3_sm80_ks5_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_BF16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 6, 1, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, bfloat16>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::bfloat16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_bfloat16_64x64x32_32x32x32_16x8x16_3_sm80_ks7_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_BF16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, bfloat16>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::bfloat16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_bfloat16_64x64x32_32x32x32_16x8x16_3_sm80_ks7_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_BF16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, bfloat16>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::bfloat16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na2d_pn_cuda_gemm_bfloat16_64x64x32_32x32x32_16x8x16_3_sm80_ks7_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_BF16
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 16, 3, 5, 1, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<80, bfloat16>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::bfloat16>;
  using Kernel = PointwiseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, height, width, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

} 
} 
} 

