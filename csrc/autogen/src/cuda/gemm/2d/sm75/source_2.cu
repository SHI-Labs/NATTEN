#include <hip/hip_runtime.h>
#include <iostream>
#include <natten/config.h>
#include <natten/cuda/gemm/na2d.cuh>
#include <natten/dtypes.cuh>
#include <natten/gemm_argpack.cuh>
namespace natten { 
namespace cuda { 
namespace gemm { 

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks33_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 16>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks35_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 17>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks35_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 17>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks35_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 17>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks37_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 18>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks37_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 18>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks37_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 18>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks39_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 19>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks39_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 19>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks39_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 19>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks41_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 20>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks41_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 20>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks41_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 20>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks43_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 21>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks43_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 21>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks43_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 21>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks45_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 22>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks45_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 22>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks45_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 22>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks47_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 23>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks47_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 23>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks47_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 23>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks49_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 24>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks49_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 24>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks49_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 24>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks51_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 25>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks51_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 25>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks51_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 25>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks53_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 26>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks53_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 26>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks53_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 26>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks55_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 27>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks55_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 27>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks55_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 27>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks57_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 28>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks57_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 28>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks57_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 28>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks59_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 29>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks59_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 29>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks59_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 29>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks61_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 30>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks61_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 30>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks61_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 30>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks63_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 31>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks63_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 31>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_nn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_ks63_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<128, 128, 32, 64, 64, 32, 16, 8, 8, 2, 11, 2, 31>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks3_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks3_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks3_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 1>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks5_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks5_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks5_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks7_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks7_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks7_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 3>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks9_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks9_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks9_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 4>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks11_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks11_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks11_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 5>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks13_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks13_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks13_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 6>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks15_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks15_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks15_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 7>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks17_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks17_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

void na2d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_ks17_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int height,
  int width,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int64_t attn_stride_3,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig2D<64, 64, 32, 32, 32, 32, 16, 8, 8, 2, 8, 4, 8>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood2D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, height, width, dim, attn_stride_0, attn_stride_1, attn_stride_2, attn_stride_3, kernel_size, dilation, scale, stream);
}

} 
} 
} 

