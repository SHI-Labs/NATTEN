#include <hip/hip_runtime.h>
#include <natten/dtypes.cuh>
#include <natten/cuda/gemm/na1d.cuh>
#include <natten/gemm_argpack.cuh>
#include <natten/config.h>
namespace natten { 
namespace cuda { 
namespace gemm { 

void na1d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 16, 8, 8, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na1d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 16, 8, 8, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na1d_in_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 16, 8, 8, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

} 
} 
} 

