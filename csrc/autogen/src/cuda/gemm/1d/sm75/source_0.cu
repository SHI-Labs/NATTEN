#include <hip/hip_runtime.h>
#include <natten/dtypes.cuh>
#include <natten/cuda/gemm/na1d.cuh>
#include <natten/gemm_argpack.cuh>
#include <natten/config.h>
namespace natten { 
namespace cuda { 
namespace gemm { 

void na1d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_align8(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig<128, 128, 32, 64, 64, 32, 16, 8, 8, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<8, 8, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na1d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_align4(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig<128, 128, 32, 64, 64, 32, 16, 8, 8, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<4, 4, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na1d_pn_cuda_gemm_half_128x128x32_64x64x32_16x8x8_2_sm75_align2(
  void * query_ptr,
  void * key_ptr,
  void * attn_ptr,
  void * bias_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig<128, 128, 32, 64, 64, 32, 16, 8, 8, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<2, 2, 1>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = PointwiseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
query_ptr, key_ptr, attn_ptr, bias_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na1d_nn_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_align8(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 16, 8, 8, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na1d_nn_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_align4(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 16, 8, 8, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

void na1d_nn_cuda_gemm_half_64x64x32_32x32x32_16x8x8_2_sm75_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {

#ifdef NATTEN_ENABLE_FP16
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 16, 8, 8, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<75, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);

#else
std::cerr << "NATTEN was not built with support for this half type."  << std::endl; 
exit(EXIT_FAILURE); 

#endif
}

} 
} 
} 

