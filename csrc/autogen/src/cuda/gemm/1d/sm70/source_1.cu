#include <hip/hip_runtime.h>
#include <natten/gemm_argpack.cuh>
#include <natten/cuda/gemm/na1d.cuh>
#include <natten/config.h>
#include <natten/dtypes.cuh>
namespace natten { 
namespace cuda { 
namespace gemm { 

void na1d_in_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 8, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);
}

void na1d_in_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 8, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);
}

void na1d_in_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int kernel_size,
  int dilation,
  float scale) {
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 8, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, length, dim, kernel_size, dilation, scale);
}

} 
} 
} 

