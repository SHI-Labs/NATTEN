#include <hip/hip_runtime.h>
#include <iostream>
#include <natten/config.h>
#include <natten/cuda/gemm/na1d.cuh>
#include <natten/dtypes.cuh>
#include <natten/gemm_argpack.cuh>
namespace natten { 
namespace cuda { 
namespace gemm { 

void na1d_nn_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_align2(
  void * attn_ptr,
  void * value_ptr,
  void * output_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 8, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = NeighborhoodNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, value_ptr, output_ptr, batch_size, heads, length, dim, attn_stride_0, attn_stride_1, attn_stride_2, kernel_size, dilation, scale, stream);
}

void na1d_in_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_align8(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 8, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 8, 8>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, length, dim, attn_stride_0, attn_stride_1, attn_stride_2, kernel_size, dilation, scale, stream);
}

void na1d_in_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_align4(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 8, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 4, 4>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, length, dim, attn_stride_0, attn_stride_1, attn_stride_2, kernel_size, dilation, scale, stream);
}

void na1d_in_cuda_gemm_half_64x64x32_32x32x32_8x8x4_2_sm70_align2(
  void * attn_ptr,
  void * d_output_ptr,
  void * d_value_ptr,
  int batch_size,
  int heads,
  int length,
  int dim,
  int64_t attn_stride_0,
  int64_t attn_stride_1,
  int64_t attn_stride_2,
  int kernel_size,
  int dilation,
  float scale,
  hipStream_t stream) {
  using GConfig = natten::gemm::detail::GemmConfig<64, 64, 32, 32, 32, 32, 8, 8, 4, 2>;
  using ArchConfig = natten::gemm::detail::ArchArgs<70, half>;
  using AConfig = natten::gemm::detail::AlignmentConfig<1, 2, 2>;
  using DConfig = natten::gemm::detail::DTypeConfig<natten::float16>;
  using Kernel = InverseNeighborhood1D<GConfig, AConfig, DConfig, ArchConfig>;
  Kernel kernel;
  kernel(
attn_ptr, d_output_ptr, d_value_ptr, batch_size, heads, length, dim, attn_stride_0, attn_stride_1, attn_stride_2, kernel_size, dilation, scale, stream);
}

} 
} 
} 

