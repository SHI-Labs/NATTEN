#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <natten/cuda/fna/kernel_forward.h>
#include <natten_autogen/cuda/fna/kernels.h>
namespace natten { 
namespace cuda { 
namespace fna { 



///////////////////////////////////////////////////////////////////
// FNA-1D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 32>::kMinBlocksPerSm)
fna1d_32x128x32_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x32_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna1d_64x64x32_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x32_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 32>::kMinBlocksPerSm)
fna1d_64x128x32_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x32_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 64>::kMinBlocksPerSm)
fna1d_32x128x64_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x64_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna1d_64x64x64_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x64_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 64>::kMinBlocksPerSm)
fna1d_64x128x64_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x64_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 128>::kMinBlocksPerSm)
fna1d_32x128x128_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x128_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna1d_64x64x128_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x128_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 128>::kMinBlocksPerSm)
fna1d_64x128x128_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x128_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 65536>::kMinBlocksPerSm)
fna1d_32x128x65536_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x65536_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 65536>::kMinBlocksPerSm)
fna1d_64x128x65536_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x65536_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna1d_64x64x65536_sm80_float32_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x65536_sm80_float32_cm_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-1D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 32>::kMinBlocksPerSm)
fna1d_32x128x32_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x32_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna1d_64x64x32_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x32_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 32>::kMinBlocksPerSm)
fna1d_64x128x32_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x32_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 64>::kMinBlocksPerSm)
fna1d_32x128x64_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x64_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna1d_64x64x64_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x64_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 64>::kMinBlocksPerSm)
fna1d_64x128x64_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x64_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 128>::kMinBlocksPerSm)
fna1d_32x128x128_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x128_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna1d_64x64x128_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x128_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 128>::kMinBlocksPerSm)
fna1d_64x128x128_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x128_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 65536>::kMinBlocksPerSm)
fna1d_32x128x65536_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x65536_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 65536>::kMinBlocksPerSm)
fna1d_64x128x65536_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x65536_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna1d_64x64x65536_sm80_float32_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x65536_sm80_float32_cm_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
} 
} 
} 

