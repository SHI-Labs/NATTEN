#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <natten/dtypes.cuh>
#include <natten/cuda/fna/kernel_forward.h>
#include <natten_autogen/cuda/fna/kernels.h>
namespace natten { 
namespace cuda { 
namespace fna { 



///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32, false>::kMinBlocksPerSm)
fna3_32x128x32_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x32_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32, false>::kMinBlocksPerSm)
fna3_64x64x32_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x32_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32, false>::kMinBlocksPerSm)
fna3_64x128x32_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x32_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64, false>::kMinBlocksPerSm)
fna3_32x128x64_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x64_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64, false>::kMinBlocksPerSm)
fna3_64x64x64_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x64_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64, false>::kMinBlocksPerSm)
fna3_64x128x64_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x64_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128, false>::kMinBlocksPerSm)
fna3_32x128x128_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x128_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128, false>::kMinBlocksPerSm)
fna3_64x64x128_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x128_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128, false>::kMinBlocksPerSm)
fna3_64x128x128_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x128_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kMinBlocksPerSm)
fna3_32x128x65536_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x65536_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kMinBlocksPerSm)
fna3_64x128x65536_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x65536_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kMinBlocksPerSm)
fna3_64x64x65536_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x65536_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kMinBlocksPerSm)
fna3_32x128x32_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x32_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kMinBlocksPerSm)
fna3_64x64x32_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x32_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kMinBlocksPerSm)
fna3_64x128x32_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x32_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kMinBlocksPerSm)
fna3_32x128x64_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x64_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kMinBlocksPerSm)
fna3_64x64x64_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x64_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kMinBlocksPerSm)
fna3_64x128x64_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x64_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kMinBlocksPerSm)
fna3_32x128x128_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x128_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kMinBlocksPerSm)
fna3_64x64x128_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x128_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kMinBlocksPerSm)
fna3_64x128x128_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x128_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kMinBlocksPerSm)
fna3_32x128x65536_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x65536_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kMinBlocksPerSm)
fna3_64x128x65536_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x65536_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kMinBlocksPerSm)
fna3_64x64x65536_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x65536_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, true>::kMinBlocksPerSm)
fna3_32x128x32_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x32_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, true>::kMinBlocksPerSm)
fna3_64x64x32_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x32_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, true>::kMinBlocksPerSm)
fna3_64x128x32_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x32_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, true>::kMinBlocksPerSm)
fna3_32x128x64_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x64_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, true>::kMinBlocksPerSm)
fna3_64x64x64_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x64_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, true>::kMinBlocksPerSm)
fna3_64x128x64_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x64_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, true>::kMinBlocksPerSm)
fna3_32x128x128_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x128_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, true>::kMinBlocksPerSm)
fna3_64x64x128_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x128_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, true>::kMinBlocksPerSm)
fna3_64x128x128_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x128_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, true>::kMinBlocksPerSm)
fna3_32x128x65536_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x65536_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, true>::kMinBlocksPerSm)
fna3_64x128x65536_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x65536_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, true>::kMinBlocksPerSm)
fna3_64x64x65536_sm50_float16_cm_0_0_0_rpb(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x65536_sm50_float16_cm_0_0_0_rpb` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kMinBlocksPerSm)
fna3_32x128x32_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x32_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kMinBlocksPerSm)
fna3_64x64x32_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x32_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kMinBlocksPerSm)
fna3_64x128x32_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x32_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kMinBlocksPerSm)
fna3_32x128x64_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x64_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kMinBlocksPerSm)
fna3_64x64x64_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x64_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kMinBlocksPerSm)
fna3_64x128x64_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x64_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kMinBlocksPerSm)
fna3_32x128x128_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x128_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kMinBlocksPerSm)
fna3_64x64x128_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x128_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kMinBlocksPerSm)
fna3_64x128x128_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x128_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kMinBlocksPerSm)
fna3_32x128x65536_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x65536_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kMinBlocksPerSm)
fna3_64x128x65536_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x65536_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kMinBlocksPerSm)
fna3_64x64x65536_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x65536_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kMinBlocksPerSm)
fna3_32x128x32_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x32_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kMinBlocksPerSm)
fna3_64x64x32_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x32_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kMinBlocksPerSm)
fna3_64x128x32_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x32_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kMinBlocksPerSm)
fna3_32x128x64_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x64_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kMinBlocksPerSm)
fna3_64x64x64_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x64_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kMinBlocksPerSm)
fna3_64x128x64_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x64_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kMinBlocksPerSm)
fna3_32x128x128_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x128_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kMinBlocksPerSm)
fna3_64x64x128_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x128_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kMinBlocksPerSm)
fna3_64x128x128_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x128_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kMinBlocksPerSm)
fna3_32x128x65536_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x65536_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kMinBlocksPerSm)
fna3_64x128x65536_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x65536_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kMinBlocksPerSm)
fna3_64x64x65536_sm50_float16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x65536_sm50_float16_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kMinBlocksPerSm)
fna3_32x128x32_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x32_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kMinBlocksPerSm)
fna3_64x64x32_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x32_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kMinBlocksPerSm)
fna3_64x128x32_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x32_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kMinBlocksPerSm)
fna3_32x128x64_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x64_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kMinBlocksPerSm)
fna3_64x64x64_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x64_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kMinBlocksPerSm)
fna3_64x128x64_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x64_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kMinBlocksPerSm)
fna3_32x128x128_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x128_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kMinBlocksPerSm)
fna3_64x64x128_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x128_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kMinBlocksPerSm)
fna3_64x128x128_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x128_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kMinBlocksPerSm)
fna3_32x128x65536_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x65536_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kMinBlocksPerSm)
fna3_64x128x65536_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x65536_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kMinBlocksPerSm)
fna3_64x64x65536_sm50_float16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x65536_sm50_float16_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kMinBlocksPerSm)
fna3_32x128x32_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x32_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kMinBlocksPerSm)
fna3_64x64x32_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x32_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kMinBlocksPerSm)
fna3_64x128x32_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x32_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kMinBlocksPerSm)
fna3_32x128x64_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x64_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kMinBlocksPerSm)
fna3_64x64x64_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x64_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kMinBlocksPerSm)
fna3_64x128x64_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x64_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kMinBlocksPerSm)
fna3_32x128x128_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x128_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kMinBlocksPerSm)
fna3_64x64x128_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x128_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kMinBlocksPerSm)
fna3_64x128x128_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x128_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kMinBlocksPerSm)
fna3_32x128x65536_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x65536_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kMinBlocksPerSm)
fna3_64x128x65536_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x65536_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kMinBlocksPerSm)
fna3_64x64x65536_sm50_float16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x65536_sm50_float16_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kMinBlocksPerSm)
fna3_32x128x32_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x32_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kMinBlocksPerSm)
fna3_64x64x32_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x32_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kMinBlocksPerSm)
fna3_64x128x32_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x32_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kMinBlocksPerSm)
fna3_32x128x64_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x64_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kMinBlocksPerSm)
fna3_64x64x64_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x64_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kMinBlocksPerSm)
fna3_64x128x64_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x64_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kMinBlocksPerSm)
fna3_32x128x128_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x128_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kMinBlocksPerSm)
fna3_64x64x128_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x128_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kMinBlocksPerSm)
fna3_64x128x128_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x128_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kMinBlocksPerSm)
fna3_32x128x65536_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x65536_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kMinBlocksPerSm)
fna3_64x128x65536_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x65536_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kMinBlocksPerSm)
fna3_64x64x65536_sm50_float16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x65536_sm50_float16_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kMinBlocksPerSm)
fna3_32x128x32_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x32_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kMinBlocksPerSm)
fna3_64x64x32_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x32_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kMinBlocksPerSm)
fna3_64x128x32_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x32_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kMinBlocksPerSm)
fna3_32x128x64_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x64_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kMinBlocksPerSm)
fna3_64x64x64_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x64_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kMinBlocksPerSm)
fna3_64x128x64_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x64_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kMinBlocksPerSm)
fna3_32x128x128_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x128_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kMinBlocksPerSm)
fna3_64x64x128_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x128_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kMinBlocksPerSm)
fna3_64x128x128_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x128_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kMinBlocksPerSm)
fna3_32x128x65536_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x65536_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kMinBlocksPerSm)
fna3_64x128x65536_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x65536_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kMinBlocksPerSm)
fna3_64x64x65536_sm50_float16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x65536_sm50_float16_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::kMinBlocksPerSm)
fna3_32x128x32_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x32_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::kMinBlocksPerSm)
fna3_64x64x32_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x32_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::kMinBlocksPerSm)
fna3_64x128x32_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x32_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::kMinBlocksPerSm)
fna3_32x128x64_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x64_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::kMinBlocksPerSm)
fna3_64x64x64_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x64_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::kMinBlocksPerSm)
fna3_64x128x64_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x64_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::kMinBlocksPerSm)
fna3_32x128x128_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x128_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::kMinBlocksPerSm)
fna3_64x64x128_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x128_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::kMinBlocksPerSm)
fna3_64x128x128_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x128_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::kMinBlocksPerSm)
fna3_32x128x65536_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_32x128x65536_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::kMinBlocksPerSm)
fna3_64x128x65536_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x128x65536_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::kMinBlocksPerSm)
fna3_64x64x65536_sm50_float16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3_64x64x65536_sm50_float16_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
} 
} 
} 

