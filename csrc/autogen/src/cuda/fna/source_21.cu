#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <natten/dtypes.cuh>
#include <natten/cuda/fna/kernel_backward.h>
#include <natten/cuda/fna/kernel_forward.h>
#include <natten_autogen/cuda/fna/kernels.h>
namespace natten { 
namespace cuda { 
namespace fna { 



///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_0_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_0_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_0_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_0_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_1_0_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_1_0_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_1_0_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_1_0_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_1_1_0_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_1_1_0_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, false>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_32x128x32_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::kMinBlocksPerSm)
fna3d_64x64x32_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::kMinBlocksPerSm)
fna3d_64x128x32_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_32x128x64_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::kMinBlocksPerSm)
fna3d_64x64x64_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::kMinBlocksPerSm)
fna3d_64x128x64_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_32x128x128_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::kMinBlocksPerSm)
fna3d_64x64x128_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::kMinBlocksPerSm)
fna3d_64x128x128_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_32x128x65536_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x128x65536_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::kMinBlocksPerSm)
fna3d_64x64x65536_sm80_bfloat16_cm_1_1_1_lse(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536, false, true>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm80_bfloat16_cm_1_1_1_lse` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-1D / float32 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna1d_backward_64x64x32_sm50_float32_cm_0(typename FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_backward_64x64x32_sm50_float32_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna1d_backward_64x64x64_sm50_float32_cm_0(typename FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_backward_64x64x64_sm50_float32_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna1d_backward_64x64x128_sm50_float32_cm_0(typename FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_backward_64x64x128_sm50_float32_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna1d_backward_64x64x65536_sm50_float32_cm_0(typename FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_backward_64x64x65536_sm50_float32_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-1D / float32 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna1d_backward_64x64x32_sm50_float32_cm_1(typename FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_backward_64x64x32_sm50_float32_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna1d_backward_64x64x64_sm50_float32_cm_1(typename FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_backward_64x64x64_sm50_float32_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna1d_backward_64x64x128_sm50_float32_cm_1(typename FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_backward_64x64x128_sm50_float32_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna1d_backward_64x64x65536_sm50_float32_cm_1(typename FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<1, CausalMask<true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_backward_64x64x65536_sm50_float32_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
} 
} 
} 

