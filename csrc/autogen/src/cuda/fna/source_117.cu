#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <natten/cuda/fna/kernel_backward.h>
#include <natten_autogen/cuda/fna/kernels.h>
namespace natten { 
namespace cuda { 
namespace fna { 



///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kMinBlocksPerSm)
fna3d_backward_128x64x128_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x128_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kMinBlocksPerSm)
fna3d_backward_128x64x128_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x128_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
} 
} 
} 

