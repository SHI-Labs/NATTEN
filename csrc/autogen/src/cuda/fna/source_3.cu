#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <natten/cuda/fna/kernel_forward.h>
#include <natten_autogen/cuda/fna/kernels.h>
namespace natten { 
namespace cuda { 
namespace fna { 



///////////////////////////////////////////////////////////////////
// FNA-2D / float16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 32>::kMinBlocksPerSm)
fna2d_32x128x32_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x32_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna2d_64x64x32_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x32_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 32>::kMinBlocksPerSm)
fna2d_64x128x32_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x32_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 64>::kMinBlocksPerSm)
fna2d_32x128x64_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x64_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna2d_64x64x64_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x64_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 64>::kMinBlocksPerSm)
fna2d_64x128x64_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x64_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 128>::kMinBlocksPerSm)
fna2d_32x128x128_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x128_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna2d_64x64x128_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x128_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128>::kMinBlocksPerSm)
fna2d_64x128x128_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x128_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kMinBlocksPerSm)
fna2d_32x128x65536_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x65536_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kMinBlocksPerSm)
fna2d_64x128x65536_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x65536_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna2d_64x64x65536_sm80_float16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x65536_sm80_float16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-2D / float16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 32>::kMinBlocksPerSm)
fna2d_32x128x32_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x32_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna2d_64x64x32_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x32_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 32>::kMinBlocksPerSm)
fna2d_64x128x32_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x32_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 64>::kMinBlocksPerSm)
fna2d_32x128x64_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x64_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna2d_64x64x64_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x64_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 64>::kMinBlocksPerSm)
fna2d_64x128x64_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x64_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 128>::kMinBlocksPerSm)
fna2d_32x128x128_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x128_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna2d_64x64x128_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x128_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128>::kMinBlocksPerSm)
fna2d_64x128x128_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x128_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kMinBlocksPerSm)
fna2d_32x128x65536_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x65536_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kMinBlocksPerSm)
fna2d_64x128x65536_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x65536_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna2d_64x64x65536_sm80_float16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x65536_sm80_float16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-2D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::kMinBlocksPerSm)
fna2d_32x128x32_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x32_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna2d_64x64x32_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x32_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::kMinBlocksPerSm)
fna2d_64x128x32_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x32_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::kMinBlocksPerSm)
fna2d_32x128x64_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x64_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna2d_64x64x64_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x64_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::kMinBlocksPerSm)
fna2d_64x128x64_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x64_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::kMinBlocksPerSm)
fna2d_32x128x128_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x128_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna2d_64x64x128_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x128_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::kMinBlocksPerSm)
fna2d_64x128x128_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x128_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kMinBlocksPerSm)
fna2d_32x128x65536_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x65536_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kMinBlocksPerSm)
fna2d_64x128x65536_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x65536_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna2d_64x64x65536_sm80_bfloat16_cm_0_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x65536_sm80_bfloat16_cm_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-2D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::kMinBlocksPerSm)
fna2d_32x128x32_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x32_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna2d_64x64x32_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x32_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::kMinBlocksPerSm)
fna2d_64x128x32_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x32_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::kMinBlocksPerSm)
fna2d_32x128x64_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x64_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna2d_64x64x64_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x64_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::kMinBlocksPerSm)
fna2d_64x128x64_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x64_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::kMinBlocksPerSm)
fna2d_32x128x128_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x128_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna2d_64x64x128_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x128_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::kMinBlocksPerSm)
fna2d_64x128x128_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x128_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kMinBlocksPerSm)
fna2d_32x128x65536_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x65536_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kMinBlocksPerSm)
fna2d_64x128x65536_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x65536_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna2d_64x64x65536_sm80_bfloat16_cm_0_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<false, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x65536_sm80_bfloat16_cm_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-2D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::kMinBlocksPerSm)
fna2d_32x128x32_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x32_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna2d_64x64x32_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x32_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::kMinBlocksPerSm)
fna2d_64x128x32_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x32_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::kMinBlocksPerSm)
fna2d_32x128x64_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x64_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna2d_64x64x64_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x64_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::kMinBlocksPerSm)
fna2d_64x128x64_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x64_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::kMinBlocksPerSm)
fna2d_32x128x128_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x128_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna2d_64x64x128_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x128_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::kMinBlocksPerSm)
fna2d_64x128x128_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x128_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kMinBlocksPerSm)
fna2d_32x128x65536_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x65536_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kMinBlocksPerSm)
fna2d_64x128x65536_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x65536_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna2d_64x64x65536_sm80_bfloat16_cm_1_0(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, false>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x65536_sm80_bfloat16_cm_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-2D / bfloat16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::kMinBlocksPerSm)
fna2d_32x128x32_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x32_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna2d_64x64x32_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x32_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::kMinBlocksPerSm)
fna2d_64x128x32_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x32_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::kMinBlocksPerSm)
fna2d_32x128x64_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x64_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna2d_64x64x64_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x64_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::kMinBlocksPerSm)
fna2d_64x128x64_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x64_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::kMinBlocksPerSm)
fna2d_32x128x128_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x128_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna2d_64x64x128_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x128_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::kMinBlocksPerSm)
fna2d_64x128x128_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x128_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::kMinBlocksPerSm)
fna2d_32x128x65536_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_32x128x65536_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::kMinBlocksPerSm)
fna2d_64x128x65536_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x128x65536_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna2d_64x64x65536_sm80_bfloat16_cm_1_1(typename FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1300
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<2, CausalMask<true, true>, cutlass::bfloat16_t, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna2d_64x64x65536_sm80_bfloat16_cm_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna3d_32x128x32_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_64x64x32_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna3d_64x128x32_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna3d_32x128x64_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_64x64x64_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna3d_64x128x64_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna3d_32x128x128_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_64x64x128_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna3d_64x128x128_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna3d_32x128x65536_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna3d_64x128x65536_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_64x64x65536_sm50_float32_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm50_float32_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna3d_32x128x32_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_64x64x32_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna3d_64x128x32_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna3d_32x128x64_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_64x64x64_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna3d_64x128x64_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna3d_32x128x128_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_64x64x128_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna3d_64x128x128_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna3d_32x128x65536_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna3d_64x128x65536_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_64x64x65536_sm50_float32_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm50_float32_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna3d_32x128x32_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_64x64x32_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna3d_64x128x32_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna3d_32x128x64_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_64x64x64_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna3d_64x128x64_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna3d_32x128x128_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_64x64x128_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna3d_64x128x128_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna3d_32x128x65536_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna3d_64x128x65536_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_64x64x65536_sm50_float32_cm_0_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm50_float32_cm_0_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna3d_32x128x32_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_64x64x32_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna3d_64x128x32_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna3d_32x128x64_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_64x64x64_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna3d_64x128x64_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna3d_32x128x128_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_64x64x128_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna3d_64x128x128_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna3d_32x128x65536_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna3d_64x128x65536_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_64x64x65536_sm50_float32_cm_0_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm50_float32_cm_0_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna3d_32x128x32_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_64x64x32_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna3d_64x128x32_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna3d_32x128x64_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_64x64x64_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna3d_64x128x64_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna3d_32x128x128_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_64x64x128_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna3d_64x128x128_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna3d_32x128x65536_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna3d_64x128x65536_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_64x64x65536_sm50_float32_cm_1_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm50_float32_cm_1_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna3d_32x128x32_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_64x64x32_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna3d_64x128x32_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna3d_32x128x64_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_64x64x64_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna3d_64x128x64_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna3d_32x128x128_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_64x64x128_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna3d_64x128x128_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna3d_32x128x65536_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna3d_64x128x65536_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_64x64x65536_sm50_float32_cm_1_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm50_float32_cm_1_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna3d_32x128x32_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_64x64x32_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna3d_64x128x32_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna3d_32x128x64_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_64x64x64_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna3d_64x128x64_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna3d_32x128x128_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_64x64x128_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna3d_64x128x128_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna3d_32x128x65536_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna3d_64x128x65536_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_64x64x65536_sm50_float32_cm_1_1_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm50_float32_cm_1_1_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna3d_32x128x32_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_64x64x32_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna3d_64x128x32_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna3d_32x128x64_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_64x64x64_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna3d_64x128x64_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna3d_32x128x128_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_64x64x128_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna3d_64x128x128_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna3d_32x128x65536_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna3d_64x128x65536_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_64x64x65536_sm50_float32_cm_1_1_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm50_float32_cm_1_1_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna3d_32x128x32_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_64x64x32_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna3d_64x128x32_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna3d_32x128x64_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_64x64x64_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna3d_64x128x64_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna3d_32x128x128_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_64x64x128_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna3d_64x128x128_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna3d_32x128x65536_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna3d_64x128x65536_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_64x64x65536_sm50_float16_cm_0_0_0(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm50_float16_cm_0_0_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna3d_32x128x32_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x32_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_64x64x32_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x32_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna3d_64x128x32_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x32_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna3d_32x128x64_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x64_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_64x64x64_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x64_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna3d_64x128x64_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x64_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna3d_32x128x128_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x128_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_64x64x128_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x128_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna3d_64x128x128_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x128_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna3d_32x128x65536_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_32x128x65536_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna3d_64x128x65536_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x128x65536_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_64x64x65536_sm50_float16_cm_0_0_1(typename FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_64x64x65536_sm50_float16_cm_0_0_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
} 
} 
} 

