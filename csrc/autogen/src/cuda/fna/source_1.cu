#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <natten/cuda/fna/kernel_forward.h>
#include <natten_autogen/cuda/fna/kernels.h>
namespace natten { 
namespace cuda { 
namespace fna { 



///////////////////////////////////////////////////////////////////
// FNA-1D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna1d_32x128x32_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x32_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna1d_64x64x32_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x32_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna1d_64x128x32_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x32_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna1d_32x128x64_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x64_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna1d_64x64x64_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x64_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna1d_64x128x64_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x64_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna1d_32x128x128_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x128_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna1d_64x64x128_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x128_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna1d_64x128x128_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x128_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna1d_32x128x65536_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x65536_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna1d_64x128x65536_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x65536_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna1d_64x64x65536_sm50_float16_cm_0(typename FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<false>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x65536_sm50_float16_cm_0` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-1D / float16 / SM50
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::kMinBlocksPerSm)
fna1d_32x128x32_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x32_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::kMinBlocksPerSm)
fna1d_64x64x32_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x32_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::kMinBlocksPerSm)
fna1d_64x128x32_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x32_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::kMinBlocksPerSm)
fna1d_32x128x64_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x64_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::kMinBlocksPerSm)
fna1d_64x64x64_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x64_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::kMinBlocksPerSm)
fna1d_64x128x64_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x64_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::kMinBlocksPerSm)
fna1d_32x128x128_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x128_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::kMinBlocksPerSm)
fna1d_64x64x128_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x128_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::kMinBlocksPerSm)
fna1d_64x128x128_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x128_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::kMinBlocksPerSm)
fna1d_32x128x65536_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 32, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_32x128x65536_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::kMinBlocksPerSm)
fna1d_64x128x65536_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 128, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x128x65536_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::kMinBlocksPerSm)
fna1d_64x64x65536_sm50_float16_cm_1(typename FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 500
#if __CUDA_ARCH__ < 700
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionKernel<1, CausalMask<true>, cutlass::half_t, cutlass::arch::Sm50, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna1d_64x64x65536_sm50_float16_cm_1` was built for SM50, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
} 
} 
} 

