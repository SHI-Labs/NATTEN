#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <natten/dtypes.cuh>
#include <natten/cuda/fna/kernel_backward.h>
#include <natten_autogen/cuda/fna/kernels.h>
namespace natten { 
namespace cuda { 
namespace fna { 



///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM75
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm75_float16_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm75_float16_cm_1_1_0` was built for SM75, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm75_float16_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm75_float16_cm_1_1_0` was built for SM75, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm75_float16_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm75_float16_cm_1_1_0` was built for SM75, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm75_float16_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm75_float16_cm_1_1_0` was built for SM75, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM75
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm75_float16_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm75_float16_cm_1_1_1` was built for SM75, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm75_float16_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm75_float16_cm_1_1_1` was built for SM75, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm75_float16_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm75_float16_cm_1_1_1` was built for SM75, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm75_float16_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 750
#if __CUDA_ARCH__ < 800
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, cutlass::half_t, cutlass::arch::Sm75, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm75_float16_cm_1_1_1` was built for SM75, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float32_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float32_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float32_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float32_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float32_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float32_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kMinBlocksPerSm)
fna3d_backward_128x64x128_sm80_float32_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x128_sm80_float32_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float32_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float32_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float32_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float32_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float32_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float32_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float32_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float32_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float32_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float32_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kMinBlocksPerSm)
fna3d_backward_128x64x128_sm80_float32_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x128_sm80_float32_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float32_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float32_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float32_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float32_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float32_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float32_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float32_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float32_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float32_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float32_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kMinBlocksPerSm)
fna3d_backward_128x64x128_sm80_float32_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x128_sm80_float32_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float32_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float32_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float32_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float32_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float32_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float32_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float32_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float32_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float32_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float32_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kMinBlocksPerSm)
fna3d_backward_128x64x128_sm80_float32_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x128_sm80_float32_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float32_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float32_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float32_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float32_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float32_cm_1_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float32_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float32_cm_1_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float32_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float32_cm_1_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float32_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kMinBlocksPerSm)
fna3d_backward_128x64x128_sm80_float32_cm_1_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x128_sm80_float32_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float32_cm_1_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float32_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float32_cm_1_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float32_cm_1_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float32_cm_1_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float32_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float32_cm_1_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float32_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float32_cm_1_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float32_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kMinBlocksPerSm)
fna3d_backward_128x64x128_sm80_float32_cm_1_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x128_sm80_float32_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float32_cm_1_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float32_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float32_cm_1_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, false, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float32_cm_1_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kMinBlocksPerSm)
fna3d_backward_128x64x128_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x128_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float32_cm_1_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, false>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float32_cm_1_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float32 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::kMinBlocksPerSm)
fna3d_backward_128x64x128_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x128_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float32_cm_1_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<true, true, true>, float, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float32_cm_1_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float16_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float16_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float16_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float16_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float16_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float16_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::kMinBlocksPerSm)
fna3d_backward_128x128x128_sm80_float16_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x128x128_sm80_float16_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float16_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float16_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float16_cm_0_0_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float16_cm_0_0_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float16_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float16_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float16_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float16_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float16_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float16_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::kMinBlocksPerSm)
fna3d_backward_128x128x128_sm80_float16_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x128x128_sm80_float16_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float16_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float16_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float16_cm_0_0_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, false, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float16_cm_0_0_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float16_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float16_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float16_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::kMinBlocksPerSm)
fna3d_backward_128x128x128_sm80_float16_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x128x128_sm80_float16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float16_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float16_cm_0_1_0(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, false>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float16_cm_0_1_0` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


///////////////////////////////////////////////////////////////////
// FNA-3D / float16 / SM80
///////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::kMinBlocksPerSm)
fna3d_backward_64x64x32_sm80_float16_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 32>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x32_sm80_float16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::kMinBlocksPerSm)
fna3d_backward_64x64x64_sm80_float16_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 64>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x64_sm80_float16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::kMinBlocksPerSm)
fna3d_backward_64x64x128_sm80_float16_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x128_sm80_float16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::kMinBlocksPerSm)
fna3d_backward_128x128x128_sm80_float16_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 128, 128>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x128x128_sm80_float16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_64x64x65536_sm80_float16_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 64, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_64x64x65536_sm80_float16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}


__global__ void __launch_bounds__(
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::kNumThreads,
    FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::kMinBlocksPerSm)
fna3d_backward_128x64x65536_sm80_float16_cm_0_1_1(typename FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::Params p) {
#ifdef __CUDA_ARCH__
#if __CUDA_ARCH__ >= 800
#if __CUDA_ARCH__ < 1000
  if (!p.advance_to_block()) {
    return;
  }
  FusedNeighborhoodAttentionBackwardKernel<3, CausalMask<false, true, true>, cutlass::half_t, cutlass::arch::Sm80, true, 128, 64, 65536>::attention_kernel(p);
  return;
#endif
#endif
    printf(
        "FATAL: FNA kernel `fna3d_backward_128x64x65536_sm80_float16_cm_0_1_1` was built for SM80, but attempted to launch from SM%d\n",
        int(__CUDA_ARCH__ + 0) / 10);
#endif
}
} 
} 
} 

