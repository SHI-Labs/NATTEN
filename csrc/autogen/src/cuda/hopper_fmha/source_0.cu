
#include <hip/hip_runtime.h>
#ifdef NATTEN_WITH_CUTLASS
#ifdef NATTEN_WITH_HOPPER_FNA
#include <cuda_runtime.h>
#include <iostream>
#include <ATen/ATen.h>
#include <ATen/cuda/CUDAContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>
#include <natten/natten.h>
#include <natten/helpers.h>
#include <natten/cuda/fmha_hopper/fmha_forward.cuh>
#include <natten_autogen/cuda/hopper_fmha/kernels.h>
namespace natten { 
namespace cuda { 
namespace fmha_hopper { 




void hopper_fmha_float16_64x128x32(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<64>, cute::Int<128>, cute::Int<32>>;
  using Kernel = natten::cuda::fmha_hopper::KernelForward<
    cutlass::half_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::NonPersistent, false>;
  using KernelWithResidualMask = natten::cuda::fmha_hopper::KernelForward<
    cutlass::half_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::NonPersistent, true>;

  bool no_mask_required = seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void hopper_fmha_float16_64x128x64(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<64>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fmha_hopper::KernelForward<
    cutlass::half_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::NonPersistent, false>;
  using KernelWithResidualMask = natten::cuda::fmha_hopper::KernelForward<
    cutlass::half_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::NonPersistent, true>;

  bool no_mask_required = seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void hopper_fmha_float16_128x128x128_coop(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fmha_hopper::KernelForward<
    cutlass::half_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSCooperative, false>;
  using KernelWithResidualMask = natten::cuda::fmha_hopper::KernelForward<
    cutlass::half_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSCooperative, true>;

  bool no_mask_required = seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void hopper_fmha_float16_128x128x128_pp(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fmha_hopper::KernelForward<
    cutlass::half_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSPingpong, false>;
  using KernelWithResidualMask = natten::cuda::fmha_hopper::KernelForward<
    cutlass::half_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSPingpong, true>;

  bool no_mask_required = seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void hopper_fmha_float16_128x64x256_coop(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<64>, cute::Int<256>>;
  using Kernel = natten::cuda::fmha_hopper::KernelForward<
    cutlass::half_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSCooperative, false>;
  using KernelWithResidualMask = natten::cuda::fmha_hopper::KernelForward<
    cutlass::half_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSCooperative, true>;

  bool no_mask_required = seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void hopper_fmha_bfloat16_64x128x32(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<64>, cute::Int<128>, cute::Int<32>>;
  using Kernel = natten::cuda::fmha_hopper::KernelForward<
    cutlass::bfloat16_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::NonPersistent, false>;
  using KernelWithResidualMask = natten::cuda::fmha_hopper::KernelForward<
    cutlass::bfloat16_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::NonPersistent, true>;

  bool no_mask_required = seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void hopper_fmha_bfloat16_64x128x64(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<64>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fmha_hopper::KernelForward<
    cutlass::bfloat16_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::NonPersistent, false>;
  using KernelWithResidualMask = natten::cuda::fmha_hopper::KernelForward<
    cutlass::bfloat16_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::NonPersistent, true>;

  bool no_mask_required = seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void hopper_fmha_bfloat16_128x128x128_coop(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fmha_hopper::KernelForward<
    cutlass::bfloat16_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSCooperative, false>;
  using KernelWithResidualMask = natten::cuda::fmha_hopper::KernelForward<
    cutlass::bfloat16_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSCooperative, true>;

  bool no_mask_required = seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void hopper_fmha_bfloat16_128x128x128_pp(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fmha_hopper::KernelForward<
    cutlass::bfloat16_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSPingpong, false>;
  using KernelWithResidualMask = natten::cuda::fmha_hopper::KernelForward<
    cutlass::bfloat16_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSPingpong, true>;

  bool no_mask_required = seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}





void hopper_fmha_bfloat16_128x64x256_coop(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using GemmShape = cute::tuple<cute::Int<128>, cute::Int<64>, cute::Int<256>>;
  using Kernel = natten::cuda::fmha_hopper::KernelForward<
    cutlass::bfloat16_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSCooperative, false>;
  using KernelWithResidualMask = natten::cuda::fmha_hopper::KernelForward<
    cutlass::bfloat16_t, GemmShape, natten::cuda::hopper::HopperKernelSchedule::WSCooperative, true>;

  bool no_mask_required = seqlen_k % get<1>(GemmShape{}) == 0;
  if (no_mask_required) {
    Kernel kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  } else {
    KernelWithResidualMask kernel;
    auto args = kernel.initialize(
        ptr_Q,
        ptr_K,
        ptr_V,
        ptr_O,
        ptr_LSE,
        batch_size,
        seqlen_q,
        seqlen_k,
        heads,
        dim,
        device_id,
        attn_scale);

    auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
    auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
    auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
    kernel.run(args, workspace_ptr, stream);
  }
}


} // namespace fmha_hopper 
} // namespace cuda 
} // namespace natten 
#endif 
#endif 

