
#include <hip/hip_runtime.h>
#ifdef NATTEN_WITH_CUTLASS
#ifdef NATTEN_WITH_BLACKWELL_FNA
#include <cuda_runtime.h>
#include <iostream>
#include <ATen/ATen.h>
#include <ATen/cuda/CUDAContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>
#include <natten/natten.h>
#include <natten/helpers.h>
#include <natten/cuda/fna_blackwell/fna_backward.cuh>
#include <natten_autogen/cuda/blackwell_fna_bwd/kernels.h>
namespace natten { 
namespace cuda { 
namespace fna_blackwell { 




void blackwell_fna2d_backward_bfloat16_128x128x64_Q16x8_KV16x8_causal0x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using KVTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x64_Q16x8_KV8x16_causal0x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using KVTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x64_Q8x16_KV16x8_causal0x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using KVTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x64_Q8x16_KV8x16_causal0x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using KVTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x64_Q16x8_KV16x8_causal1x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::true_type, cute::false_type>;
  using QTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using KVTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x64_Q16x8_KV8x16_causal1x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::true_type, cute::false_type>;
  using QTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using KVTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x64_Q8x16_KV16x8_causal1x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::true_type, cute::false_type>;
  using QTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using KVTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x64_Q8x16_KV8x16_causal1x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::true_type, cute::false_type>;
  using QTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using KVTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x64_Q16x8_KV16x8_causal1x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::true_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using KVTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x64_Q16x8_KV8x16_causal1x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::true_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using KVTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x64_Q8x16_KV16x8_causal1x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::true_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using KVTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x64_Q8x16_KV8x16_causal1x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::true_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using KVTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<64>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x128_Q16x8_KV16x8_causal0x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::false_type>;
  using QTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using KVTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x128_Q16x8_KV8x16_causal0x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::false_type>;
  using QTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using KVTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x128_Q8x16_KV16x8_causal0x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::false_type>;
  using QTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using KVTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x128_Q8x16_KV8x16_causal0x0(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::false_type>;
  using QTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using KVTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x128_Q16x8_KV16x8_causal0x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using KVTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x128_Q16x8_KV8x16_causal0x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using KVTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x128_Q8x16_KV16x8_causal0x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using KVTileShape = cute::tuple<cute::Int<16>, cute::Int<8>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}





void blackwell_fna2d_backward_bfloat16_128x128x128_Q8x16_KV8x16_causal0x1(
      void* ptr_Q,
      void* ptr_K,
      void* ptr_V,
      void* ptr_O,
      void* ptr_LSE,
      void* ptr_dQ,
      void* ptr_dK,
      void* ptr_dV,
      void* ptr_dO,
      int batch_size,
      int seqlen_q,
      int seqlen_k,
      int heads,
      int dim,
      cute::tuple<int, int> q_shape,
      cute::tuple<int, int> kv_shape,
      cute::tuple<int, int> qkv_shape,
      cute::tuple<int, int> window_size,
      cute::tuple<int, int> stride,
      cute::tuple<int, int> dilation,
      int device_id,
      float attn_scale,
      cudaStream_t stream,
      at::TensorOptions tensor_options) {

  using Causal = cute::tuple<cute::false_type, cute::true_type>;
  using QTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using KVTileShape = cute::tuple<cute::Int<8>, cute::Int<16>>;
  using Config = cute::tuple<cute::Int<128>, cute::Int<128>, cute::Int<128>>;
  using Kernel = natten::cuda::fna_blackwell::KernelBackward<
    cutlass::bfloat16_t, Causal, QTileShape, KVTileShape, Config>;

  Kernel kernel;
  auto args = kernel.initialize(
      ptr_Q,
      ptr_K,
      ptr_V,
      ptr_O,
      ptr_LSE,
      ptr_dQ,
      ptr_dK,
      ptr_dV,
      ptr_dO,
      batch_size,
      seqlen_q,
      seqlen_k,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale);

  auto bytes = static_cast<int64_t>(kernel.get_workspace_size(args));
  auto workspace = at::empty({bytes}, tensor_options.dtype(at::ScalarType::Byte));
  auto workspace_ptr = static_cast<void*>(workspace.data_ptr());
  kernel.run(args, workspace_ptr, stream);
}


} // namespace fna_blackwell 
} // namespace cuda 
} // namespace natten 
#endif 
#endif 

