#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2024 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *

 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
/*! \file
    Example usage:
      $ ./examples/88_hopper_fmha/88_hopper_fmha \
            --b=2048 --h=2048 --d=2048 --q=2048 --k=2048
*/

#include <iostream>
#include <hip/hip_runtime.h>

#include "cute/tensor.hpp"
#include "cute/util/print.hpp"
#include "cutlass/cutlass.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/tensor_fill.h"

#include "reference.hpp"
#include "bwd_reference.hpp"
#include "error.hpp"

#include "flash.h"

#include "flash_fwd_sm80_test.cu"
#include "flash_bwd_sm80_test.cu"

#include "flash_fwd_launch_template.h"
#include "flash_bwd_launch_template.h"

using namespace cute;
using namespace natten::cuda::flash;

///////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void print_O_and_LSE(void* block_O, void* block_ref_O,
    void* block_LSE, void* block_ref_LSE){
    if (thread0()){
      print("============= DEBUG ============"); print("\n");
      print("block_O[0]="); print(*((cutlass::half_t*)block_O + 0)); print("\n");
      print("block_O[1]="); print(*((cutlass::half_t*)block_O + 1)); print("\n");
      print("block_O[2]="); print(*((cutlass::half_t*)block_O + 2)); print("\n");
      print("block_ref_O[0]="); print(*((cutlass::half_t*)block_ref_O + 0)); print("\n");
      print("block_ref_O[1]="); print(*((cutlass::half_t*)block_ref_O + 1)); print("\n");
      print("block_ref_O[2]="); print(*((cutlass::half_t*)block_ref_O + 2)); print("\n");
      print("block_LSE[0]="); print(*((float*)block_LSE + 0)); print("\n");
      print("block_LSE[1]="); print(*((float*)block_LSE + 1)); print("\n");
      print("block_LSE[2]="); print(*((float*)block_LSE + 2)); print("\n");
      print("block_ref_LSE[0]="); print(*((float*)block_ref_LSE + 0)); print("\n");
      print("block_ref_LSE[1]="); print(*((float*)block_ref_LSE + 1)); print("\n");
      print("block_ref_LSE[2]="); print(*((float*)block_ref_LSE + 2)); print("\n");
    }
}

__global__ void print_dQ_dK_dV(void* block_dQ, void* block_ref_dQ,
                             void* block_dK, void* block_ref_dK,
                             void* block_dV, void* block_ref_dV) {
    if (thread0()) {
        print("============= DEBUG ============"); print("\n");

        // dQ
        print("block_dQ[0]="); print(*((cutlass::half_t*)block_dQ + 0)); print("\n");
        print("block_dQ[1]="); print(*((cutlass::half_t*)block_dQ + 1)); print("\n");
        print("block_dQ[2]="); print(*((cutlass::half_t*)block_dQ + 2)); print("\n");

        print("block_ref_dQ[0]="); print(*((cutlass::half_t*)block_ref_dQ + 0)); print("\n");
        print("block_ref_dQ[1]="); print(*((cutlass::half_t*)block_ref_dQ + 1)); print("\n");
        print("block_ref_dQ[2]="); print(*((cutlass::half_t*)block_ref_dQ + 2)); print("\n");

        // dK
        print("block_dK[0]="); print(*((cutlass::half_t*)block_dK + 0)); print("\n");
        print("block_dK[1]="); print(*((cutlass::half_t*)block_dK + 1)); print("\n");
        print("block_dK[2]="); print(*((cutlass::half_t*)block_dK + 2)); print("\n");

        print("block_ref_dK[0]="); print(*((cutlass::half_t*)block_ref_dK + 0)); print("\n");
        print("block_ref_dK[1]="); print(*((cutlass::half_t*)block_ref_dK + 1)); print("\n");
        print("block_ref_dK[2]="); print(*((cutlass::half_t*)block_ref_dK + 2)); print("\n");

        // dV
        print("block_dV[0]="); print(*((cutlass::half_t*)block_dV + 0)); print("\n");
        print("block_dV[1]="); print(*((cutlass::half_t*)block_dV + 1)); print("\n");
        print("block_dV[2]="); print(*((cutlass::half_t*)block_dV + 2)); print("\n");

        print("block_ref_dV[0]="); print(*((cutlass::half_t*)block_ref_dV + 0)); print("\n");
        print("block_ref_dV[1]="); print(*((cutlass::half_t*)block_ref_dV + 1)); print("\n");
        print("block_ref_dV[2]="); print(*((cutlass::half_t*)block_ref_dV + 2)); print("\n");
    }
}

__global__ void print_tensors(void** blocks, int num_tensors) {
    if (thread0()) {
        print("============= DEBUG ============"); print("\n");

        for (int i = 0; i < num_tensors; ++i) {
            print("Tensor "); print(i); print(":\n");

            print("  [0]=");
            print(*((cutlass::half_t*)blocks[i] + 0)); print("\n");

            print("  [1]=");
            print(*((cutlass::half_t*)blocks[i] + 1)); print("\n");

            print("  [2]=");
            print(*((cutlass::half_t*)blocks[i] + 2)); print("\n");
        }
    }
}

/// Command line options parsing
struct Options {

  bool help;
  bool error;

  int b, h, q, k, d;
  int iterations;
  bool verify;
  bool verbose;
  bool causal;
  bool residual;
  bool bwd;

  Options():
    help(false),
    error(false),
    b(16), h(16), q(1024), k(1024), d(128),
    iterations(20), verify(true),
    causal(false), residual(false), bwd(false), verbose(false)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    Options defaults;

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("d", d, defaults.d);
    cmd.get_cmd_line_argument("h", h, -1);
    if (h == -1) h = 2048 / d;

    cmd.get_cmd_line_argument("q", q, -1);
    cmd.get_cmd_line_argument("k", k, -1);
    if (q == -1) q = k;
    if (k == -1) k = q;
    if (q == -1 && k == -1) q = k = defaults.q;

    cmd.get_cmd_line_argument("b", b, -1);
    if (b == -1) b = 16384 / k;
    if (b == 0) b = 1;

    cmd.get_cmd_line_argument("iterations", iterations, defaults.iterations);
    verify = true;
    verbose = cmd.check_cmd_line_flag("verbose");

    std::string mask;
    cmd.get_cmd_line_argument<std::string>("mask", mask, "");
    if (mask == "no" || mask == "") {
      causal = residual = false;
    }
    else if (mask == "causal") {
      residual = false;
      causal = true;
    }
    else if (mask == "residual") {
      residual = true;
      causal = false;
    }

    bwd = cmd.check_cmd_line_flag("bwd");
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "88_hopper_fmha\n\n"
      << "  This example showcases the use of CUTLASS's collective operation builders to easily construct\n"
      << "  fused multi-head attention forward-pass kernels targeting NVIDIA's Hopper architecture.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement\n\n"
      << "  --b=<int>                   Sets the B extent\n"
      << "  --h=<int>                   Sets the H extent\n"
      << "  --q=<int>                   Sets the Q extent\n"
      << "  --k=<int>                   Sets the K extent\n"
      << "  --d=<int>                   Sets the D extent\n"
      << "  --iterations=<int>          Benchmarking iterations\n"
      << "  --verify                    Verify results\n"
      << "  --verbose                   Print smem and execution time per kernel\n"
      << "  --mask=<no|residual|causal> Enables masking\n"
      << "  --bwd                       Runs the backwards pass\n"
      << "\n";

    return out;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to initialize a block of device data
template <class Element>
bool initialize_block(
  cutlass::DeviceAllocation<Element>& block,
  uint64_t seed=2023, bool init_one=false) {

  if (init_one) {
    cutlass::reference::device::BlockFillRandomUniform(
      block.get(), block.size(), seed, (Element) 1, (Element) 1);
  } else {
    cutlass::reference::device::BlockFillRandomGaussian(
      block.get(), block.size(), seed, (Element) 0, (Element) 1);
  }

  return true;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

struct ExampleResult {
  bool passed = false;
  bool verified = false;
  float runtime_ms = 0;
  double tflops_s = 0;
  size_t smem_size = 0;
};

///////////////////////////////////////////////////////////////////////////////////////////////////

// #if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

struct DefaultFusion {
  template<class BlkCoord, class TileShape, class ProblemSize>
  CUTLASS_DEVICE
  int get_trip_count(
    BlkCoord const& blk_coord,
    TileShape const& tile_shape,
    ProblemSize const& problem_size
  ) {
    return ceil_div(get<3>(problem_size), get<1>(tile_shape));
  }

  template<class BlkCoord, class TileShape, class ProblemSize>
  CUTLASS_DEVICE
  int get_masked_trip_count(
    BlkCoord const& blk_coord,
    TileShape const& tile_shape,
    ProblemSize const& problem_size
  ) {
    return get_trip_count(blk_coord, tile_shape, problem_size);
  }

  template<class BlkCoord, class TileShape, class ProblemSize>
  CUTLASS_DEVICE
  int get_unmasked_trip_count(
    BlkCoord const& blk_coord,
    TileShape const& tile_shape,
    ProblemSize const& problem_size
  ) {
    return 0;
  }

  template<class AccQK, class IndexQK, class ProblemSize>
  CUTLASS_DEVICE
  void before_softmax(
    AccQK& acc_qk,
    IndexQK const& index_qk,
    ProblemSize const& problem_size

  ) {
    return;
  }
};


template<
  class TileShape,
  class... KernelOptions
>
struct FwdRunner {

#ifdef FP8
  using Element = cutlass::float_e4m3_t;
  using ElementAccumulatorQK = find_option_t<Tag::kAccQK, float, KernelOptions...>;
#else
  using Element = cutlass::half_t;
  using ElementAccumulatorQK = float;
#endif

  using ElementAccumulatorPV = float;

  // B H Q K D
  using ProblemShapeType = cute::tuple<int, int, int, int, int>;

  
  using StrideQ = cute::tuple<int, _1, cute::tuple<int, int>>;  // Q D (B H)
  using StrideK = cute::tuple<int, _1, cute::tuple<int, int>>;  // K D (B H)
  using StrideV = std::conditional_t<sizeof(Element) == 1,
    cute::tuple<_1, int, cute::tuple<int, int>>,
    cute::tuple<int, _1, cute::tuple<int, int>>>;  // K D (B H)
  using StrideO = cute::tuple<int, _1, cute::tuple<int, int>>; // Q D (B H)
  using StrideLSE = cute::tuple<_1, cute::tuple<int, int>>; // Q (B H)

  // using Operation = cutlass::device::Universal<
  //   typename cutlass::fmha::kernel::FmhaBuilder<
  //     Element, ElementAccumulatorQK, ElementAccumulatorPV,
  //     TileShape, StrideQ, StrideK, StrideV,
  //     ActiveFusion, DispatchPolicy, KernelOptions...
  //   >::Kernel>;

  //
  // Data members
  //

  /// Initialization
  StrideQ stride_Q;
  StrideK stride_K;
  StrideV stride_V;
  StrideO stride_O;
  StrideLSE stride_LSE;
  uint64_t seed = 0;

  cutlass::DeviceAllocation<Element> block_Q;
  cutlass::DeviceAllocation<Element> block_K;
  cutlass::DeviceAllocation<Element> block_V;
  cutlass::DeviceAllocation<Element> block_O;
  cutlass::DeviceAllocation<ElementAccumulatorPV> block_LSE;
  cutlass::DeviceAllocation<Element> block_ref_O;
  cutlass::DeviceAllocation<ElementAccumulatorPV> block_ref_LSE;

  //
  // Methods
  //
  bool verify(const ProblemShapeType& problem_size) {
    auto [B, H, Q, K, D] = problem_size;

    Tensor mQ = make_tensor(make_gmem_ptr(block_Q.get()),
      make_shape(Q, D, make_shape(B, H)),
      stride_Q);

    Tensor mK = make_tensor(make_gmem_ptr(block_K.get()),
      make_shape(K, D, make_shape(B, H)),
      stride_K);

    Tensor mV = make_tensor(make_gmem_ptr(block_V.get()),
      make_shape(K, D, make_shape(B, H)),
      stride_V);

    Tensor mO = make_tensor(make_gmem_ptr(block_ref_O.get()),
      make_shape(Q, D, make_shape(B, H)),
      stride_O);

    Tensor mLSE = make_tensor(make_gmem_ptr(block_ref_LSE.get()),
      make_shape(Q, make_shape(B, H)),
      stride_LSE);

    fmha_reference(problem_size, mQ, mK, mV, mO, mLSE, DefaultFusion{});
    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    const double kMaxDiffThresh = sizeof(Element) == 1 ? 1e-1 : 1e-2;
    const double kMeanDiffThresh = sizeof(Element) == 1 ? 1e-1 : 1e-3;

    dim3 blk(1, 1, 1);
    dim3 grd(1, 1, 1);
    print_O_and_LSE<<<grd, blk>>>(block_O.get(), block_ref_O.get(), block_LSE.get(),
        block_ref_LSE.get());
    // Check if output from CUTLASS kernel and reference kernel are equal or not
    double max_diff = 0;
    double mean_diff = 0;
    reference_abs_diff(block_O, block_ref_O, max_diff, mean_diff);
    bool passed_O = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if (! passed_O) {
      std::cerr << "failed O: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    reference_abs_diff(block_LSE, block_ref_LSE, max_diff, mean_diff);
    bool passed_LSE = (max_diff < kMaxDiffThresh) && (mean_diff < kMeanDiffThresh);
    if ( ! passed_LSE) {
      std::cerr << "failed LSE: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    std::cout << "passed_O: " << passed_O << std::endl;
    std::cout << "passed_LSE: " << passed_LSE << std::endl;

    // if (!(passed_O && passed_LSE)){
    // }

    return passed_O && passed_LSE;
  }


  void initialize_stride(cute::tuple<int, int, int> const& shape, cute::tuple<_1, cute::tuple<int, int>>& stride) {
    auto [B, H, Q] = shape;
    stride = make_stride(_1{}, make_stride(H*Q, Q));
  }

  void initialize_stride(cute::tuple<int, int, int, int> const& shape, cute::tuple<int, _1, cute::tuple<int, int>>& stride) {
    auto [B, H, Q, D] = shape;
    stride = make_stride(D, _1{}, make_stride(H*Q*D, Q*D));
  }

  void initialize_stride(cute::tuple<int, int, int, int> const& shape, cute::tuple<_1, int, cute::tuple<int, int>>& stride) {
    auto [B, H, Q, D] = shape;
    stride = make_stride(_1{}, Q, make_stride(H*Q*D, Q*D));
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(const ProblemShapeType& problem_size) {
    auto [B, H, Q, K, D] = problem_size;
    D = cutlass::round_up(D, 8); // Alignment

    auto shape_QO = cute::make_shape(B, H, Q, D);
    auto shape_KV = cute::make_shape(B, H, K, D);
    auto shape_LSE = cute::make_shape(B, H, Q);

    initialize_stride(shape_QO, stride_Q);
    initialize_stride(shape_KV, stride_K);
    initialize_stride(shape_KV, stride_V);
    initialize_stride(shape_QO, stride_O);
    initialize_stride(shape_LSE, stride_LSE);

    block_Q.reset(size(shape_QO));
    block_K.reset(size(shape_KV));
    block_V.reset(size(shape_KV));
    block_O.reset(size(shape_QO));
    block_LSE.reset(size(shape_LSE));
    block_ref_O.reset(size(shape_QO));
    block_ref_LSE.reset(size(shape_LSE));

    initialize_block(block_Q, seed + 2023, false);
    initialize_block(block_K, seed + 2022, false);
    initialize_block(block_V, seed + 2021, false);
  }

  // ExampleResult run(const Options& options, const cutlass::KernelHardwareInfo& hw_info) {
  ExampleResult run(const Options& options) {
    ProblemShapeType problem_size = ProblemShapeType{options.b, options.h, options.q, options.k, options.d};

    initialize(problem_size);

    Flash_fwd_params params = set_flash_fwd_params_for_testing(
      problem_size,
      block_Q.get(),
      block_K.get(),
      block_V.get(),
      block_O.get(),
      block_LSE.get(),
      stride_Q,
      stride_K,
      stride_V,
      stride_O,
      stride_LSE,
      1.0 / sqrt(double(options.d)) /* softmax_scale */ 
    );
    // print_strides(params);

    ExampleResult example_result;
    example_result.smem_size = 0; // Operation::Kernel::SharedStorageSize;

    hipError_t result = hipDeviceSynchronize();
    cutlass::Status status;
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamDefault);


    // What needs to be done here:
    // Initialize and run a flash attention kernel and check if it correctly achieves the same throughput

    // Test scope:
    // Arch: SM80
    // Dtype: FP16
    // HeadDim: 128

    auto flash_fwd_fn = run_flash_fwd<
      /* Arch= */ 80,
      /* kHeadDim= */ 128,
      /* kHeadDimV= */ 128,
      /* kBlockM= */ 128,
      /* kBlockN= */ 112,
      /* T= */ cutlass::half_t,
      /* T_out= */ cutlass::half_t,
      /* PackGQA= */ false, 
      /* V_colmajor= */ false
      // /* Is_causal= */ false, 
      // /* Is_local= */ false,
      // /* Has_softcap= */ false, 
      // /* Varlen= */ false,
      // /* AppendKV= */ false,
      // /* PagedKVNonTMA= */ false, 
      // /* HasQv= */ false,
      // /* Split= */ false, 
    >;

    hipEvent_t events[2];

    for (auto & event : events) {
      result = hipEventCreate(&event);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }
    }

    // Record an event at the start of a series of GEMMs
    result = hipEventRecord(events[0]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    for (int i = 0; i < options.iterations; i++) {
      // status = op.run();
      flash_fwd_fn(params, stream);
      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                  << hipGetErrorString(hipGetLastError()) << std::endl;
        return example_result;
      }
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMMs are complete
    result = hipEventRecord(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Wait for work on the device to complete.
    result = hipEventSynchronize(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Measure elapsed runtime
    float runtime_ms = 0;
    result = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    runtime_ms /= static_cast<float>(options.iterations);

    double flops = 4.0;
    flops *= static_cast<double>(get<0>(problem_size));
    flops *= static_cast<double>(get<1>(problem_size));
    flops *= static_cast<double>(get<2>(problem_size));
    flops *= static_cast<double>(get<3>(problem_size));
    flops *= static_cast<double>(get<4>(problem_size));
    double tflops_s = flops * 1e-12 /*tera*/ / (runtime_ms * 1e-3 /*ms*/);
    example_result.tflops_s = tflops_s;
    example_result.runtime_ms = runtime_ms;

    result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Verify that the result is correct
    bool passed = true;
    if (options.verify) {
      passed = verify(problem_size);
      if (passed) example_result.verified = true;
    }
    
    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
      return example_result;
    }

    example_result.passed = true;

    return example_result;
  }

};

///////////////////////////////////////////////////////////////////////////////////////////////////

template<
  class TileShape,
  class... KernelOptions
>
struct BwdRunner {

  using Element = cutlass::half_t;
  using ElementAccumulator = float;

  // B H Q K D
  using ProblemShapeType = cute::tuple<int, int, int, int, int>;

  // using Operation = cutlass::fmha::device::FmhaBwd<Element, ElementAccumulator, TileShape, ActiveFusion, KernelOptions...>;
  
  // Just like forward
  using StrideQ = cute::tuple<int, int, int, _1>; // B H Q D
  using StrideK = cute::tuple<int, int, int, _1>; // B H K D
  using StrideV = cute::tuple<int, int, int, _1>; // B H K D
  using StrideO = cute::tuple<int, int, int, _1>; // B H Q D
  using StrideLSE = cute::tuple<int, int, _1>; // B H Q

  // Backwards specific
  using StrideDQ = cute::tuple<int, int, int, _1>; // B H Q D
  using StrideDK = cute::tuple<int, int, int, _1>; // B H K D
  using StrideDV = cute::tuple<int, int, int, _1>; // B H K D
  using StrideDO = cute::tuple<int, int, int, _1>; // B H Q D

  //
  // Data members
  //

  /// Initialization
  StrideQ stride_Q;
  StrideK stride_K;
  StrideV stride_V;
  StrideO stride_O;
  StrideLSE stride_LSE;

  StrideDQ stride_dQ;
  StrideDK stride_dK;
  StrideDV stride_dV;
  StrideDO stride_dO;

  uint64_t seed = 0;

  cutlass::DeviceAllocation<Element> block_Q;
  cutlass::DeviceAllocation<Element> block_K;
  cutlass::DeviceAllocation<Element> block_V;
  cutlass::DeviceAllocation<Element> block_O;
  cutlass::DeviceAllocation<ElementAccumulator> block_LSE;
  cutlass::DeviceAllocation<int> block_dQ_semaphore;

  static constexpr int kBlockM = 64;
  static constexpr int kBlockN = 128;

  // (aditya) NOTE: Specifically for Flash Attention
  cutlass::DeviceAllocation<ElementAccumulator> block_LSE_log2;
  cutlass::DeviceAllocation<ElementAccumulator> block_dsoftmax_sum;
  cutlass::DeviceAllocation<ElementAccumulator> block_dQ_accum;

  cutlass::DeviceAllocation<Element> block_dQ;
  cutlass::DeviceAllocation<Element> block_dK;
  cutlass::DeviceAllocation<Element> block_dV;
  cutlass::DeviceAllocation<Element> block_dO;

  cutlass::DeviceAllocation<Element> block_ref_dQ;
  cutlass::DeviceAllocation<Element> block_ref_dK;
  cutlass::DeviceAllocation<Element> block_ref_dV;

  //
  // Methods
  //
  bool verify(const ProblemShapeType& problem_size) {
    auto [B, H, Q, K, D] = problem_size;

    Tensor mQ = make_tensor(make_gmem_ptr(block_Q.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_Q), get<3>(stride_Q), make_stride(get<0>(stride_Q), get<1>(stride_Q))));

    Tensor mK = make_tensor(make_gmem_ptr(block_K.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_K), get<3>(stride_K), make_stride(get<0>(stride_K), get<1>(stride_K))));

    Tensor mV = make_tensor(make_gmem_ptr(block_V.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_V), get<3>(stride_V), make_stride(get<0>(stride_V), get<1>(stride_V))));

    Tensor mO = make_tensor(make_gmem_ptr(block_O.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_O), get<3>(stride_O), make_stride(get<0>(stride_O), get<1>(stride_O))));

    Tensor mLSE = make_tensor(make_gmem_ptr(block_LSE.get()),
      make_shape(Q, make_shape(B, H)),
      make_stride(get<2>(stride_LSE), make_stride(get<0>(stride_LSE), get<1>(stride_LSE))));

    Tensor mDQ = make_tensor(make_gmem_ptr(block_ref_dQ.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_dQ), get<3>(stride_dQ), make_stride(get<0>(stride_dQ), get<1>(stride_dQ))));

    Tensor mDK = make_tensor(make_gmem_ptr(block_ref_dK.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_dK), get<3>(stride_dK), make_stride(get<0>(stride_dK), get<1>(stride_dK))));

    Tensor mDV = make_tensor(make_gmem_ptr(block_ref_dV.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_dV), get<3>(stride_dV), make_stride(get<0>(stride_dV), get<1>(stride_dV))));

    Tensor mDO = make_tensor(make_gmem_ptr(block_dO.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_dO), get<3>(stride_dO), make_stride(get<0>(stride_dO), get<1>(stride_dO))));

    fmha_bwd_reference(problem_size, mQ, mK, mV, mO, mLSE, mDO, mDQ, mDK, mDV, DefaultFusion{});

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }
    // dim3 blk(1, 1, 1);
    // dim3 grd(1, 1, 1);
    // print_dQ_dK_dV<<<grd, blk>>>(block_dQ.get(), block_ref_dQ.get(),
    //                              block_dK.get(), block_ref_dK.get(),
    //                              block_dV.get(), block_ref_dV.get());
    // 
    result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Reference kernel failed. Last CUDA error: "
                << hipGetErrorString(result) << std::endl;
      return false;
    }

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    double max_diff = 0;
    double mean_diff = 0;
    reference_abs_diff(block_dQ, block_ref_dQ, max_diff, mean_diff);
    std::cout << "dQ max_diff=" << max_diff << " mean_diff=" << mean_diff << std::endl;
    bool passed_dQ = (max_diff < 1e-2) && (mean_diff < 1e-3);
    if (! passed_dQ) {
      std::cerr << "failed dQ: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    reference_abs_diff(block_dK, block_ref_dK, max_diff, mean_diff);
    std::cout << "dK max_diff=" << max_diff << " mean_diff=" << mean_diff << std::endl;
    bool passed_dK = (max_diff < 1e-2) && (mean_diff < 1e-3);
    if (! passed_dK) {
      std::cerr << "failed dK: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    reference_abs_diff(block_dV, block_ref_dV, max_diff, mean_diff);
    std::cout << "dV max_diff=" << max_diff << " mean_diff=" << mean_diff << std::endl;
    bool passed_dV = (max_diff < 1e-2) && (mean_diff < 1e-3);
    if (! passed_dV) {
      std::cerr << "failed dV: max diff " << max_diff 
                << " mean " << mean_diff << std::endl;
    }

    std::cout << "passed_dQ: " << passed_dQ << std::endl;
    std::cout << "passed_dK: " << passed_dK << std::endl;
    std::cout << "passed_dV: " << passed_dV << std::endl;

    if (!(passed_dQ && passed_dK && passed_dV)) {
      dim3 blk(1, 1, 1);
      dim3 grd(1, 1, 1);
      print_dQ_dK_dV<<<grd, blk>>>(block_dQ.get(), block_ref_dQ.get(),
                                 block_dK.get(), block_ref_dK.get(),
                                 block_dV.get(), block_ref_dV.get());
    }



    return passed_dQ && passed_dK && passed_dV;
  }

  /// Initialize operands to be used in the GEMM and reference GEMM
  void initialize(const ProblemShapeType& problem_size) {
    auto [B, H, Q, K, D] = problem_size;
    // Q = cutlass::round_up(Q, 8);  // Alignment
    auto Q_rounded = cutlass::round_up(Q, kBlockM);

    auto shape_QO = cute::make_shape(B, H, Q, D);
    auto shape_KV = cute::make_shape(B, H, K, D);
    auto shape_LSE = cute::make_shape(B, H, Q);
    auto shape_LSE_rounded = cute::make_shape(B, H, Q_rounded);
    auto shape_dQ_accum = cute::make_shape(B, H, Q_rounded * D);
    auto shape_dQ_semaphore = cute::make_shape((Q + kBlockM - 1) / kBlockM, B, H);

    stride_Q = cute::compact_row_major(shape_QO);
    stride_K = cute::compact_row_major(shape_KV);
    stride_V = cute::compact_row_major(shape_KV);
    stride_O = cute::compact_row_major(shape_QO);
    stride_LSE = cute::compact_row_major(shape_LSE);

    stride_dQ = stride_Q;
    stride_dK = stride_K;
    stride_dV = stride_V;
    stride_dO = stride_O;

    block_Q.reset(size(shape_QO));
    block_K.reset(size(shape_KV));
    block_V.reset(size(shape_KV));
    block_O.reset(size(shape_QO));
    block_LSE.reset(size(shape_LSE));
    block_LSE_log2.reset(size(shape_LSE_rounded));
    block_dsoftmax_sum.reset(size(shape_LSE_rounded));
    block_dQ_accum.reset(size(shape_dQ_accum));
    block_dQ_semaphore.reset(size(shape_dQ_semaphore));

    block_dQ.reset(size(shape_QO));
    block_dK.reset(size(shape_KV));
    block_dV.reset(size(shape_KV));
    block_dO.reset(size(shape_QO));

    block_ref_dQ.reset(size(shape_QO));
    block_ref_dK.reset(size(shape_KV));
    block_ref_dV.reset(size(shape_KV));

    initialize_block(block_Q, seed + 2023, false);
    initialize_block(block_K, seed + 2022, false);
    initialize_block(block_V, seed + 2021, false);
    initialize_block(block_dO, seed + 2020, false);

    Tensor mQ = make_tensor(make_gmem_ptr(block_Q.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_Q), get<3>(stride_Q), make_stride(get<0>(stride_Q), get<1>(stride_Q))));

    Tensor mK = make_tensor(make_gmem_ptr(block_K.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_K), get<3>(stride_K), make_stride(get<0>(stride_K), get<1>(stride_K))));

    Tensor mV = make_tensor(make_gmem_ptr(block_V.get()),
      make_shape(K, D, make_shape(B, H)),
      make_stride(get<2>(stride_V), get<3>(stride_V), make_stride(get<0>(stride_V), get<1>(stride_V))));

    Tensor mO = make_tensor(make_gmem_ptr(block_O.get()),
      make_shape(Q, D, make_shape(B, H)),
      make_stride(get<2>(stride_O), get<3>(stride_O), make_stride(get<0>(stride_O), get<1>(stride_O))));

    Tensor mLSE = make_tensor(make_gmem_ptr(block_LSE.get()),
      make_shape(Q, make_shape(B, H)),
      make_stride(get<2>(stride_LSE), make_stride(get<0>(stride_LSE), get<1>(stride_LSE))));

    fmha_reference(problem_size, mQ, mK, mV, mO, mLSE, DefaultFusion{});
  }

  ExampleResult run(const Options& options) {
    ProblemShapeType problem_size = ProblemShapeType{options.b, options.h, options.q, options.k, options.d};

    initialize(problem_size);

    // Set params here
    Flash_bwd_params params = set_flash_bwd_params_for_testing(
      problem_size,
      block_Q.get(),
      block_K.get(),
      block_V.get(),
      block_O.get(),
      block_LSE.get(),
      block_dQ.get(),
      block_dK.get(),
      block_dV.get(),
      block_dO.get(),
      block_dsoftmax_sum.get(),
      block_LSE_log2.get(),
      block_dQ_accum.get(),
      block_dQ_semaphore.get(),
      stride_Q,
      stride_K,
      stride_V,
      stride_O,
      stride_LSE,
      (float) 1.0 / sqrt(double(options.d)) /* softmax_scale */ 
    );

    ExampleResult example_result;

    example_result.smem_size = 0; //Operation::Operation::Kernel::SharedStorageSize;

    hipError_t result = hipDeviceSynchronize();
    cutlass::Status status;
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamDefault);

    // auto flash_bwd_fn = run_flash_bwd<
    //   /* Arch= */ 80,
    //   /* kHeadDim= */ 128,
    //   /* kBlockM= */ kBlockM,
    //   /* kBlockN= */ kBlockN,
    //   /* Element= */ cutlass::half_t,
    //   /* Is_causal= */ false,
    //   /* Is_local= */ false,
    //   /* Has_softcap= */ false,
    //   /* Varlen= */ false,
    //   /* Deterministic= */ false,
    //   /* GQA= */ false,
    //   /* Stages_dO= */ 2,
    //   /* Stages_dS_or_QSm80= */ 2,
    //   /* SdP_swapAB= */ true,
    //   /* dKV_swapAB= */ false,
    //   /* dQ_swapAB= */ false,
    //   /* NumMmaWarpGroups= */ 2,
    //   /* AtomLayoutMSdP= */ 2,
    //   /* AtomLayoutNdKV= */ 2,
    //   /* AtomLayoutMdQ= */ 2,
    //   /* V_in_regs= */ false
    // >;

    auto flash_bwd_fn = run_flash_bwd<
      /* Arch= */ 80,
      /* kHeadDim= */ 128,
      /* kBlockM= */ kBlockM,
      /* kBlockN= */ kBlockN,
      /* Element= */ cutlass::half_t,
      /* Deterministic= */ false,
      /* GQA= */ false,
      /* Stages_dO= */ 2,
      /* Stages_dS_or_QSm80= */ 2,
      /* SdP_swapAB= */ true,
      /* dKV_swapAB= */ false,
      /* dQ_swapAB= */ false,
      /* NumMmaWarpGroups= */ 2,
      /* AtomLayoutMSdP= */ 2,
      /* AtomLayoutNdKV= */ 2,
      /* AtomLayoutMdQ= */ 2,
      /* V_in_regs= */ false
    >;

    hipMemset(block_dQ.get(), 0, block_dQ.size() * sizeof(Element));
    hipMemset(block_dQ_accum.get(), 0, block_dQ_accum.size() * sizeof(ElementAccumulator));
    hipMemset(block_dK.get(), 0, block_dK.size() * sizeof(Element));
    hipMemset(block_dV.get(), 0, block_dV.size() * sizeof(Element));

    hipEvent_t events[2];

    for (auto & event : events) {
      result = hipEventCreate(&event);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result) << std::endl;
        return example_result;
      }
    }

    // Record an event at the start of a series of GEMMs
    result = hipEventRecord(events[0]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    for (int i = 0; i < options.iterations; i++) {
      flash_bwd_fn(params, stream);
      // verify(problem_size);
      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to launch the CUTLASS kernel. Last CUDA error is: "
                  << hipGetErrorString(hipGetLastError()) << std::endl;
        return example_result;
      }
    }

    //
    // Stop profiling loop
    //

    // Record an event when the GEMMs are complete
    result = hipEventRecord(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Wait for work on the device to complete.
    result = hipEventSynchronize(events[1]);
    if (result != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    hipMemset(block_dQ.get(), 0, block_dQ.size() * sizeof(Element));
    hipMemset(block_dQ_accum.get(), 0, block_dQ.size() * sizeof(ElementAccumulator));
    hipMemset(block_dK.get(), 0, block_dK.size() * sizeof(Element));
    hipMemset(block_dV.get(), 0, block_dV.size() * sizeof(Element));
    flash_bwd_fn(params, stream);
    result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "hipDeviceSynchronize() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }
    

    // Measure elapsed runtime
    float runtime_ms = 0;
    result = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    runtime_ms /= static_cast<float>(options.iterations);

    double flops = 10.0;
    flops *= static_cast<double>(get<0>(problem_size));
    flops *= static_cast<double>(get<1>(problem_size));
    flops *= static_cast<double>(get<2>(problem_size));
    flops *= static_cast<double>(get<3>(problem_size));
    flops *= static_cast<double>(get<4>(problem_size));
    double tflops_s = flops * 1e-12 /*tera*/ / (runtime_ms * 1e-3 /*ms*/);
    example_result.tflops_s = tflops_s;
    example_result.runtime_ms = runtime_ms;

    result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Error running the CUTLASS kernel. Last CUDA error is: "
                << hipGetErrorString(result) << std::endl;
      return example_result;
    }

    // Verify that the result is correct
    bool passed = true;
    if (options.verify) {
      passed = verify(problem_size);
      if (passed) example_result.verified = true;
    }
    
    if (!passed) {
      std::cerr << "Reference check failed" << std::endl;
      return example_result;
    }

    example_result.passed = true;

    return example_result;
  }

};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Helper to print a description of the example run and its result
void print_result(const std::string& description, ExampleResult result, bool verbose) {
  std::ios fmt(nullptr);
  fmt.copyfmt(std::cout);
  std::cout << (result.passed ? (result.verified ? " [OK]  " : " [--] ") : "[FAIL] ");
  std::cout << std::setw(32) << std::left << description;
  std::cout.copyfmt(fmt);
  std::cout <<std::endl;
  std::cout << " \t " << "Throughput: " << result.tflops_s << " TFLOPS/s" << std::endl;
  std::cout << " \t " << "Runtime:    " << result.runtime_ms << " ms" << std::endl;
  std::cout << " \t "<<  "SMEM size:  " << result.smem_size << "b" << std::endl;
}

///////////////////////////////////////////////////////////////////////////////////////////////////

// using KernelTma = cutlass::gemm::KernelTma;
// using KernelCooperative = cutlass::gemm::KernelTmaWarpSpecializedCooperative;
// using KernelPingpong = cutlass::gemm::KernelTmaWarpSpecializedPingpong;

///////////////////////////////////////////////////////////////////////////////////////////////////

void run_fwd_32(Options const & options) {
  auto run = [&](auto shape, const char* name, auto... kernel_options) {
    FwdRunner<decltype(shape), decltype(kernel_options)...> runner;
    auto result = runner.run(options);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _32;

  run(Shape< _64, _128, HeadDim>{}, "tma 64x128x32");
  run(Shape< _128, _64, HeadDim>{}, "tma ws cooperative 128x64x32");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

void run_fwd_64(Options const & options) {
  auto run = [&](auto shape, const char* name, auto... kernel_options) {
    FwdRunner<decltype(shape), decltype(kernel_options)...> runner;
    auto result = runner.run(options);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _64;

  run(Shape< _64, _128, HeadDim>{}, "tma 64x128x64");
  run(Shape< _128, _64, HeadDim>{}, "tma ws cooperative 128x64x64");
  run(Shape< _128, _64, HeadDim>{}, "tma ws ping-pong 128x64x64");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

void run_fwd_128(Options const & options) {
  auto run = [&](auto shape, const char* name, auto... kernel_options) {
    FwdRunner<decltype(shape), decltype(kernel_options)...> runner;
    auto result = runner.run(options);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _128;

  run(Shape<_128, _128, HeadDim>{}, "tma ws cooperative 128x128x128");
  // run(Shape<_128, _128, HeadDim>{}, "tma ws ping-pong 128x128x128");
#ifdef FP8
  // run(Shape<_128, _256, HeadDim>{}, "tma ws cooperative 128x256x128 acc fp16", Option<Tag::kAccQK, cutlass::half_t>{});
  // run(Shape<_128, _256, HeadDim>{}, "tma ws cooperative 128x256x128 acc fp32");
#endif
}

///////////////////////////////////////////////////////////////////////////////////////////////////

void run_fwd_256(Options const & options) {
  auto run = [&](auto shape, const char* name, auto... kernel_options) {
    FwdRunner<decltype(shape), decltype(kernel_options)...> runner;
    auto result = runner.run(options);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _256;

#ifdef FP8
  run(Shape<_128, _128, HeadDim>{}, "tma ws cooperative 128x128x256");
  run(Shape<_128, _128, HeadDim>{}, "tma ws ping-pong 128x128x256");
#else
  run(Shape<_128, _64, HeadDim>{}, "tma ws cooperative 128x64x256");
#endif
}

///////////////////////////////////////////////////////////////////////////////////////////////////

void run_bwd_32(Options const & options) {
  auto run = [&](auto shape, const char* name, auto... kernel_options) {
    BwdRunner<decltype(shape), decltype(kernel_options)...> runner;
    auto result = runner.run(options);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _32;

  run(Shape< _64, _128, HeadDim>{}, "tma ws cooperative 64x128x32");
  run(Shape<_128, _128, HeadDim>{}, "tma ws cooperative 128x128x32");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

void run_bwd_64(Options const & options) {
  auto run = [&](auto shape, const char* name, auto... kernel_options) {
    BwdRunner<decltype(shape), decltype(kernel_options)...> runner;
    auto result = runner.run(options);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _64;

  run(Shape< _64, _128, HeadDim>{}, "tma ws cooperative 64x128x64");
  run(Shape<_128, _128, HeadDim>{}, "tma ws cooperative 128x128x64");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

void run_bwd_128(Options const & options) {
  auto run = [&](auto shape, const char* name, auto... kernel_options) {
    BwdRunner<decltype(shape), decltype(kernel_options)...> runner;
    auto result = runner.run(options);
    print_result(name, result, options.verbose);
  };

  using HeadDim = _128;

  run(Shape<_64, _128, HeadDim>{}, "tma ws cooperative 64x128x128");
}

///////////////////////////////////////////////////////////////////////////////////////////////////

// #endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

///////////////////////////////////////////////////////////////////////////////////////////////////

int main_single(int argc, char const **args) {

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  //
  // Parse options
  //

  Options options;

  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.error) {
    std::cerr << "Aborting execution." << std::endl;
    return -1;
  }

// #if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  //
  // Run examples
  //

  std::cout << "###### B " << options.b << " H " << options.h << " Q " << options.q << " K " << options.k << " D " << options.d << " ";
  std::cout << (options.bwd ? "Backward" : "Forward") << " " << (options.causal ? "Causal" : "Full") << " ";

  // with_fusion([&](auto fusion) {
  if (options.bwd) {
#ifndef FP8
    if (options.d <= 32) {
      run_bwd_32(options);
    } else if (options.d <= 64) {
      run_bwd_64(options);
    } else if (options.d <= 128) {
      run_bwd_128(options);
    } else
#endif
   {
#ifdef FP8
      std::cout << "Backward is not implemented for FP8." << std::endl;
#else
      std::cout << "No backward kernel instantiated for d=" << options.d << std::endl;
#endif
    }
  } else {
#ifndef FP8
    if (options.d <= 32) {
      run_fwd_32(options);
    } else
    if (options.d <= 64) {
      run_fwd_64(options);
    } else
#endif
    if (options.d <= 128) {
      run_fwd_128(options);
    } else
    if (options.d <= 256) {
      run_fwd_256(options);
    }
    else {
      std::cout << "No forward kernel instantiated for d=" << options.d << std::endl;
    }
  }
// };
  //);
// #endif // defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char const **args) {
  std::vector<std::string> full_arguments(args, args + argc);

  int result = 0;

  bool recursed = false;
  for (size_t i = 1; i < full_arguments.size(); i++) {
    if (full_arguments[i].find(',') != std::string::npos) {
      auto arg = full_arguments[i];
      size_t eq_pos = arg.find('=');
      std::string prefix = eq_pos == std::string::npos ? "" : arg.substr(0, eq_pos+1);
      std::string rest = eq_pos == std::string::npos ? arg : arg.substr(eq_pos+1);
      for (;;) {
        size_t comma_pos = rest.find(',');
        std::string current = rest.substr(0, comma_pos);
        full_arguments[i] = prefix + current;
        std::vector<const char*> next_args;
        for (auto& elem : full_arguments) { next_args.push_back(elem.data()); }
        main(argc, next_args.data());
        if (comma_pos == std::string::npos) break;
        rest = rest.substr(comma_pos+1);
      }
      recursed = true;
      break;
    }
  }

  if (! recursed) {
    main_single(argc, args);
  }

  return result;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
