/***************************************************************************************************
 * Copyright (c) 2022-2025 Ali Hassani.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 *all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 **************************************************************************************************/
/*! \file
    \brief Hopper FNA Torch interface
*/

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>

#include <natten/helpers.h>
#include <natten/natten.h>

#include <natten/cuda/hopper_fmha_fna.h>
#ifdef NATTEN_WITH_CUTLASS
#include <natten_autogen/cuda/hopper_fna/interface.h>
#include <natten/cuda/fna_hopper/fna_forward.cuh>

template <typename StdTuple>
auto std_tuple_to_cute_tuple(StdTuple a) {
  static_assert(
      std::tuple_size_v<StdTuple> > 0 && std::tuple_size_v<StdTuple> < 4);

  if constexpr (std::tuple_size_v<StdTuple> == 1) {
    return cute::make_tuple(std::get<0>(a));
  } else if constexpr (std::tuple_size_v<StdTuple> == 2) {
    return cute::make_tuple(std::get<0>(a), std::get<1>(a));
  } else {
    return cute::make_tuple(std::get<0>(a), std::get<1>(a), std::get<2>(a));
  }
}
#endif

namespace natten {

#ifdef NATTEN_WITH_CUTLASS
#ifdef NATTEN_WITH_HOPPER_FNA
namespace {} // namespace
#endif
#endif

template <class StdNADim, class StdCausal>
void hopper_fna_generic_forward(
    at::Tensor& out,
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value,
    const at::optional<at::Tensor>& logsumexp,
    const StdNADim& kernel_size,
    const StdNADim& stride_,
    const StdNADim& dilation_,
    const StdCausal& is_causal_,
    float attn_scale,
    const StdNADim& q_shape_, // after token permute and padding
    const StdNADim& kv_shape_, // after token permute and padding
    const StdNADim& qkv_shape_, // before token permute and padding
    const StdNADim& query_tile_shape_,
    const StdNADim& key_tile_shape_,
    int kernel_type) {
  static_assert(
      std::tuple_size_v<StdNADim> > 0 && std::tuple_size_v<StdNADim> < 4);
  static constexpr int kNADim = std::tuple_size_v<StdNADim>;
  static_assert(std::tuple_size_v<StdCausal> == kNADim);

#ifdef NATTEN_WITH_CUTLASS
#ifdef NATTEN_WITH_HOPPER_FNA
  AssertDimsAre128BitAligned(query, value);

  CHECK_CONTIGUOUS(query);
  CHECK_CONTIGUOUS(key);
  CHECK_CONTIGUOUS(value);
  CHECK_CONTIGUOUS(out);

  CHECK_CUDA(query);
  CHECK_CUDA(key);
  CHECK_CUDA(value);
  CHECK_CUDA(out);

  at::cuda::OptionalCUDAGuard device_guard(query.device());

  CheckArgs(kernel_size, stride_, dilation_);
  CheckIfPropertiesMatch(query, key, value);

  // NOTE (alih): q and kv might have slightly different shapes because we're
  // padding to multiples of the tile shape. Seqlen mode must be flattened.
  CheckIfTensorShapesMatch<1>(query, out);
  CheckIfTensorShapesMatch<1>(key, value);

  TORCH_CHECK(query.dim() == 4, "Tensors must be 4-D.");
  TORCH_CHECK(key.dim() == 4, "Tensors must be 4-D.");
  TORCH_CHECK(value.dim() == 4, "Tensors must be 4-D.");

  int batch_size = query.size(0);
  int seqlen_q = query.size(1);
  int seqlen_kv = key.size(1);
  int heads = query.size(2);
  int dim = query.size(3);

  CheckArgsAgainstDim(qkv_shape_, kernel_size, dilation_);

  auto qkv_shape = std_tuple_to_cute_tuple(qkv_shape_);
  auto q_shape = std_tuple_to_cute_tuple(q_shape_);
  auto kv_shape = std_tuple_to_cute_tuple(kv_shape_);

  auto query_tile_shape = std_tuple_to_cute_tuple(query_tile_shape_);
  auto key_tile_shape = std_tuple_to_cute_tuple(key_tile_shape_);

  auto window_size = std_tuple_to_cute_tuple(kernel_size);
  auto stride = std_tuple_to_cute_tuple(stride_);
  auto dilation = std_tuple_to_cute_tuple(dilation_);
  auto is_causal = std_tuple_to_cute_tuple(is_causal_);

  TORCH_CHECK(
      size(q_shape) == seqlen_q,
      "Q's sequence length (q.shape[1]) must match the size of Q shape.");
  TORCH_CHECK(
      size(kv_shape) == seqlen_kv,
      "KV's sequence length ({k,v}.shape[1]) must match the size of KV shape.");

  if (logsumexp.has_value()) {
    CheckLogSumExp<1>(out, logsumexp.value());
    CHECK_CUDA(logsumexp.value());
  }

  TORCH_CHECK(
      dim == 32 || dim == 64 || dim == 128 || dim == 256,
      "Hopper FNA only supports head dims 32, 64, 128, and 256 for now.");

  hipDeviceProp_t* device_props =
      at::cuda::getDeviceProperties(query.device().index());
  const int cc = device_props->major * 10 + device_props->minor;
  TORCH_CHECK(
      cc == 90,
      "This operation can only run on the Hopper architecture (SM90).");

  TORCH_CHECK(
      query.scalar_type() == key.scalar_type() &&
          query.scalar_type() == value.scalar_type() &&
          query.scalar_type() == out.scalar_type(),
      "Query, key, value, and output must match in dtype.");

  TORCH_CHECK(
      query.scalar_type() == torch::kFloat16 ||
          query.scalar_type() == torch::kBFloat16,
      "Only FP16/BF16 is supported for now.");

  int device_id = query.device().index();
  auto hip_stream = at::cuda::getCurrentCUDAStream(device_id);

  auto kernel_type_ =
      natten::cuda::hopper::kernel_type_int_to_enum_type(kernel_type);
  TORCH_CHECK(
      kernel_type_ != natten::cuda::hopper::HopperKernelSchedule::Invalid,
      "Got invalid kernel_type argument.");

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  DISPATCH_HOPPER_FNA_FORWARD(
      kNADim,
      query.scalar_type(),
      dim,
      is_causal,
      query_tile_shape,
      key_tile_shape,
      kernel_type_,
      static_cast<void*>(query.data_ptr()),
      static_cast<void*>(key.data_ptr()),
      static_cast<void*>(value.data_ptr()),
      static_cast<void*>(out.data_ptr()),
      logsumexp.has_value() ? static_cast<void*>(logsumexp.value().data_ptr())
                            : nullptr,
      batch_size,
      seqlen_q,
      seqlen_kv,
      heads,
      dim,
      q_shape,
      kv_shape,
      qkv_shape,
      window_size,
      stride,
      dilation,
      device_id,
      attn_scale,
      hip_stream,
      query.options());

#else
  TORCH_CHECK(
      false,
      "libnatten was not compiled with CUTLASS_ARCH_MMA_SM90_SUPPORTED.");
#endif
#else
  TORCH_CHECK(false, "libnatten was not compiled for Hopper (SM90).");
#endif
#else
  TORCH_CHECK(false, "libnatten not compiled with CUTLASS.");
#endif
}

void hopper_na1d_forward(
    at::Tensor& out,
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value,
    const at::optional<at::Tensor>& logsumexp,
    const std::tuple<int32_t>& kernel_size,
    const std::tuple<int32_t>& stride,
    const std::tuple<int32_t>& dilation,
    const std::tuple<bool>& is_causal,
    float attn_scale,
    const std::tuple<int32_t>& q_shape,
    const std::tuple<int32_t>& kv_shape,
    const std::tuple<int32_t>& qkv_shape,
    const std::tuple<int32_t>& query_tile_shape,
    const std::tuple<int32_t>& key_tile_shape,
    int kernel_type) {
  TORCH_CHECK(query.dim() == 4, "Tensors must be 4-D.");

  hopper_fna_generic_forward(
      out,
      query,
      key,
      value,
      logsumexp,
      kernel_size,
      stride,
      dilation,
      is_causal,
      attn_scale,
      q_shape,
      kv_shape,
      qkv_shape,
      query_tile_shape,
      key_tile_shape,
      kernel_type);
}

void hopper_na2d_forward(
    at::Tensor& out,
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value,
    const at::optional<at::Tensor>& logsumexp,
    const std::tuple<int32_t, int32_t>& kernel_size,
    const std::tuple<int32_t, int32_t>& stride,
    const std::tuple<int32_t, int32_t>& dilation,
    const std::tuple<bool, bool>& is_causal,
    float attn_scale,
    const std::tuple<int32_t, int32_t>& q_shape,
    const std::tuple<int32_t, int32_t>& kv_shape,
    const std::tuple<int32_t, int32_t>& qkv_shape,
    const std::tuple<int32_t, int32_t>& query_tile_shape,
    const std::tuple<int32_t, int32_t>& key_tile_shape,
    int kernel_type) {
  TORCH_CHECK(query.dim() == 4, "Tensors must be 4-D.");

  hopper_fna_generic_forward(
      out,
      query,
      key,
      value,
      logsumexp,
      kernel_size,
      stride,
      dilation,
      is_causal,
      attn_scale,
      q_shape,
      kv_shape,
      qkv_shape,
      query_tile_shape,
      key_tile_shape,
      kernel_type);
}

void hopper_na3d_forward(
    at::Tensor& out,
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value,
    const at::optional<at::Tensor>& logsumexp,
    const std::tuple<int32_t, int32_t, int32_t>& kernel_size,
    const std::tuple<int32_t, int32_t, int32_t>& stride,
    const std::tuple<int32_t, int32_t, int32_t>& dilation,
    const std::tuple<bool, bool, bool>& is_causal,
    float attn_scale,
    const std::tuple<int32_t, int32_t, int32_t>& q_shape,
    const std::tuple<int32_t, int32_t, int32_t>& kv_shape,
    const std::tuple<int32_t, int32_t, int32_t>& qkv_shape,
    const std::tuple<int32_t, int32_t, int32_t>& query_tile_shape,
    const std::tuple<int32_t, int32_t, int32_t>& key_tile_shape,
    int kernel_type) {
  TORCH_CHECK(query.dim() == 4, "Tensors must be 4-D.");

  hopper_fna_generic_forward(
      out,
      query,
      key,
      value,
      logsumexp,
      kernel_size,
      stride,
      dilation,
      is_causal,
      attn_scale,
      q_shape,
      kv_shape,
      qkv_shape,
      query_tile_shape,
      key_tile_shape,
      kernel_type);
}

} // namespace natten
