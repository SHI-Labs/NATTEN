/***************************************************************************************************
 * Copyright (c) 2022-2025 Ali Hassani.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 *all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 **************************************************************************************************/
/*! \file
    \brief Hopper FMHA interface
*/

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <torch/extension.h>

#include <natten/helpers.h>
#include <natten/natten.h>

#ifdef NATTEN_WITH_CUTLASS
#include <natten_autogen/cuda/hopper_fmha/interface.h>
#include <natten/cuda/fmha_hopper/fmha_forward.cuh>
#endif

namespace natten {

#ifdef NATTEN_WITH_CUTLASS
#ifdef NATTEN_WITH_HOPPER_FNA
namespace {

auto kernel_type_int_to_enum_type(int kernel_type) {
  switch (kernel_type) {
    case 0:
      return natten::cuda::fmha_hopper::HopperFmhaKernelType::NonPersistent;
    case 1:
      return natten::cuda::fmha_hopper::HopperFmhaKernelType::WSCooperative;
    case 2:
      return natten::cuda::fmha_hopper::HopperFmhaKernelType::WSPingpong;
  }
  std::cerr
      << "Invalid value for argument kernel_type; expected either 0, 1, or 2, got "
      << kernel_type << ".\n";
  return natten::cuda::fmha_hopper::HopperFmhaKernelType::Invalid;
}

} // namespace
#endif
#endif

void hopper_fmha_forward(
    at::Tensor& out,
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value,
    const at::optional<at::Tensor>& logsumexp,
    float attn_scale,
    int query_tile_size,
    int key_tile_size,
    int kernel_type) {
#ifdef NATTEN_WITH_CUTLASS
#ifdef NATTEN_WITH_HOPPER_FNA
  AssertDimsAre128BitAligned(query, value);

  CHECK_CONTIGUOUS(query);
  CHECK_CONTIGUOUS(key);
  CHECK_CONTIGUOUS(value);
  CHECK_CONTIGUOUS(out);

  CHECK_CUDA(query);
  CHECK_CUDA(key);
  CHECK_CUDA(value);
  CHECK_CUDA(out);

  at::cuda::OptionalCUDAGuard device_guard(query.device());

  CheckIfPropertiesMatch(query, key, value);
  CheckIfTensorShapesMatch<1>(key, value);
  CheckIfTensorShapesMatch<1>(query, out);

  TORCH_CHECK(query.dim() == 4, "Tensors must be 4-D.");
  TORCH_CHECK(key.dim() == 4, "Tensors must be 4-D.");
  TORCH_CHECK(value.dim() == 4, "Tensors must be 4-D.");

  int batch_size = query.size(0);
  int seqlen_q = query.size(1);
  int seqlen_kv = key.size(1);
  int heads = query.size(2);
  int dim = query.size(3);

  if (logsumexp.has_value()) {
    CheckLogSumExp<1>(out, logsumexp.value());
    CHECK_CUDA(logsumexp.value());
  }

  TORCH_CHECK(
      dim == 32 || dim == 64 || dim == 128 || dim == 256,
      "Hopper FMHA only supports head dims 32, 64, 128, and 256 for now.");

  hipDeviceProp_t* device_props =
      at::cuda::getDeviceProperties(query.device().index());
  const int cc = device_props->major * 10 + device_props->minor;
  TORCH_CHECK(
      cc == 90,
      "This operation can only run on the Hopper architecture (SM90).");

  TORCH_CHECK(
      query.scalar_type() == key.scalar_type() &&
          query.scalar_type() == value.scalar_type() &&
          query.scalar_type() == out.scalar_type(),
      "Query, key, value, and output must match in dtype.");

  TORCH_CHECK(
      query.scalar_type() == torch::kFloat16 ||
          query.scalar_type() == torch::kBFloat16,
      "Only FP16/BF16 is supported for now.");

  int device_id = query.device().index();
  auto hip_stream = at::cuda::getCurrentCUDAStream(device_id);

  auto kernel_type_ = kernel_type_int_to_enum_type(kernel_type);
  TORCH_CHECK(
      kernel_type_ != natten::cuda::fmha_hopper::HopperFmhaKernelType::Invalid,
      "Got invalid kernel_type argument.");

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)

  DISPATCH_HOPPER_FMHA_FORWARD(
      query.scalar_type(),
      dim,
      query_tile_size,
      key_tile_size,
      kernel_type_,
      static_cast<void*>(query.data_ptr()),
      static_cast<void*>(key.data_ptr()),
      static_cast<void*>(value.data_ptr()),
      static_cast<void*>(out.data_ptr()),
      logsumexp.has_value() ? static_cast<void*>(logsumexp.value().data_ptr())
                            : nullptr,
      batch_size,
      seqlen_q,
      seqlen_kv,
      heads,
      dim,
      device_id,
      attn_scale,
      hip_stream,
      query.options());

#else
  TORCH_CHECK(
      false,
      "libnatten was not compiled with CUTLASS_ARCH_MMA_SM90_SUPPORTED.");
#endif
#else
  TORCH_CHECK(false, "libnatten was not compiled for Hopper (SM90).");
#endif
#else
  TORCH_CHECK(false, "libnatten not compiled with CUTLASS.");
#endif
}

} // namespace natten
